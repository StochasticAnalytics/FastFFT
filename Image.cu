#include "hip/hip_runtime.h"
#include "Image.cuh"

template < class wanted_real_type, class wanted_complex_type >
Image<wanted_real_type, wanted_complex_type>::Image(short4 wanted_size)
{

  size = wanted_size;

  if (wanted_size.x % 2 == 0) padding_jump_value = 2;
  else padding_jump_value = 1;

  size.w = (size.x + padding_jump_value) / 2;

  is_in_memory = false;
  is_in_real_space = true;
  is_cufft_planned = false;
  is_fftw_planned = false;



}

template < class wanted_real_type, class wanted_complex_type >
Image<wanted_real_type, wanted_complex_type>::~Image()
{
  if (is_in_memory) 
  {
    fftwf_free(real_values);
     is_in_memory = false;
  } 
  if (is_fftw_planned)
  {
    fftwf_destroy_plan(plan_fwd);
    fftwf_destroy_plan(plan_bwd);
    is_fftw_planned = false;
  }
  if (is_cufft_planned)
  {
    cudaErr_img(hipfftDestroy(cuda_plan_inverse));
    cudaErr_img(hipfftDestroy(cuda_plan_forward));
    is_cufft_planned = false;
  }

  if (is_set_clip_into_mask)
  {
    cudaErr_img(hipFree(clipIntoMask));
    is_set_clip_into_mask = false;
  }
}

template < class wanted_real_type, class wanted_complex_type >
void Image<wanted_real_type, wanted_complex_type>::SetClipIntoMask(short4 input_size, short4 output_size)
{
  // Allocate the mask
  int pjv;
  int address = 0;
  int n_values = output_size.w*2*output_size.y;
  bool* tmpMask = new bool[n_values];

  precheck_img
  cudaErr_img(hipMalloc(&clipIntoMask, (n_values)*sizeof(bool)));
  postcheck_img

  if (output_size.x % 2 == 0) pjv = 2;
  else pjv = 1;

  for (int j = 0 ; j < output_size.y ; j++)
  {
    for (int i = 0 ; i < output_size.x ; i++)
    {
      if (i < input_size.x && j < input_size.y) tmpMask[address] = true;
      else tmpMask[address] = false;
      address++;
    }
    tmpMask[address] = false; 
    address++;
    if (pjv > 1) {tmpMask[address] = false;  address++;}
  }


  cudaErr_img(hipMemcpyAsync(clipIntoMask, tmpMask, n_values*sizeof(bool),hipMemcpyHostToDevice,hipStreamPerThread));
  hipStreamSynchronize(hipStreamPerThread);

  delete [] tmpMask;
  is_set_clip_into_mask = true;


}

// template < class wanted_real_type, class wanted_complex_type >
// Image<class wanted_real_type, class wanted_complex_type >::Image()
// {

// }

template < class wanted_real_type, class wanted_complex_type >
void Image<wanted_real_type, wanted_complex_type>::Allocate(bool set_fftw_plan)
{
  real_values = (wanted_real_type *) fftwf_malloc(sizeof(wanted_real_type) * real_memory_allocated);
  complex_values = (wanted_complex_type*) real_values;  // Set the complex_values to point at the newly allocated real values;

  // This will only work for single precision, should probably add a check on this, but for now rely on the user to make sure they are using single precision.
  if (set_fftw_plan)
  {
    plan_fwd = fftwf_plan_dft_r2c_3d(size.z, size.y, size.x, real_values, reinterpret_cast<fftwf_complex*>(complex_values), FFTW_ESTIMATE);
    plan_bwd = fftwf_plan_dft_c2r_3d(size.z, size.y, size.x, reinterpret_cast<fftwf_complex*>(complex_values), real_values, FFTW_ESTIMATE);
    is_fftw_planned = true;
  }


  is_in_memory = true;
}


template < class wanted_real_type, class wanted_complex_type >
void Image<wanted_real_type, wanted_complex_type>::FwdFFT()
{
  if (is_fftw_planned)
  {
    // Now let's do the forward FFT on the host and check that the result is correct.
    fftwf_execute_dft_r2c(plan_fwd, real_values, reinterpret_cast<fftwf_complex*>(complex_values));
  }
  else {std::cout << "Error: FFTW plan not set up." << std::endl; exit(1);}

  is_in_real_space = false;

}

template < class wanted_real_type, class wanted_complex_type >
void Image<wanted_real_type, wanted_complex_type>::InvFFT()
{
  if (is_fftw_planned)
  {
    // Now let's do the forward FFT on the host and check that the result is correct.
    fftwf_execute_dft_c2r(plan_bwd, reinterpret_cast<fftwf_complex*>(complex_values), real_values);
  }
  else {std::cout << "Error: FFTW plan not set up." << std::endl; exit(1);}

  is_in_real_space = true;
}

template < class wanted_real_type, class wanted_complex_type >
void Image<wanted_real_type, wanted_complex_type>::MakeCufftPlan()
{

  // TODO for alternate precisions.

  hipfftCreate(&cuda_plan_forward);
  hipfftCreate(&cuda_plan_inverse);

  hipfftSetStream(cuda_plan_forward, hipStreamPerThread);
  hipfftSetStream(cuda_plan_inverse, hipStreamPerThread);

  int rank = 2; int iBatch = 1;
  long long int* fftDims = new long long int[rank];
  long long int*inembed = new long long int[rank];
  long long int*onembed = new long long int[rank];

  fftDims[0] = size.y;
  fftDims[1] = size.x;

  inembed[0] = size.y;
  inembed[1] = size.w;

  onembed[0] = size.y;
  onembed[1] = size.w;

  (hipfftXtMakePlanMany(cuda_plan_forward, rank, fftDims,
    NULL, NULL, NULL, HIP_R_32F,
    NULL, NULL, NULL, HIP_C_32F, iBatch, &cuda_plan_worksize_forward, HIP_C_32F));
    (hipfftXtMakePlanMany(cuda_plan_inverse, rank, fftDims,
    NULL, NULL, NULL, HIP_C_32F,
    NULL, NULL, NULL, HIP_R_32F, iBatch, &cuda_plan_worksize_inverse, HIP_R_32F));

    delete [] fftDims;
    delete [] inembed;
    delete [] onembed;

    is_cufft_planned = true;
}

typedef struct _CB_realLoadAndClipInto_params
{
  bool* mask;
	hipfftReal*	target;

} CB_realLoadAndClipInto_params;


static __device__ hipfftReal CB_realLoadAndClipInto(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr);

static __device__ hipfftReal CB_realLoadAndClipInto(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr)
{

	 CB_realLoadAndClipInto_params* my_params = (CB_realLoadAndClipInto_params *)callerInfo;

  if (my_params->mask[offset])
  {
    return my_params->target[offset];

  }
  else
  {
    return 0.0f;
  }



}

__device__ hipfftCallbackLoadR d_realLoadAndClipInto = CB_realLoadAndClipInto;

template < class wanted_real_type, class wanted_complex_type >
void Image<wanted_real_type, wanted_complex_type>::SetClipIntoCallback(hipfftReal* image_to_insert, int image_to_insert_size_x, int image_to_insert_size_y,int image_to_insert_pitch)
{


  // // First make the mask
  short4 wanted_size = make_short4(image_to_insert_size_x, image_to_insert_size_y, 1, image_to_insert_pitch);
  SetClipIntoMask(wanted_size, size );

  if (!is_cufft_planned) {std::cout << "Cufft plan must be made before setting callback function." << std::endl; exit(-1);}

  hipfftCallbackLoadR h_realLoadAndClipInto;
  CB_realLoadAndClipInto_params* d_params;
  CB_realLoadAndClipInto_params h_params;

  precheck_img
  h_params.target = (hipfftReal *)image_to_insert;
  h_params.mask = (bool*) clipIntoMask;
  cudaErr_img(hipMalloc((void **)&d_params,sizeof(CB_realLoadAndClipInto_params)));
  postcheck_img

  precheck_img
  cudaErr_img(hipMemcpyAsync(d_params, &h_params, sizeof(CB_realLoadAndClipInto_params), hipMemcpyHostToDevice, hipStreamPerThread));
  postcheck_img

  precheck_img
  cudaErr_img(hipMemcpyFromSymbol(&h_realLoadAndClipInto,HIP_SYMBOL(d_realLoadAndClipInto), sizeof(h_realLoadAndClipInto)));
  postcheck_img

  precheck_img
  cudaErr_img(hipStreamSynchronize(hipStreamPerThread));
  postcheck_img

  precheck_img
  cudaErr_img(hipfftXtSetCallback(cuda_plan_forward, (void **)&h_realLoadAndClipInto, HIPFFT_CB_LD_REAL, (void **)&d_params));
  postcheck_img


}