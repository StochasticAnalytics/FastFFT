#include "hip/hip_runtime.h"
#include "Image.cu"
#include "../src/FastFFT.cu"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

// The Fourier transform of a constant should be a unit impulse, and on back fft, without normalization, it should be a constant * N.
// It is assumed the input/output have the same dimension (i.e. no padding)
void const_image_test(std::vector<int> size)
{

  bool all_passed = true;
  std::vector<bool> init_passed(size.size(), true);
  std::vector<bool> FFTW_passed(size.size(), true);
  std::vector<bool> FastFFT_forward_passed(size.size(), true);
  std::vector<bool> FastFFT_roundTrip_passed(size.size(), true);

  for (int n = 0; n < size.size() ; n++)
  {

    short4 input_size = make_short4(size[n],size[n],1,0);
    short4 output_size = make_short4(size[n],size[n],1,0);

    bool test_passed = true;
    long address = 0;
    float sum;
    const float acceptable_epsilon = 1e-4;
    float2 sum_complex;

    Image< float, float2 > host_input(input_size);
    Image< float, float2 > host_output(output_size);
    Image< float, float2 > device_output(output_size);


      // Pointers to the arrays on the host -- maybe make this a struct of some sort? I'm sure there is a parallel in cuda, look into cuarray/texture code

    // We just make one instance of the FourierTransformer class, with calc type float.
    // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
    FastFFT::FourierTransformer<float, float, float> FT;
    
    // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
    FT.SetInputDimensionsAndType(input_size.x,input_size.y,input_size.z,true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
    FT.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);

      // The padding (dims.w) is calculated based on the setup
    short4 dims_in = FT.ReturnInputDimensions();
    short4 dims_out = FT.ReturnOutputDimensions();

    // Determine how much memory we need, working with FFTW/CUDA style in place transform padding.
    // Note: there is no reason we really need this, because the xforms will always be out of place. 
    //       For now, this is just in place because all memory in cisTEM is allocated accordingly.
    host_input.real_memory_allocated = FT.ReturnInputMemorySize();
    host_output.real_memory_allocated = FT.ReturnOutputMemorySize();
    
    // On the device, we will always allocate enough memory for the larger of input/output including the buffer array.
    // Minmize the number of calls to malloc which are slow and can lead to fragmentation.
    device_output.real_memory_allocated = std::max(host_input.real_memory_allocated, host_output.real_memory_allocated);
    
    
    // In your own programs, you will be handling this memory allocation yourself. We'll just make something here.
    // I think fftwf_malloc may potentially create a different alignment than new/delete, but kinda doubt it. For cisTEM consistency...
    bool set_fftw_plan = true;
    host_input.Allocate(set_fftw_plan);
    host_output.Allocate(set_fftw_plan);

      
    // Set our input host memory to a constant. Then FFT[0] = host_input_memory_allocated
    FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 1.0f);

      

    
    // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
    // ensures faster transfer. If false, it will be pinned for you.
    FT.SetInputPointer(host_output.real_values, false);
    sum = ReturnSumOfReal(host_output.real_values, dims_out);
    if (sum != dims_out.x*dims_out.y*dims_out.z) {all_passed = false; init_passed[n] = false;}

    // MyFFTDebugAssertTestTrue( sum == dims_out.x*dims_out.y*dims_out.z,"Unit impulse Init ");
    
    // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
    FT.CopyHostToDevice();
      
    host_output.FwdFFT();
    
    test_passed = true;
    for (long index = 1; index < host_output.real_memory_allocated/2; index++)
    {
      if (host_output.complex_values[index].x != 0.0f && host_output.complex_values[index].y != 0.0f) { std::cout << host_output.complex_values[index].x  << " " << host_output.complex_values[index].y << " " << std::endl; test_passed = false;}
    }
    if (host_output.complex_values[0].x != (float)dims_out.x * (float)dims_out.y * (float)dims_out.z) test_passed = false;
    // for (int i = 0; i < 10; i++)
    // {
    //   std::cout << "FFTW unit " << host_output.complex_values[i].x << " " << host_output.complex_values[i].y << std::endl;
    // }
    if (test_passed == false) {all_passed = false; FFTW_passed[n] = false;}
    // MyFFTDebugAssertTestTrue( test_passed, "FFTW unit impulse forward FFT");
    
    // Just to make sure we don't get a false positive, set the host memory to some undesired value.
    FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 2.0f);
    
    // This method will call the regular FFT kernels given the input/output dimensions are equal when the class is instantiated.
    bool swap_real_space_quadrants = false;
    FT.FwdFFT(swap_real_space_quadrants);
    
    // in buffer, do not deallocate, do not unpin memory
    FT.CopyDeviceToHost( false, false);
    test_passed = true;
    for (long index = 1; index < host_output.real_memory_allocated/2; index++)
    {
      if (host_output.complex_values[index].x != 0.0f && host_output.complex_values[index].y != 0.0f) {test_passed = false;} // std::cout << host_output.complex_values[index].x  << " " << host_output.complex_values[index].y << " " << std::endl;}
    }
    if (host_output.complex_values[0].x != (float)dims_out.x * (float)dims_out.y * (float)dims_out.z) test_passed = false;
    // int n=0;
    // for (int x = 0; x <  host_output.size.y ; x++)
    // {
      
    //   std::cout << x << "[ ";
    //   for (int y = 0; y < host_output.size.w; y++)
    //   {  
    //     std::cout << host_output.complex_values[x + y*host_output.size.y].x << "," << host_output.complex_values[x + y*host_output.size.y].y << " ";
    //     n++;
    //     if (n == 34) {n = 0; std::cout << std::endl ;} // line wrapping
    //   }
    //   std::cout << "] " << std::endl;
    //   n = 0;
    // }

    if (test_passed == false) {all_passed = false; FastFFT_forward_passed[n] = false;}
    // MyFFTDebugAssertTestTrue( test_passed, "FastFFT unit impulse forward FFT");
    FT.SetToConstant<float>(host_input.real_values, host_input.real_memory_allocated, 2.0f);
    

    FT.InvFFT();
    FT.CopyDeviceToHost( true, true);
    
    // Assuming the outputs are always even dimensions, padding_jump_val is always 2.
    sum = ReturnSumOfReal(host_output.real_values, dims_out);

    // COMPLEX TODO make these functions.
    //   int n=0;
    //   for (int x = 0; x <  host_output.size.y ; x++)
    // {
      
    //   std::cout << x << "[ ";
    //   for (int y = 0; y < host_output.size.w; y++)
    //   {  
    //     std::cout << host_output.complex_values[x + y*host_output.size.y].x << "," << host_output.complex_values[x + y*host_output.size.y].y << " ";
    //     n++;
    //     if (n == 34) {n = 0; std::cout << std::endl ;} // line wrapping
    //   }
    //   std::cout << "] " << std::endl;
    //   n = 0;
    // }
      // REAL
    //  int n=0;
    // for (int x = 0; x <  host_output.size.x ; x++)
    // {
      
    //   std::cout << x << "[ ";
    //   for (int y = 0; y < host_output.size.y; y++)
    //   {  
    //     std::cout << host_output.real_values[x + y*host_output.size.w*2] <<  " ";
    //     n++;
    //     if (n == 32) {n = 0; std::cout << std::endl ;} // line wrapping
    //   }
    //   std::cout << "] " << std::endl;
    //   n = 0;
    // } 
    if (sum != powf(dims_in.x*dims_in.y*dims_in.z,2)) {all_passed = false; FastFFT_roundTrip_passed[n] = false;}
    // MyFFTDebugAssertTestTrue( sum == powf(dims_in.x*dims_in.y*dims_in.z,2),"FastFFT constant image round trip failed for size");
  }
  
  if (all_passed)
  {
    std::cout << "    All const_image tests passed!" << std::endl;
  }
  else  
  {
    for (int n = 0; n < size.size() ; n++)
    {
      if ( ! init_passed[n] ) std::cout << "    Initialization failed for size " << size[n] << std::endl;
      if ( ! FFTW_passed[n] ) std::cout << "    FFTW failed for size " << size[n] << std::endl;
      if ( ! FastFFT_forward_passed[n] ) std::cout << "    FastFFT failed for forward transform size " << size[n] << std::endl;
      if ( ! FastFFT_roundTrip_passed[n] ) std::cout << "    FastFFT failed for roundtrip transform size " << size[n] << std::endl;

    }
  }
}

void unit_impulse_test(std::vector<int>size)
{

  bool all_passed = true;
  std::vector<bool> init_passed(size.size(), true);
  std::vector<bool> FFTW_passed(size.size(), true);
  std::vector<bool> FastFFT_forward_passed(size.size(), true);
  std::vector<bool> FastFFT_roundTrip_passed(size.size(), true);

  for (int iSize = 0; iSize < size.size() - 1 ; iSize++)
  {
    int oSize = iSize + 1;
    while (oSize < size.size())
    {
      std::cout << std::endl << "Testing padding from  " << size[iSize] << " to " << size[oSize] << std::endl;
      short4 input_size = make_short4(size[iSize],size[iSize],1,0);
      short4 output_size = make_short4(size[oSize],size[oSize],1,0);   


  bool test_passed = true;
  long address = 0;

  float sum;
  float2 sum_complex;

  Image< float, float2 > host_input(input_size);
  Image< float, float2 > host_output(output_size);
  Image< float, float2 > device_output(output_size);
  

  // We just make one instance of the FourierTransformer class, with calc type float.
  // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
  FastFFT::FourierTransformer<float, float, float> FT;
  // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
  FT.SetInputDimensionsAndType(input_size.x,input_size.y,input_size.z,true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
  FT.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural); 
 
  // The padding (dims.w) is calculated based on the setup
  short4 dims_in = FT.ReturnInputDimensions();
  short4 dims_out = FT.ReturnOutputDimensions();
  // Determine how much memory we need, working with FFTW/CUDA style in place transform padding.
  // Note: there is no reason we really need this, because the xforms will always be out of place. 
  //       For now, this is just in place because all memory in cisTEM is allocated accordingly.
  host_input.real_memory_allocated = FT.ReturnInputMemorySize();
  host_output.real_memory_allocated = FT.ReturnOutputMemorySize();



  // On the device, we will always allocate enough memory for the larger of input/output including the buffer array.
  // Minmize the number of calls to malloc which are slow and can lead to fragmentation.
  device_output.real_memory_allocated = std::max(host_input.real_memory_allocated, host_output.real_memory_allocated);
  
  // In your own programs, you will be handling this memory allocation yourself. We'll just make something here.
  // I think fftwf_malloc may potentially create a different alignment than new/delete, but kinda doubt it. For cisTEM consistency...
  bool set_fftw_plan = true;
  host_input.Allocate(set_fftw_plan);
  host_output.Allocate(set_fftw_plan);
  
  // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
  // ensures faster transfer. If false, it will be pinned for you.
  FT.SetInputPointer(host_input.real_values, false);
  
  // Set a unit impulse at the center of the input array.
  FT.SetToConstant<float>(host_input.real_values, host_input.real_memory_allocated, 0.0f);
  FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 0.0f);

  sum = ReturnSumOfReal(host_output.real_values, dims_out);
  // host_input.real_values[ dims_in.y/2 * (dims_in.x+host_input.padding_jump_value) + dims_in.x/2] = 1.0f;
  // short4 wanted_center = make_short4(0,0,0,0);
  // ClipInto(host_input.real_values, host_output.real_values, dims_in ,  dims_out,  wanted_center, 0.f);

  // FT.SetToConstant<float>(host_input.real_values, host_input.real_memory_allocated, 0.0f);
  host_input.real_values[0] = 1.0f;
  host_output.real_values[0] = 1.0f;


  sum = ReturnSumOfReal(host_output.real_values, dims_out);
  if (sum != 1) {all_passed = false; init_passed[iSize] = false;}

  // MyFFTDebugAssertTestTrue( sum == 1,"Unit impulse Init ");
  
  // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
  FT.CopyHostToDevice();

  host_output.FwdFFT();
  
  host_output.fftw_epsilon = ReturnSumOfComplexAmplitudes(host_output.complex_values, host_output.real_memory_allocated/2);  
  // std::cout << "host " << host_output.fftw_epsilon << " " << host_output.real_memory_allocated<< std::endl;

  host_output.fftw_epsilon -= (host_output.real_memory_allocated/2 );
  if (std::abs(host_output.fftw_epsilon) > 1e-8 ) {all_passed = false; FFTW_passed[iSize] = false;}

  // MyFFTDebugAssertTestTrue( std::abs(host_output.fftw_epsilon) < 1e-8 , "FFTW unit impulse forward FFT");
  
  // Just to make sure we don't get a false positive, set the host memory to some undesired value.
  FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 2.0f);
  
  // This method will call the regular FFT kernels given the input/output dimensions are equal when the class is instantiated.
  bool swap_real_space_quadrants = true;
  FT.FwdFFT(swap_real_space_quadrants);
  
  // do not deallocate, do not unpin memory

  FT.CopyDeviceToHost(host_output.real_values, false, false);

  // int n=0;
  // for (int x = 0; x <  host_output.size.y ; x++)
  // {
    
  //   std::cout << x << "[ ";
  //   for (int y = 0; y < host_output.size.w; y++)
  //   {  
  //     std::cout << host_output.complex_values[x + y*host_output.size.y].x << "," << host_output.complex_values[x + y*host_output.size.y].y << " ";
  //     n++;
  //     if (n == 32) {n = 0; std::cout << std::endl ;} // line wrapping
  //   }
  //   std::cout << "] " << std::endl;
  //   n = 0;
  // }

  sum = ReturnSumOfComplexAmplitudes(host_output.complex_values, host_output.real_memory_allocated/2); 
  // std::cout << sum << " " << host_output.real_memory_allocated<< std::endl;

  sum -= (host_output.real_memory_allocated/2 );


  // std::cout << "FFT Unit Impulse Forward FFT: " << sum <<  " epsilon " << host_output.fftw_epsilon << std::endl;
  // std::cout << "epsilon " << abs(sum - host_output.fftw_epsilon) << std::endl;
  if (abs(sum - host_output.fftw_epsilon) > 1e-8) {all_passed = false; FastFFT_forward_passed[iSize] = false;}

  // MyFFTDebugAssertTestTrue( abs(sum - host_output.fftw_epsilon) < 1e-8, "FastFFT unit impulse forward FFT");
  FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 2.0f);
  

  FT.InvFFT();
  FT.CopyDeviceToHost(host_output.real_values, true, true);
  // for (int x = 0; x < 128; x++)
  // {
  //   int n=0;
  //   std::cout << x << "[ ";
  //   for (int y = 0; y < 128; y++)
  //   {  
  //     std::cout << host_output[x + y*130]<< " ";
  //   }
  //   std::cout << "] " << n << std::endl;
  // }
  // Assuming the outputs are always even dimensions, padding_jump_val is always 2.
  sum = ReturnSumOfReal(host_output.real_values, dims_out);
  if (sum != dims_out.x*dims_out.y*dims_out.z) {all_passed = false; FastFFT_roundTrip_passed[iSize] = false;}

  // MyFFTDebugAssertTestTrue( sum == dims_out.x*dims_out.y*dims_out.z,"FastFFT unit impulse round trip FFT");
    oSize++;
    } // while loop over pad to size
  } // for loop over pad from size


  if (all_passed)
  {
    std::cout << "    All unit impulse tests passed!" << std::endl;
  }
  else  
  {
    for (int n = 0; n < size.size() ; n++)
    {
      if ( ! init_passed[n] ) std::cout << "    Initialization failed for size " << size[n] << std::endl;
      if ( ! FFTW_passed[n] ) std::cout << "    FFTW failed for size " << size[n] << std::endl;
      if ( ! FastFFT_forward_passed[n] ) std::cout << "    FastFFT failed for forward transform size " << size[n] << std::endl;
      if ( ! FastFFT_roundTrip_passed[n] ) std::cout << "    FastFFT failed for roundtrip transform size " << size[n] << std::endl;

    }
  }

}

void compare_libraries(short4 input_size, short4 output_size)
{

  bool set_padding_callback = false; // the padding callback is slower than pasting in b/c the read size of the pointers is larger than the actual data. do not use.
  bool set_conjMult_callback = true;
  if (input_size.x == output_size.x && input_size.y == output_size.y && input_size.z == output_size.z) 
  {
    // Also will change the path called in FastFFT to just be fwd/inv xform.
    set_conjMult_callback = false;
  }

  bool test_passed = true;
  long address = 0;

  float sum;
  float2 sum_complex;

  Image< float, float2 > FT_input(input_size);
  Image< float, float2 > FT_output(output_size);
  Image< float, float2 > cuFFT_input(input_size);
  Image< float, float2 > cuFFT_output(output_size);

  Image< float, float2> target_search_image(output_size);
  Image< float, float2> positive_control(output_size);


   // We just make one instance of the FourierTransformer class, with calc type float.
  // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
  FastFFT::FourierTransformer<float, float, float> FT;
    // Create an instance to copy memory also for the cufft tests.
  FastFFT::FourierTransformer<float, float, float> cuFFT;
  FastFFT::FourierTransformer<float, float, float> targetFT;

  // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
  FT.SetInputDimensionsAndType(input_size.x,input_size.y,input_size.z,true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
  FT.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
  cuFFT.SetInputDimensionsAndType(input_size.x,input_size.y,input_size.z,true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
  cuFFT.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
  
  targetFT.SetInputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
  targetFT.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);


  FT_input.real_memory_allocated = FT.ReturnInputMemorySize();
  FT_output.real_memory_allocated = FT.ReturnOutputMemorySize();
  
  cuFFT_input.real_memory_allocated = cuFFT.ReturnInputMemorySize();
  cuFFT_output.real_memory_allocated = cuFFT.ReturnOutputMemorySize();

  target_search_image.real_memory_allocated = targetFT.ReturnInputMemorySize();
  positive_control.real_memory_allocated = targetFT.ReturnOutputMemorySize();


  bool set_fftw_plan = false;
  FT_input.Allocate(set_fftw_plan);
  FT_output.Allocate(set_fftw_plan);

  cuFFT_input.Allocate(set_fftw_plan);
  cuFFT_output.Allocate(set_fftw_plan);

  target_search_image.Allocate(true);
  positive_control.Allocate(true);


  // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
  // ensures faster transfer. If false, it will be pinned for you.
  FT.SetInputPointer(FT_input.real_values, false);
  cuFFT.SetInputPointer(cuFFT_input.real_values, false);
  targetFT.SetInputPointer(target_search_image.real_values, false);

  // Set a unit impulse at the center of the input array.
  FT.SetToConstant<float>(FT_input.real_values, FT_input.real_memory_allocated, 0.0f);
  FT.SetToConstant<float>(cuFFT_input.real_values, cuFFT_input.real_memory_allocated, 0.0f);
  FT.SetToConstant<float>(FT_output.real_values, FT_input.real_memory_allocated, 0.0f);
  FT.SetToConstant<float>(cuFFT_output.real_values, cuFFT_input.real_memory_allocated, 0.0f);
  FT.SetToConstant<float>(target_search_image.real_values, target_search_image.real_memory_allocated, 0.0f);
  FT.SetToConstant<float>(positive_control.real_values, target_search_image.real_memory_allocated, 0.0f);


  // Place these values at the origin of the image and after convolution, should be at 0,0,0.
  float testVal_1 = 2.0f;
  float testVal_2 = 3.0f;
  FT_input.real_values[0] = testVal_1;
  cuFFT_input.real_values[0] = testVal_1;
  target_search_image.real_values[0] = testVal_2;//target_search_image.size.w*2*target_search_image.size.y/2 + target_search_image.size.x/2] = testVal_2;
  positive_control.real_values[0] = testVal_1;//target_search_image.size.w*2*target_search_image.size.y/2 + target_search_image.size.x/2] = testVal_1;

  // Transform the target on the host prior to transfer.
  target_search_image.FwdFFT();

  // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
  FT.CopyHostToDevice();
  cuFFT.CopyHostToDevice();
  targetFT.CopyHostToDevice();
  // Wait on the transfers to finish.
  cudaErr(hipStreamSynchronize(hipStreamPerThread));  

  // Positive control on the host.
  positive_control.FwdFFT();
  positive_control.MultiplyConjugateImage(target_search_image.complex_values);
  positive_control.InvFFT();


  // address = 0;
  // test_passed = true;
  // for (int z = 1; z <  positive_control.size.z ; z++)
  // {   
  //   for (int y = 1; y < positive_control.size.y; y++)
  //   {  
  //     for (int x = 1; x < positive_control.size.x; x++)
  //     {
  //       if (positive_control.real_values[address] != 0.0f) test_passed = false;
  //     }
  //   }
  // }
  // if (test_passed) 
  // {
  //   if (positive_control.real_values[address] == positive_control.size.x*positive_control.size.y*positive_control.size.z*testVal_1*testVal_2)
  //   {
  //     std::cout << "Test passed for FFTW positive control.\n" << std::endl;
  //   }
  //   else
  //   {
  //     std::cout << "Test failed for FFTW positive control. Value at zero is  " << positive_control.real_values[address] << std::endl;
  //   }
  // }
  // else
  // {
  //   std::cout << "Test failed for positive control, non-zero values found away from the origin." << std::endl;
  // }


  cuFFT_output.create_timing_events(); 
  cuFFT_input.MakeCufftPlan();
  cuFFT_output.MakeCufftPlan();

  //////////////////////////////////////////
  //////////////////////////////////////////
  // Warm up and check for accuracy
  if (set_conjMult_callback)
  {
    FT.CrossCorrelate(targetFT.d_ptr.momentum_space_buffer, false);
  }
  else
  {
    FT.FwdFFT();
    FT.InvFFT();
  }
  FT.CopyDeviceToHost(FT_output.real_values,false, false);

  // address = 0;
  // test_passed = true;
  // for (int z = 1; z <  FT_output.size.z ; z++)
  // {   
  //   for (int y = 1; y < FT_output.size.y; y++)
  //   {  
  //     for (int x = 1; x < FT_output.size.x; x++)
  //     {
  //       if (FT_output.real_values[address] != 0.0f) test_passed = false;
  //     }
  //   }
  // }
  // if (test_passed) 
  // {
  //   if (FT_output.real_values[address] == FT_output.size.x*FT_output.size.y*FT_output.size.z*testVal_1*testVal_2)
  //   {
  //     std::cout << "Test passed for FastFFT positive control.\n" << std::endl;
  //   }
  //   else
  //   {
  //     std::cout << "Test failed for FastFFT positive control. Value at zero is  " << FT_output.real_values[address] << std::endl;
  //   }
  // }
  // else
  // {
  //   std::cout << "Test failed for FastFFT control, non-zero values found away from the origin." << std::endl;
  // }

  //////////////////////////////////////////
  // //////////////////////////////////////////
  // int n = 0;
  // for (int x = 0; x <  FT_output.size.x ; x++)
  // {
    
  //   std::cout << x << "[ ";
  //   for (int y = 0; y < FT_output.size.y; y++)
  //   {  
  //     std::cout << FT_output.real_values[x + y*FT_output.size.w*2] << " ";
  //     n++;
  //     if (n == 32) {n = 0; std::cout << std::endl ;} // line wrapping
  //   }
  //   std::cout << "] " << std::endl;
  //   n = 0;
  // }


  const int n_loops = 10000;
  cuFFT_output.record_start();
  for (int i = 0; i < n_loops; ++i)
  {
    if (set_conjMult_callback)
    {
      FT.CrossCorrelate(targetFT.d_ptr.momentum_space_buffer, false);
    }
    else
    {
      FT.FwdFFT();
      FT.InvFFT();
    }
  }
  cuFFT_output.record_stop();
  cuFFT_output.synchronize();
  cuFFT_output.print_time("FastFFT");
  float FastFFT_time = cuFFT_output.elapsed_gpu_ms;

  if (set_padding_callback) 
  {
    precheck
    hipfftReal* overlap_pointer;
    overlap_pointer = cuFFT.d_ptr.position_space;
    cuFFT_output.SetClipIntoCallback(overlap_pointer, cuFFT_input.size.x, cuFFT_input.size.y, cuFFT_input.size.w*2);
    postcheck
  }

  if (set_conjMult_callback)
  {
    precheck
    // FIXME scaling factor
    cuFFT_output.SetComplexConjMultiplyAndLoadCallBack( (hipfftComplex *) targetFT.d_ptr.momentum_space_buffer, 1.0f);
    postcheck
  }


  //////////////////////////////////////////
  //////////////////////////////////////////
  // Warm up and check for accuracy
  cuFFT.ClipIntoTopLeft();
  // cuFFT.ClipIntoReal(cuFFT_output.size.x/2, cuFFT_output.size.y/2, cuFFT_output.size.z/2);

  cuFFT.CopyDeviceToHost(cuFFT_output.real_values,false, false);
  // cuFFT.ClipIntoReal(input_size.x/2, input_size.y/2, input_size.z/2);
  precheck
  cudaErr(hipfftExecR2C(cuFFT_output.cuda_plan_forward, (hipfftReal*)cuFFT.d_ptr.position_space, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer));
  postcheck
  precheck
  cudaErr(hipfftExecC2R(cuFFT_output.cuda_plan_inverse, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer, (hipfftReal*)cuFFT.d_ptr.position_space));
  postcheck  
  cuFFT.CopyDeviceToHost(cuFFT_output.real_values,false, false);

  // address = 0;
  // test_passed = true;
  // for (int z = 1; z <  cuFFT_output.size.z ; z++)
  // {   
  //   for (int y = 1; y < cuFFT_output.size.y; y++)
  //   {  
  //     for (int x = 1; x < cuFFT_output.size.x; x++)
  //     {
  //       if (cuFFT_output.real_values[address] != 0.0f) test_passed = false;
  //     }
  //   }
  // }
  // if (test_passed) 
  // {
  //   if (cuFFT_output.real_values[address] == cuFFT_output.size.x*cuFFT_output.size.y*cuFFT_output.size.z*testVal_1*testVal_2)
  //   {
  //     std::cout << "Test passed for cuFFT positive control.\n" << std::endl;
  //   }
  //   else
  //   {
  //     std::cout << "Test failed for cuFFT positive control. Value at zero is  " << cuFFT_output.real_values[address] << std::endl;
  //   }
  // }
  // else
  // {
  //   std::cout << "Test failed for cuFFT control, non-zero values found away from the origin." << std::endl;
  // }
  //////////////////////////////////////////
  //////////////////////////////////////////
  // n = 0;
  // for (int x = 0; x <  cuFFT_output.size.x ; x++)
  // {
    
  //   std::cout << x << "[ ";
  //   for (int y = 0; y < cuFFT_output.size.y; y++)
  //   {  
  //     std::cout << cuFFT_output.real_values[x + y*cuFFT_output.size.w*2] << " ";
  //     n++;
  //     if (n == 32) {n = 0; std::cout << std::endl ;} // line wrapping
  //   }
  //   std::cout << "] " << std::endl;
  //   n = 0;
  // }

  cuFFT_output.record_start();
  for (int i = 0; i < n_loops; ++i)
  {
    if (set_conjMult_callback) cuFFT.ClipIntoTopLeft();
    // cuFFT.ClipIntoReal(input_size.x/2, input_size.y/2, input_size.z/2);

    precheck
    cudaErr(hipfftExecR2C(cuFFT_output.cuda_plan_forward, (hipfftReal*)cuFFT.d_ptr.position_space, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer));
    postcheck

    precheck
    cudaErr(hipfftExecC2R(cuFFT_output.cuda_plan_inverse, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer, (hipfftReal*)cuFFT.d_ptr.position_space));
    postcheck
  }
  cuFFT_output.record_stop();
  cuFFT_output.synchronize();
  cuFFT_output.print_time("cuFFT");

  std::cout << "Ratio cuFFT/FastFFT : " << cuFFT_output.elapsed_gpu_ms/FastFFT_time << std::endl;

}

void run_oned(std::vector<int> size)
{

  // Override the size to be one dimensional in x
  std::cout << "Running one-dimensional tests\n" << std::endl;

  bool test_passed = true;
  long address = 0;

  float sum;
  float2 sum_complex;

  for (int n : size)
  {
    short4 input_size = make_short4(n,1,1,0);
    short4 output_size = make_short4(n,1,1,0);

    Image< float, float2 > FT_input(input_size);
    Image< float, float2 > FT_output(output_size);
    Image< float2, float2 > FT_input_complex(input_size);
    Image< float2, float2 > FT_output_complex(output_size);

    // We just make one instance of the FourierTransformer class, with calc type float.
    // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
    FastFFT::FourierTransformer<float, float, float> FT;
    FastFFT::FourierTransformer<float, float2, float2> FT_complex;

    // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
    FT.SetInputDimensionsAndType(input_size.x,input_size.y,input_size.z,true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
    FT.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);

    FT_complex.SetInputDimensionsAndType(input_size.x,input_size.y,input_size.z,true, false, FastFFT::FourierTransformer<float, float2 ,float2>::OriginType::natural);
    FT_complex.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, FastFFT::FourierTransformer<float, float2 ,float2>::OriginType::natural);

    FT_input.real_memory_allocated = FT.ReturnInputMemorySize();
    FT_output.real_memory_allocated = FT.ReturnOutputMemorySize();

    FT_input_complex.real_memory_allocated = FT_complex.ReturnInputMemorySize();
    FT_output_complex.real_memory_allocated = FT_complex.ReturnOutputMemorySize();
    std::cout << "Allocated " << FT_input_complex.real_memory_allocated << " bytes for input.\n";
    std::cout << "Allocated complex " << FT_output_complex.real_memory_allocated << " bytes for input.\n";

    bool set_fftw_plan = true;
    FT_input.Allocate(set_fftw_plan);
    FT_output.Allocate(set_fftw_plan);

    FT_input_complex.Allocate();
    FT_output_complex.Allocate();



    // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
    // ensures faster transfer. If false, it will be pinned for you.
    FT.SetInputPointer(FT_input.real_values, false);
    FT_complex.SetInputPointer(FT_input_complex.complex_values, false);


    FT.SetToConstant<float>(FT_input.real_values, FT_input.real_memory_allocated, 1.f);

    // Set a unit impulse at the center of the input array.
    // FT.SetToConstant<float>(FT_input.real_values, FT_input.real_memory_allocated, 1.0f);
    float2 const_val = make_float2(1.0f,0.0f);
    FT_complex.SetToConstant<float2>(FT_input_complex.complex_values, FT_input.real_memory_allocated, const_val);
    for (int i=0; i<10; i++)
    {
      std::cout << FT_input_complex.complex_values[i].x << "," << FT_input_complex.complex_values[i].y << std::endl;
    }


    FT.CopyHostToDevice();
    FT_complex.CopyHostToDevice();
    cudaErr(hipStreamSynchronize(hipStreamPerThread));  

        // Set the outputs to a clearly wrong answer.
        FT.SetToConstant<float>(FT_output.real_values, FT_input.real_memory_allocated, 2.0f);
        const_val = make_float2(2.0f,2.0f);
        FT_complex.SetToConstant<float2>(FT_output_complex.complex_values, FT_output.real_memory_allocated, const_val);

    FT_input.FwdFFT();

    for (int i = 0; i < 5; ++i) std::cout << "FFTW fwd " << FT_input.real_values[i] << std::endl;
    std::cout << std::endl;


    bool transpose_output = false;
    bool swap_real_space_quadrants = false;
    FT.FwdFFT(swap_real_space_quadrants, transpose_output);
    FT_complex.FwdFFT(swap_real_space_quadrants, transpose_output);

    FT.CopyDeviceToHost(FT_output.real_values, false, false);
    FT_complex.CopyDeviceToHost(FT_output_complex.real_values, false, false);

    for (int i = 0; i < 10; ++i) {std::cout << "FT fwd " << FT_output.real_values[i] << std::endl;}
    for (int i = 0; i < 10; ++i) {std::cout << "FT complex fwd "<< FT_output_complex.real_values[i].x << "," << FT_output_complex.real_values[i].y << std::endl;}
  

    FT_input.InvFFT();

    for (int i = 0; i < 5; ++i) {std::cout << "FFTW inv " << FT_input.real_values[i] << std::endl;}
    std::cout << std::endl;



    FT.InvFFT(transpose_output);
    FT_complex.InvFFT(transpose_output);
    FT.CopyDeviceToHost(FT_output.real_values, true, true);
    FT_complex.CopyDeviceToHost(FT_output_complex.real_values, true, true);

    for (int i = 0; i < 10; i++) {std::cout << "Ft inv " << FT_output.real_values[i] << std::endl;}
    for (int i = 0; i < 10; i++) {std::cout << "Ft complex inv " << FT_output_complex.real_values[i].x << "," << FT_output_complex.real_values[i].y << std::endl;}


  }


}
int main(int argc, char** argv) {

  std::printf("Entering main in tests.cpp\n");
  std::printf("Standard is %i\n\n",__cplusplus);


  bool run_validation_tests;
  bool run_performance_tests;

  if (argc > 1)
  {
    run_validation_tests = false;
    run_performance_tests = true;
    std::cout << "Running performance tests.\n";
  }
  else
  {
    run_validation_tests = true;
    run_performance_tests = false;
    std::cout << "Running validation tests.\n";
  }
  // Input and output dimensions, with simple checks. I'm sure there are better checks on argv.
  short4 input_size;
  short4 output_size;

  std::vector<int> test_size = { 64, 128, 256, 512, 1024, 2048, 4096};

  std::vector<int> test_sizes =  {32};//,64,128,256,320,480,512,544,608,768,1024,1056,1536,2048,2560,3072,3584,4096,5120,6144};

  if (run_validation_tests)  {

    // change onde these to just report the pass/fail.
    // run_oned(test_sizes);
    // exit(0);



    const_image_test(test_size);
    unit_impulse_test(test_size);


  } // end of validation tests


  if (run_performance_tests) {

    #ifdef HEAVYERRORCHECKING_FFT
      std::cout << "Running performance tests with heavy error checking.\n";
      std::cout << "This doesn't make sense as the synchronizations are invalidating.\n";
      // exit(1);
    #endif

    for (int iSize = 0; iSize < test_size.size(); iSize++) {

      std::cout << std::endl << "Testing cufft comparison " << test_size[iSize] << " x" << std::endl;
      input_size = make_short4(test_size[iSize],test_size[iSize],1,0);
      output_size = make_short4(test_size[iSize],test_size[iSize],1,0);

      compare_libraries(input_size, output_size);

    }
 
    for (int iSize = 0; iSize < test_size.size() - 1; iSize++) {
      int oSize = iSize + 1;
      while (oSize < test_size.size())
      {
        std::cout << std::endl << "Testing padding from  " << test_size[iSize] << " to " << test_size[oSize] << std::endl;
        input_size = make_short4(test_size[iSize],test_size[iSize],1,0);
        output_size = make_short4(test_size[oSize],test_size[oSize],1,0);
    
        compare_libraries(input_size, output_size);
        oSize++;
      }
    }
  }

}
