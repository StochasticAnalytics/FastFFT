#include "hip/hip_runtime.h"
#include "Image.cu"
#include "../src/FastFFT.cu"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#define MyTestPrintAndExit(...)	{std::cerr << __VA_ARGS__  << " From: " << __FILE__  << " " << __LINE__  << " " << __PRETTY_FUNCTION__ << std::endl; exit(-1);}


void PrintArray( float2* array, short NX, short NY, short NZ, int line_wrapping = 34)
{
    // COMPLEX TODO make these functions.
    int n=0;
    for (int z = 0; z < NZ ; z ++)
    {
      for (int x = 0; x <  NX ; x++)
      {
        
        std::cout << x << "[ ";
        for (int y = 0; y < NY; y++)
        {  
          std::cout << array[x + NX*(y + z*NY)].x << "," << array[x + NX*(y + z*NY)].y << " ";
          n++;
          if (n == line_wrapping) {n = 0; std::cout << std::endl ;} // line wrapping
        }
        std::cout << "] " << std::endl;
        n = 0;
      }
      if (NZ > 0) std::cout << " ... ... ... " << z << " ... ... ..." << std::endl;
    }

};

void PrintArray(float* array, short NX, short NY, short NZ, short NW, int line_wrapping = 34)
{
  int n=0;
  for (int z = 0; z < NZ ; z ++)
  {
    for (int x = 0; x <  NX ; x++)
    {

      std::cout << x << "[ ";
      for (int y = 0; y < NY; y++)
      {  
        std::cout << array[x + (2*NW)*(y + z*NY)] <<  " ";
        n++;
        if (n == line_wrapping) {n = 0; std::cout << std::endl ;} // line wrapping
      }
      std::cout << "] " << std::endl;
      n = 0;
    } 
    if (NZ > 0) std::cout << " ... ... ... " << z << " ... ... ..." << std::endl;
  }
};

// The Fourier transform of a constant should be a unit impulse, and on back fft, without normalization, it should be a constant * N.
// It is assumed the input/output have the same dimension (i.e. no padding)
void const_image_test(std::vector<int> size, bool do_3d = false)
{

  bool all_passed = true;
  std::vector<bool> init_passed(size.size(), true);
  std::vector<bool> FFTW_passed(size.size(), true);
  std::vector<bool> FastFFT_forward_passed(size.size(), true);
  std::vector<bool> FastFFT_roundTrip_passed(size.size(), true);

  for (int n = 0; n < size.size() ; n++)
  {

    short4 input_size;
    short4 output_size;
    long full_sum = long(size[n]);
    if (do_3d)
    {
      input_size = make_short4(size[n],size[n],size[n],0);
      output_size = make_short4(size[n],size[n],size[n],0);
      full_sum =  full_sum*full_sum*full_sum*full_sum*full_sum*full_sum;
    }
    else
    {
      input_size = make_short4(size[n],size[n],1,0);
      output_size = make_short4(size[n],size[n],1,0);
      full_sum = full_sum*full_sum*full_sum;
    }


    bool test_passed = true;
    long address = 0;
    float sum;
    const float acceptable_epsilon = 1e-4;
    float2 sum_complex;

    Image< float, float2 > host_input(input_size);
    Image< float, float2 > host_output(output_size);
    Image< float, float2 > device_output(output_size);


      // Pointers to the arrays on the host -- maybe make this a struct of some sort? I'm sure there is a parallel in cuda, look into cuarray/texture code

    // We just make one instance of the FourierTransformer class, with calc type float.
    // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
    FastFFT::FourierTransformer<float, float, float> FT;
    
    // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
    FT.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, output_size.x,output_size.y,output_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
    FT.SetInverseFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);

      // The padding (dims.w) is calculated based on the setup
    short4 dims_in = FT.ReturnFwdInputDimensions();
    short4 dims_out = FT.ReturnFwdOutputDimensions();

    // Determine how much memory we need, working with FFTW/CUDA style in place transform padding.
    // Note: there is no reason we really need this, because the xforms will always be out of place. 
    //       For now, this is just in place because all memory in cisTEM is allocated accordingly.
    host_input.real_memory_allocated = FT.ReturnInputMemorySize();
    host_output.real_memory_allocated = FT.ReturnInvOutputMemorySize();

    // On the device, we will always allocate enough memory for the larger of input/output including the buffer array.
    // Minmize the number of calls to malloc which are slow and can lead to fragmentation.
    device_output.real_memory_allocated = std::max(host_input.real_memory_allocated, host_output.real_memory_allocated);
    
    // In your own programs, you will be handling this memory allocation yourself. We'll just make something here.
    // I think fftwf_malloc may potentially create a different alignment than new/delete, but kinda doubt it. For cisTEM consistency...
    bool set_fftw_plan = true;
    host_input.Allocate(set_fftw_plan);
    host_output.Allocate(set_fftw_plan);

      
    // Set our input host memory to a constant. Then FFT[0] = host_input_memory_allocated
    FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 1.0f);

      

    
    // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
    // ensures faster transfer. If false, it will be pinned for you.
    FT.SetInputPointer(host_output.real_values, false);
    sum = ReturnSumOfReal(host_output.real_values, dims_out);

      if (sum != long(dims_in.x)*long(dims_in.y)*long(dims_in.z)) {all_passed = false; init_passed[n] = false;}
    
    // MyFFTDebugAssertTestTrue( sum == dims_out.x*dims_out.y*dims_out.z,"Unit impulse Init ");
    
    // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
    FT.CopyHostToDevice();
      
    host_output.FwdFFT();
    
    test_passed = true;
    for (long index = 1; index < host_output.real_memory_allocated/2; index++)
    {
      if (host_output.complex_values[index].x != 0.0f && host_output.complex_values[index].y != 0.0f) { std::cout << host_output.complex_values[index].x  << " " << host_output.complex_values[index].y << " " << std::endl; test_passed = false;}
    }
    if (host_output.complex_values[0].x != (float)dims_out.x * (float)dims_out.y * (float)dims_out.z) test_passed = false;

    if (test_passed == false) {all_passed = false; FFTW_passed[n] = false;}
    // MyFFTDebugAssertTestTrue( test_passed, "FFTW unit impulse forward FFT");
    
    // Just to make sure we don't get a false positive, set the host memory to some undesired value.
    FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 2.0f);
    
    // This method will call the regular FFT kernels given the input/output dimensions are equal when the class is instantiated.
    bool swap_real_space_quadrants = false;
    FT.FwdFFT(swap_real_space_quadrants);
    
    // in buffer, do not deallocate, do not unpin memory
    FT.CopyDeviceToHost( false, false);
    test_passed = true;
    for (long index = 1; index < host_output.real_memory_allocated/2; index++)
    {
      if (host_output.complex_values[index].x != 0.0f && host_output.complex_values[index].y != 0.0f) {test_passed = false;} // std::cout << host_output.complex_values[index].x  << " " << host_output.complex_values[index].y << " " );}
    }
    if (host_output.complex_values[0].x != (float)dims_out.x * (float)dims_out.y * (float)dims_out.z) test_passed = false;


    #if DEBUG_FFT_STAGE == 0
      PrintArray(host_output.real_values, dims_out.x, dims_in.y, dims_in.z, dims_out.w);
      MyTestPrintAndExit( "stage 0 " );
    #elif DEBUG_FFT_STAGE == 1
      PrintArray(host_output.complex_values, dims_in.y, dims_out.w, dims_in.z);
      MyTestPrintAndExit( "stage 1 " );
      #elif DEBUG_FFT_STAGE == 2
      PrintArray(host_output.complex_values, dims_in.y, dims_out.w, dims_out.z);
      MyTestPrintAndExit( "stage 2 " );      
    #elif DEBUG_FFT_STAGE == 3
      PrintArray(host_output.complex_values, dims_in.y, dims_out.w, dims_out.z);
      MyTestPrintAndExit( "stage 3 " );
    #endif   
    

    if (test_passed == false) {all_passed = false; FastFFT_forward_passed[n] = false;}
    // MyFFTDebugAssertTestTrue( test_passed, "FastFFT unit impulse forward FFT");
    FT.SetToConstant<float>(host_input.real_values, host_input.real_memory_allocated, 2.0f);
    

    FT.InvFFT();
    FT.CopyDeviceToHost( true, true);
 

    #if DEBUG_FFT_STAGE == 4
      PrintArray(host_output.complex_values, dims_out.y, dims_out.w, dims_out.z);
      MyTestPrintAndExit( "stage 4 " );
    #elif DEBUG_FFT_STAGE == 5
      PrintArray(host_output.complex_values, dims_out.y, dims_out.w, dims_out.z);
      MyTestPrintAndExit( "stage 5 " );
    #elif DEBUG_FFT_STAGE == 6
      PrintArray(host_output.complex_values, dims_out.y, dims_out.w, dims_out.z);
      MyTestPrintAndExit( "stage 6 " );      
    #elif DEBUG_FFT_STAGE == 7
      PrintArray(host_output.real_values, dims_out.x, dims_out.y,dims_out.z, dims_out.w);
      MyTestPrintAndExit( "stage 7 " );
    #elif DEBUG_FFT_STAGE > 7
      // No debug, keep going
    #else
      MyTestPrintAndExit( " This block is only valid for DEBUG_FFT_STAGE == 4, 5, 7 " );
    #endif   


    // Assuming the outputs are always even dimensions, padding_jump_val is always 2.
    sum = ReturnSumOfReal(host_output.real_values, dims_out);

    if (sum != full_sum) {all_passed = false; FastFFT_roundTrip_passed[n] = false;}
    MyFFTDebugAssertTestTrue( sum == full_sum,"FastFFT constant image round trip for size " + std::to_string(dims_in.x));
  } // loop over sizes
  
  if (all_passed)
  {
    std::cout << "    All const_image tests passed!" << std::endl;
  }
  else  
  {
    for (int n = 0; n < size.size() ; n++)
    {
      if ( ! init_passed[n] ) std::cout << "    Initialization failed for size " << size[n] << std::endl;
      if ( ! FFTW_passed[n] ) std::cout << "    FFTW failed for size " << size[n] << std::endl;
      if ( ! FastFFT_forward_passed[n] ) std::cout << "    FastFFT failed for forward transform size " << size[n] << std::endl;
      if ( ! FastFFT_roundTrip_passed[n] ) std::cout << "    FastFFT failed for roundtrip transform size " << size[n] << std::endl;

    }
  }
}

void unit_impulse_test(std::vector<int>size, bool do_increase_size)
{

  bool all_passed = true;
  std::vector<bool> init_passed(size.size(), true);
  std::vector<bool> FFTW_passed(size.size(), true);
  std::vector<bool> FastFFT_forward_passed(size.size(), true);
  std::vector<bool> FastFFT_roundTrip_passed(size.size(), true);

  short4 input_size;
  short4 output_size;
  for (int iSize = 0; iSize < size.size() - 1 ; iSize++)
  {
    int oSize = iSize + 1;
    while (oSize < size.size())
    {

      // std::cout << std::endl << "Testing padding from  " << size[iSize] << " to " << size[oSize] << std::endl;
      if (do_increase_size)
      {
        input_size  = make_short4(size[iSize],size[iSize],1,0);
        output_size = make_short4(size[oSize],size[oSize],1,0);  
      }
      else
      {
        output_size = make_short4(size[iSize],size[iSize],1,0);
        input_size  = make_short4(size[oSize],size[oSize],1,0);  
      }


  bool test_passed = true;
  long address = 0;

  float sum;
  float2 sum_complex;

  Image< float, float2 > host_input(input_size);
  Image< float, float2 > host_output(output_size);
  Image< float, float2 > device_output(output_size);
  

  // We just make one instance of the FourierTransformer class, with calc type float.
  // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
  FastFFT::FourierTransformer<float, float, float> FT;
  // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
  FT.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, output_size.x,output_size.y,output_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
  FT.SetInverseFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural); 
 
  // The padding (dims.w) is calculated based on the setup
  short4 dims_in = FT.ReturnFwdInputDimensions();
  short4 dims_out = FT.ReturnFwdOutputDimensions();
  // Determine how much memory we need, working with FFTW/CUDA style in place transform padding.
  // Note: there is no reason we really need this, because the xforms will always be out of place. 
  //       For now, this is just in place because all memory in cisTEM is allocated accordingly.
  host_input.real_memory_allocated = FT.ReturnInputMemorySize();
  host_output.real_memory_allocated = FT.ReturnInvOutputMemorySize();



  // On the device, we will always allocate enough memory for the larger of input/output including the buffer array.
  // Minmize the number of calls to malloc which are slow and can lead to fragmentation.
  device_output.real_memory_allocated = std::max(host_input.real_memory_allocated, host_output.real_memory_allocated);
  
  // In your own programs, you will be handling this memory allocation yourself. We'll just make something here.
  // I think fftwf_malloc may potentially create a different alignment than new/delete, but kinda doubt it. For cisTEM consistency...
  bool set_fftw_plan = true;
  host_input.Allocate(set_fftw_plan);
  host_output.Allocate(set_fftw_plan);
  
  // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
  // ensures faster transfer. If false, it will be pinned for you.
  FT.SetInputPointer(host_input.real_values, false);
  
  // Set a unit impulse at the center of the input array.
  FT.SetToConstant<float>(host_input.real_values, host_input.real_memory_allocated, 0.0f);
  FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 0.0f);

  sum = ReturnSumOfReal(host_output.real_values, dims_out);
  // host_input.real_values[ dims_in.y/2 * (dims_in.x+host_input.padding_jump_value) + dims_in.x/2] = 1.0f;
  // short4 wanted_center = make_short4(0,0,0,0);
  // ClipInto(host_input.real_values, host_output.real_values, dims_in ,  dims_out,  wanted_center, 0.f);

  // FT.SetToConstant<float>(host_input.real_values, host_input.real_memory_allocated, 0.0f);
  host_input.real_values[0] = 1.0f;
  host_output.real_values[0] = 1.0f;


  sum = ReturnSumOfReal(host_output.real_values, dims_out);
  if (sum != 1) {all_passed = false; init_passed[iSize] = false;}

  // MyFFTDebugAssertTestTrue( sum == 1,"Unit impulse Init ");
  
  // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
  FT.CopyHostToDevice();

  host_output.FwdFFT();
  
  host_output.fftw_epsilon = ReturnSumOfComplexAmplitudes(host_output.complex_values, host_output.real_memory_allocated/2);  
  // std::cout << "host " << host_output.fftw_epsilon << " " << host_output.real_memory_allocated<< std::endl;

  host_output.fftw_epsilon -= (host_output.real_memory_allocated/2 );
  if (std::abs(host_output.fftw_epsilon) > 1e-8 ) {all_passed = false; FFTW_passed[iSize] = false;}

  // MyFFTDebugAssertTestTrue( std::abs(host_output.fftw_epsilon) < 1e-8 , "FFTW unit impulse forward FFT");
  
  // Just to make sure we don't get a false positive, set the host memory to some undesired value.
  FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 2.0f);
  
  // This method will call the regular FFT kernels given the input/output dimensions are equal when the class is instantiated.
  bool swap_real_space_quadrants = true;
  
  FT.FwdFFT(swap_real_space_quadrants);

  int n=0;
  if (do_increase_size)
  {
    FT.CopyDeviceToHost(host_output.real_values, false, false);

      #if DEBUG_FFT_STAGE == 0
        PrintArray(host_output.real_values,  dims_in.x, dims_in.y, dims_in.z, dims_in.w);
        MyTestPrintAndExit( "stage 0 " );
      #elif DEBUG_FFT_STAGE == 1
        // If we are doing a fwd increase, the data will have only been expanded along the (transposed) X dimension at this point
        // So the (apparent) X is dims_in.y not dims_out.y
        PrintArray(host_output.complex_values, dims_in.y, dims_in.z, dims_out.w);
        MyTestPrintAndExit( "stage 1 " );
      #elif DEBUG_FFT_STAGE == 2
        // If we are doing a fwd increase, the data will have only been expanded along the (transposed) X dimension at this point
        // So the (apparent) X is dims_in.y not dims_out.y
        PrintArray(host_output.complex_values, dims_in.y, dims_out.z, dims_out.w);
        MyTestPrintAndExit( "stage 2 " );
      #elif DEBUG_FFT_STAGE == 3
        // Now the array is fully expanded to dims_out, but still transposed
        PrintArray(host_output.complex_values, dims_out.y, dims_out.z, dims_out.w);
        MyTestPrintAndExit( "stage 3 " );
      #endif    
      sum = ReturnSumOfComplexAmplitudes(host_output.complex_values, host_output.real_memory_allocated/2); 
  
  }
  else
  {
    FT.CopyDeviceToHost(false, false, FT.ReturnInputMemorySize());
    #if DEBUG_FFT_STAGE == 0
      PrintArray(host_input.real_values, dims_in.x, dims_in.y, dims_in.z, dims_in.w);
      MyTestPrintAndExit( "stage 0 " );
    #elif DEBUG_FFT_STAGE == 1
      // If we are doing a fwd increase, the data will have only been expanded along the (transposed) X dimension at this point
      // So the (apparent) X is dims_in.y not dims_out.y
      PrintArray(host_input.complex_values, dims_in.y, dims_in.z, dims_out.w);
      MyTestPrintAndExit( "stage 1 " );
    #elif DEBUG_FFT_STAGE == 2
      // If we are doing a fwd increase, the data will have only been expanded along the (transposed) X dimension at this point
      // So the (apparent) X is dims_in.y not dims_out.y
      PrintArray(host_input.complex_values, dims_in.y, dims_out.z, dims_out.w);
      MyTestPrintAndExit( "stage 2 " );      
    #elif DEBUG_FFT_STAGE == 3
      // Now the array is fully expanded to dims_out, but still transposed
      PrintArray(host_input.complex_values, dims_out.y, dims_out.z, dims_out.w);
      MyTestPrintAndExit( "stage 3 " );
    #endif   
    sum = ReturnSumOfComplexAmplitudes(host_input.complex_values, host_input.real_memory_allocated/2); 

  }


  sum -= (host_output.real_memory_allocated/2 );

  // std::cout << "sum " << sum << std::endl;
  // std::cout << "FFT Unit Impulse Forward FFT: " << sum <<  " epsilon " << host_output.fftw_epsilon << std::endl;
  // std::cout << "epsilon " << abs(sum - host_output.fftw_epsilon) << std::endl;
  if (abs(sum) > 1e-8) {all_passed = false; FastFFT_forward_passed[iSize] = false;}

  // MyFFTDebugAssertTestTrue( abs(sum - host_output.fftw_epsilon) < 1e-8, "FastFFT unit impulse forward FFT");
  FT.SetToConstant<float>(host_output.real_values, host_output.real_memory_allocated, 2.0f);
  

  FT.InvFFT();
  FT.CopyDeviceToHost(host_output.real_values, true, true);

  
  #if DEBUG_FFT_STAGE == 5
    PrintArray(host_output.complex_values, dims_out.y, dims_out.z, dims_out.w);
    MyTestPrintAndExit( "stage 5 " );
  #endif
  #if DEBUG_FFT_STAGE == 6
    PrintArray(host_output.complex_values, dims_out.y, dims_out.z, dims_out.w);
    MyTestPrintAndExit( "stage 6 " );    
  #elif DEBUG_FFT_STAGE == 7
    PrintArray(host_output.real_values, dims_out.x, dims_out.y, dims_out.z, dims_out.w);
    MyTestPrintAndExit( "stage 7 " );
  #elif DEBUG_FFT_STAGE > 7
    // No debug, keep going      
  #else
    MyTestPrintAndExit( " This block is only valid for DEBUG_FFT_STAGE == 3 || 4 " );
  #endif   

  sum = ReturnSumOfReal(host_output.real_values, dims_out);
  if (sum != dims_out.x*dims_out.y*dims_out.z) {all_passed = false; FastFFT_roundTrip_passed[iSize] = false;}

  // std::cout << "size in/out " << dims_in.x << ", " << dims_out.x << std::endl;
  // MyFFTDebugAssertTestTrue( sum == dims_out.x*dims_out.y*dims_out.z,"FastFFT unit impulse round trip FFT");

    oSize++;
    } // while loop over pad to size
  } // for loop over pad from size


  if (all_passed)
  {
    if ( ! do_increase_size) std::cout << "    All size_decrease unit impulse tests passed!" << std::endl;
    else std::cout << "    All size_increase unit impulse tests passed!" << std::endl;
  }
  else  
  {
    for (int n = 0; n < size.size() ; n++)
    {
      if ( ! init_passed[n] ) std::cout << "    Initialization failed for size " << size[n] << std::endl;
      if ( ! FFTW_passed[n] ) std::cout << "    FFTW failed for size " << size[n] << std::endl;
      if ( ! FastFFT_forward_passed[n] ) std::cout << "    FastFFT failed for forward transform size " << size[n] << std::endl;
      if ( ! FastFFT_roundTrip_passed[n] ) std::cout << "    FastFFT failed for roundtrip transform size " << size[n] << std::endl;

    }
  }

}

void compare_libraries(std::vector<int>size, int size_change_type)
{

  bool skip_cufft_for_profiling = false;
  bool set_padding_callback = false; // the padding callback is slower than pasting in b/c the read size of the pointers is larger than the actual data. do not use.
  bool set_conjMult_callback = true;
  bool is_size_change_decrease = false;

  if (size_change_type < 0) { is_size_change_decrease = true; }

  short4 input_size;
  short4 output_size;
  for (int iSize = 0; iSize < size.size() - 1 ; iSize++)
  {
    int oSize;
    int loop_size;
    if (size_change_type != 0) 
    {
      oSize = iSize + 1;
      loop_size = size.size();

    }
    else 
    {
      oSize = iSize;
      loop_size = oSize + 1;
    }

    while (oSize < loop_size)
    {

      if (is_size_change_decrease)
      {
        output_size = make_short4(size[iSize],size[iSize],1,0);
        input_size  = make_short4(size[oSize],size[oSize],1,0);  
      }
      else
      {
        input_size  = make_short4(size[iSize],size[iSize],1,0);
        output_size = make_short4(size[oSize],size[oSize],1,0);  

      }
      std::cout << std::endl << "Testing padding from  " << input_size.x << " to " << output_size.x << std::endl;



      if ( ( input_size.x == output_size.x && input_size.y == output_size.y && input_size.z == output_size.z ) )
      {
        // Also will change the path called in FastFFT to just be fwd/inv xform.
        set_conjMult_callback = false;
      }

      bool test_passed = true;
      long address = 0;

      float sum;
      float2 sum_complex;

      Image< float, float2 > FT_input(input_size);
      Image< float, float2 > FT_output(output_size);
      Image< float, float2 > cuFFT_input(input_size);
      Image< float, float2 > cuFFT_output(output_size);

      short4 target_size;

      if (is_size_change_decrease) target_size = input_size; // assuming xcorr_fwd_NONE_inv_DECREASE
      else target_size = output_size;


    
      Image< float, float2> target_search_image(target_size);
      Image< float, float2> positive_control(target_size);


      // We just make one instance of the FourierTransformer class, with calc type float.
      // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
      FastFFT::FourierTransformer<float, float, float> FT;
        // Create an instance to copy memory also for the cufft tests.
      FastFFT::FourierTransformer<float, float, float> cuFFT;
      FastFFT::FourierTransformer<float, float, float> targetFT;

      if ( is_size_change_decrease )
      {
        FT.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, input_size.x,input_size.y,input_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        FT.SetInverseFFTPlan(input_size.x,input_size.y,input_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);

        // For the subset of outputs this is just the input size, assuming the program then accesses just the valid data (could explicitly put into a new array which would be even slower.)
        cuFFT.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, input_size.x,input_size.y,input_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        cuFFT.SetInverseFFTPlan(input_size.x,input_size.y,input_size.z, input_size.x,input_size.y,input_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        
        targetFT.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, input_size.x,input_size.y,input_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        targetFT.SetInverseFFTPlan(input_size.x,input_size.y,input_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);        
      }
      else
      {
        FT.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, output_size.x,output_size.y,output_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        FT.SetInverseFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);

        cuFFT.SetForwardFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        cuFFT.SetInverseFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        
        targetFT.SetForwardFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
        targetFT.SetInverseFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
      }

      short4 fwd_dims_in = FT.ReturnFwdInputDimensions();
      short4 fwd_dims_out = FT.ReturnFwdOutputDimensions();
      short4 inv_dims_in = FT.ReturnInvInputDimensions();
      short4 inv_dims_out = FT.ReturnInvOutputDimensions();

      FT_input.real_memory_allocated = FT.ReturnInputMemorySize();
      FT_output.real_memory_allocated = FT.ReturnInvOutputMemorySize();


      cuFFT_input.real_memory_allocated = cuFFT.ReturnInputMemorySize();
      cuFFT_output.real_memory_allocated = cuFFT.ReturnInvOutputMemorySize();


      if (is_size_change_decrease) target_search_image.real_memory_allocated = targetFT.ReturnInputMemorySize();
      else target_search_image.real_memory_allocated = targetFT.ReturnInvOutputMemorySize(); // the larger of the two.

      positive_control.real_memory_allocated = target_search_image.real_memory_allocated; // this won't change size


      bool set_fftw_plan = false;
      FT_input.Allocate(set_fftw_plan);
      FT_output.Allocate(set_fftw_plan);

      cuFFT_input.Allocate(set_fftw_plan);
      cuFFT_output.Allocate(set_fftw_plan);

      target_search_image.Allocate(true);
      positive_control.Allocate(true);


      // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
      // ensures faster transfer. If false, it will be pinned for you.
      FT.SetInputPointer(FT_input.real_values, false);
      cuFFT.SetInputPointer(cuFFT_input.real_values, false);
      targetFT.SetInputPointer(target_search_image.real_values, false);

      // Set a unit impulse at the center of the input array.
      // For now just considering the real space image to have been implicitly quadrant swapped so the center is at the origin.
      FT.SetToConstant<float>(FT_input.real_values, FT_input.real_memory_allocated, 0.0f);
      FT.SetToConstant<float>(cuFFT_input.real_values, cuFFT_input.real_memory_allocated, 0.0f);
      FT.SetToConstant<float>(FT_output.real_values, FT_output.real_memory_allocated, 0.0f);
      FT.SetToConstant<float>(cuFFT_output.real_values, cuFFT_output.real_memory_allocated, 0.0f);
      FT.SetToConstant<float>(target_search_image.real_values, target_search_image.real_memory_allocated, 0.0f);
      FT.SetToConstant<float>(positive_control.real_values, target_search_image.real_memory_allocated, 0.0f);

  
      // Place these values at the origin of the image and after convolution, should be at 0,0,0.
      float testVal_1 = 2.0f;
      float testVal_2 = 3.0f;
      FT_input.real_values[0] = testVal_1;
      cuFFT_input.real_values[0] = testVal_1;
      target_search_image.real_values[0] = testVal_2;//target_search_image.size.w*2*target_search_image.size.y/2 + target_search_image.size.x/2] = testVal_2;
      positive_control.real_values[0] = testVal_1;//target_search_image.size.w*2*target_search_image.size.y/2 + target_search_image.size.x/2] = testVal_1;



      // Transform the target on the host prior to transfer.
      target_search_image.FwdFFT();


      // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
      FT.CopyHostToDevice();
      FT.CopyDeviceToHost(false, false);

      

      cuFFT.CopyHostToDevice();
      

      targetFT.CopyHostToDevice();
      

      // Wait on the transfers to finish.
      cudaErr(hipStreamSynchronize(hipStreamPerThread));  

      // Positive control on the host.
      positive_control.FwdFFT();
      positive_control.MultiplyConjugateImage(target_search_image.complex_values);
      positive_control.InvFFT();



      // address = 0;
      test_passed = true;
      for (int z = 1; z <  positive_control.size.z ; z++)
      {   
        for (int y = 1; y < positive_control.size.y; y++)
        {  
          for (int x = 1; x < positive_control.size.x; x++)
          {
            if (positive_control.real_values[address] != 0.0f) test_passed = false;
          }
        }
      }
      if (test_passed) 
      {
        if (positive_control.real_values[address] == positive_control.size.x*positive_control.size.y*positive_control.size.z*testVal_1*testVal_2)
        {
          std::cout << "Test passed for FFTW positive control.\n" << std::endl;
        }
        else
        {
          std::cout << "Test failed for FFTW positive control. Value at zero is  " << positive_control.real_values[address] << std::endl;
        }
      }
      else
      {
        std::cout << "Test failed for positive control, non-zero values found away from the origin." << std::endl;
      }


      cuFFT_output.create_timing_events(); 
      cuFFT_input.MakeCufftPlan();
      cuFFT_output.MakeCufftPlan();


      //////////////////////////////////////////
      //////////////////////////////////////////
      // Warm up and check for accuracy
      if (set_conjMult_callback || is_size_change_decrease ) // we set set_conjMult_callback = false 
      {
        FT.CrossCorrelate(targetFT.d_ptr.momentum_space, false);
      }
      else
      {
        FT.FwdFFT();
        FT.InvFFT();
      }      


      
      if (is_size_change_decrease)
      {
        FT.CopyDeviceToHost(false, false);
        #if DEBUG_FFT_STAGE == 0

          PrintArray(FT_input.real_values, fwd_dims_in.x, fwd_dims_in.y, fwd_dims_in.z, fwd_dims_in.w);
          MyTestPrintAndExit(" Stage 0");
        #elif DEBUG_FFT_STAGE == 1

          PrintArray(FT_input.complex_values, fwd_dims_in.y, fwd_dims_in.z, fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 1");
        #elif DEBUG_FFT_STAGE == 2
          PrintArray(FT_input.complex_values, fwd_dims_in.y, fwd_dims_out.z, fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 2");          
        #elif DEBUG_FFT_STAGE == 3

          PrintArray(FT_input.complex_values, fwd_dims_in.y, fwd_dims_out.z, fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 3");
        #elif DEBUG_FFT_STAGE == 4
        
          PrintArray(FT_input.complex_values, fwd_dims_in.y, fwd_dims_out.z,fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 4");
        #elif DEBUG_FFT_STAGE == 5
          PrintArray(FT_input.complex_values,inv_dims_out.y, inv_dims_in.z, inv_dims_in.w);
          MyTestPrintAndExit(" Stage 5");
        #elif DEBUG_FFT_STAGE == 6
          PrintArray(FT_input.complex_values,inv_dims_out.y, inv_dims_out.z, inv_dims_in.w);
          MyTestPrintAndExit(" Stage 6");          
        #elif DEBUG_FFT_STAGE == 7
          PrintArray(FT_input.real_values,inv_dims_out.x, inv_dims_out.y, inv_dims_out.z, inv_dims_out.w);
          MyTestPrintAndExit(" Stage 7");
        #elif DEBUG_FFT_STAGE > 7
          // Do nothing, we are doing all ops and not debugging.
        #else
          MyTestPrintAndExit("DEBUG_FFT_STAGE not recognized " + std::to_string(DEBUG_FFT_STAGE));
        #endif
      }
      else
      {
        // the output is equal or > the input, so we can always copy there.
        FT.CopyDeviceToHost(FT_output.real_values,false, false);

        #if DEBUG_FFT_STAGE == 0
          PrintArray(FT_output.real_values, fwd_dims_in.x, fwd_dims_in.y, fwd_dims_in.z, fwd_dims_in.w);
          MyTestPrintAndExit(" Stage 0");
        #elif DEBUG_FFT_STAGE == 1
          PrintArray(FT_output.complex_values, fwd_dims_in.y, fwd_dims_in.z, fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 1");
        #elif DEBUG_FFT_STAGE == 2
          PrintArray(FT_output.complex_values, fwd_dims_in.y, fwd_dims_out.z, fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 2");          
        #elif DEBUG_FFT_STAGE == 3
          PrintArray(FT_output.complex_values, fwd_dims_out.y, fwd_dims_out.z, fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 3");
        #elif DEBUG_FFT_STAGE == 4
          PrintArray(FT_output.complex_values, fwd_dims_out.y, fwd_dims_out.z, fwd_dims_out.w);
          MyTestPrintAndExit(" Stage 4");
        #elif DEBUG_FFT_STAGE == 5
          PrintArray(FT_output.complex_values, inv_dims_out.y, inv_dims_in.z, inv_dims_out.w);
          MyTestPrintAndExit(" Stage 5");
        #elif DEBUG_FFT_STAGE == 6
          PrintArray(FT_output.complex_values, inv_dims_out.y, inv_dims_out.z, inv_dims_out.w);
          MyTestPrintAndExit(" Stage 6");          
        #elif DEBUG_FFT_STAGE == 7
          PrintArray(FT_output.real_values, inv_dims_out.x, inv_dims_out.y, inv_dims_out.z,  inv_dims_out.w);
          MyTestPrintAndExit(" Stage 7");
        #elif DEBUG_FFT_STAGE > 7
          // Do nothing, we are doing all ops and not debugging.
        #else
          MyTestPrintAndExit("DEBUG_FFT_STAGE not recognized " + std::to_string(DEBUG_FFT_STAGE));
        #endif
      }
      
      address = 0;
      test_passed = true;
      if (is_size_change_decrease)
      {
        for (int z = 1; z <  FT_input.size.z ; z++)
        {   
          for (int y = 1; y < FT_input.size.y; y++)
          {  
            for (int x = 1; x < FT_input.size.x; x++)
            {
              if (FT_input.real_values[address] != 0.0f) test_passed = false;
            }
          }
        }
        if (test_passed) 
        {
          if (FT_input.real_values[address] == FT_input.size.x*FT_input.size.y*FT_input.size.z*testVal_1*testVal_2)
          {
            std::cout << "Test passed for FastFFT positive control.\n" << std::endl;
          }
          else
          {
            std::cout << "Test failed for FastFFT positive control. Value at zero is  " << FT_input.real_values[address] << std::endl;
          }
        }
        else
        {
          std::cout << "Test failed for FastFFT control, non-zero values found away from the origin." << std::endl;
        }
      }
      else
      {
        for (int z = 1; z <  FT_output.size.z ; z++)
        {   
          for (int y = 1; y < FT_output.size.y; y++)
          {  
            for (int x = 1; x < FT_output.size.x; x++)
            {
              if (FT_output.real_values[address] != 0.0f) test_passed = false;
            }
          }
        }
        if (test_passed) 
        {
          if (FT_output.real_values[address] == FT_output.size.x*FT_output.size.y*FT_output.size.z*testVal_1*testVal_2)
          {
            std::cout << "Test passed for FastFFT positive control.\n" << std::endl;
          }
          else
          {
            std::cout << "Test failed for FastFFT positive control. Value at zero is  " << FT_output.real_values[address] << std::endl;
          }
        }
        else
        {
          std::cout << "Test failed for FastFFT control, non-zero values found away from the origin." << std::endl;
        }
      }

      ////////////////////////////////////////
      //////////////////////////////////////////

      #if DEBUG_FFT_STAGE == 0

          PrintArray(FT_output.real_values, fwd_dims_in.x, fwd_dims_in.y, fwd_dims_in.z, fwd_dims_in.w);

        MyTestPrintAndExit( "stage 0 " );
      #elif DEBUG_FFT_STAGE == 1
        // If we are doing a fwd increase, the data will have only been expanded along the (transposed) X dimension at this point
        // So the (apparent) X is dims_in.y not output_size.y
        // decrease is currently just tested on the output. Really, to simplify there should be 3 different functions, fwd_none_inv_decrease (current decrease), fwd_decrease_inc_decrease (not yet) fwd_increase_inv_none
        if (is_size_change_decrease) 
        {
          MyTestPrintAndExit( "stage 1 decrease" );
          PrintArray(FT_output.complex_values, fwd_dims_out.y, fwd_dims_out.z, fwd_dims_out.w);
        }
        else 
        {
          MyTestPrintAndExit( "stage 1 increase" );
          PrintArray(FT_output.complex_values, fwd_dims_in.y, fwd_dims_in.z, fwd_dims_out.w);
        }

      #elif DEBUG_FFT_STAGE == 2
        // Now the array is fully expanded to output_size, but still transposed
        PrintArray(FT_output.complex_values, fwd_dims_out.y, fwd_dims_out.z, fwd_dims_out.w);
        MyTestPrintAndExit( "stage 2 " );

      #elif DEBUG_FFT_STAGE == 3
        PrintArray(FT_output.complex_values,inv_dims_out.y, inv_dims_out.z, inv_dims_out.w);
        MyTestPrintAndExit( "stage 3 " );
      #elif DEBUG_FFT_STAGE == 4
        PrintArray(FT_output.real_values, inv_dims_out.x, inv_dims_out.y, inv_dims_out.z, inv_dims_out.w);
        MyTestPrintAndExit( "stage 4 " );
      #elif DEBUG_FFT_STAGE > 7
        // This is the final stage, the data is fully expanded and transposed
      #else
        MyTestPrintAndExit( "This blah blah");
      #endif   



      const int n_loops = 3000;
      cuFFT_output.record_start();
      for (int i = 0; i < n_loops; ++i)
      {
        if (set_conjMult_callback || is_size_change_decrease )
        {
          FT.CrossCorrelate(targetFT.d_ptr.momentum_space_buffer, false);
        }
        else
        {
          FT.FwdFFT();
          FT.InvFFT();
        }
      }
      cuFFT_output.record_stop();
      cuFFT_output.synchronize();
      cuFFT_output.print_time("FastFFT");
      float FastFFT_time = cuFFT_output.elapsed_gpu_ms;

      if (set_padding_callback) 
      {
        precheck
        hipfftReal* overlap_pointer;
        overlap_pointer = cuFFT.d_ptr.position_space;
        cuFFT_output.SetClipIntoCallback(overlap_pointer, cuFFT_input.size.x, cuFFT_input.size.y, cuFFT_input.size.w*2);
        postcheck
      }

      if (set_conjMult_callback)
      {
        precheck
        // FIXME scaling factor
        cuFFT_output.SetComplexConjMultiplyAndLoadCallBack( (hipfftComplex *) targetFT.d_ptr.momentum_space_buffer, 1.0f);
        postcheck
      }

      

      if (! skip_cufft_for_profiling)
      {
        //////////////////////////////////////////
        //////////////////////////////////////////
        // Warm up and check for accuracy
        if (is_size_change_decrease)
        {

          precheck
          cudaErr(hipfftExecR2C(cuFFT_input.cuda_plan_forward, (hipfftReal*)cuFFT.d_ptr.position_space, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer));
          postcheck
          
  
          precheck
          cudaErr(hipfftExecC2R(cuFFT_input.cuda_plan_inverse, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer, (hipfftReal*)cuFFT.d_ptr.position_space));
          postcheck  
          
        }
        else
        {
          // cuFFT.ClipIntoTopLeft();
          // cuFFT.ClipIntoReal(cuFFT_output.size.x/2, cuFFT_output.size.y/2, cuFFT_output.size.z/2);
          // cuFFT.CopyDeviceToHost(cuFFT_output.real_values,false, false);

          precheck
          cudaErr(hipfftExecR2C(cuFFT_output.cuda_plan_forward, (hipfftReal*)cuFFT.d_ptr.position_space, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer));
          postcheck
          
  
          precheck
          cudaErr(hipfftExecC2R(cuFFT_output.cuda_plan_inverse, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer, (hipfftReal*)cuFFT.d_ptr.position_space));
          postcheck  
           
        }
        

        


        cuFFT_output.record_start();
        for (int i = 0; i < n_loops; ++i)
        {
          // std::cout << i << "i / " << n_loops << "n_loops" << std::endl;
          if (set_conjMult_callback) cuFFT.ClipIntoTopLeft();
          // cuFFT.ClipIntoReal(input_size.x/2, input_size.y/2, input_size.z/2);

          if (is_size_change_decrease)
          {
            precheck
            cudaErr(hipfftExecR2C(cuFFT_input.cuda_plan_forward, (hipfftReal*)cuFFT.d_ptr.position_space, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer));
            postcheck
  
            precheck
            cudaErr(hipfftExecC2R(cuFFT_input.cuda_plan_inverse, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer, (hipfftReal*)cuFFT.d_ptr.position_space));
            postcheck
          }
          else
          {
            precheck
            cudaErr(hipfftExecR2C(cuFFT_output.cuda_plan_forward, (hipfftReal*)cuFFT.d_ptr.position_space, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer));
            postcheck
  
            precheck
            cudaErr(hipfftExecC2R(cuFFT_output.cuda_plan_inverse, (hipfftComplex*)cuFFT.d_ptr.momentum_space_buffer, (hipfftReal*)cuFFT.d_ptr.position_space));
            postcheck
          }

        }
        cuFFT_output.record_stop();
        cuFFT_output.synchronize();
        cuFFT_output.print_time("cuFFT");
      } // end of if (! skip_cufft_for_profiling)
      std::cout << "For size " << input_size.x << " to "<< output_size.x << ": " << std::endl;
      std::cout << "Ratio cuFFT/FastFFT : " << cuFFT_output.elapsed_gpu_ms/FastFFT_time << std::endl;

      oSize++;
      // We don't want to loop if the size is not actually changing.
      } // while loop over pad to size
  } // for loop over pad from size

}

void run_oned(std::vector<int> size)
{

  // Override the size to be one dimensional in x
  std::cout << "Running one-dimensional tests\n" << std::endl;

  bool test_passed = true;
  long address = 0;

  float sum;
  float2 sum_complex;

  for (int n : size)
  {
    short4 input_size = make_short4(n,1,1,0);
    short4 output_size = make_short4(n,1,1,0);

    Image< float, float2 > FT_input(input_size);
    Image< float, float2 > FT_output(output_size);
    Image< float2, float2 > FT_input_complex(input_size);
    Image< float2, float2 > FT_output_complex(output_size);

    // We just make one instance of the FourierTransformer class, with calc type float.
    // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
    FastFFT::FourierTransformer<float, float, float> FT;
    FastFFT::FourierTransformer<float, float2, float2> FT_complex;

    // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
    FT.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, output_size.x,output_size.y,output_size.z, true, false, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);
    FT.SetInverseFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float ,float>::OriginType::natural);

    FT_complex.SetForwardFFTPlan(input_size.x,input_size.y,input_size.z, output_size.x,output_size.y,output_size.z, true, false, FastFFT::FourierTransformer<float, float2 ,float2>::OriginType::natural);
    FT_complex.SetInverseFFTPlan(output_size.x,output_size.y,output_size.z, output_size.x,output_size.y,output_size.z, true, FastFFT::FourierTransformer<float, float2 ,float2>::OriginType::natural);

    FT_input.real_memory_allocated = FT.ReturnInputMemorySize();
    FT_output.real_memory_allocated = FT.ReturnInvOutputMemorySize();

    FT_input_complex.real_memory_allocated = FT_complex.ReturnInputMemorySize();
    FT_output_complex.real_memory_allocated = FT_complex.ReturnInvOutputMemorySize();
    std::cout << "Allocated " << FT_input_complex.real_memory_allocated << " bytes for input.\n";
    std::cout << "Allocated complex " << FT_output_complex.real_memory_allocated << " bytes for input.\n";

    bool set_fftw_plan = true;
    FT_input.Allocate(set_fftw_plan);
    FT_output.Allocate(set_fftw_plan);

    FT_input_complex.Allocate();
    FT_output_complex.Allocate();



    // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
    // ensures faster transfer. If false, it will be pinned for you.
    FT.SetInputPointer(FT_input.real_values, false);
    FT_complex.SetInputPointer(FT_input_complex.complex_values, false);


    FT.SetToConstant<float>(FT_input.real_values, FT_input.real_memory_allocated, 1.f);

    // Set a unit impulse at the center of the input array.
    // FT.SetToConstant<float>(FT_input.real_values, FT_input.real_memory_allocated, 1.0f);
    float2 const_val = make_float2(1.0f,0.0f);
    FT_complex.SetToConstant<float2>(FT_input_complex.complex_values, FT_input.real_memory_allocated, const_val);
    for (int i=0; i<10; i++)
    {
      std::cout << FT_input_complex.complex_values[i].x << "," << FT_input_complex.complex_values[i].y << std::endl;
    }


    FT.CopyHostToDevice();
    FT_complex.CopyHostToDevice();
    cudaErr(hipStreamSynchronize(hipStreamPerThread));  

        // Set the outputs to a clearly wrong answer.
        FT.SetToConstant<float>(FT_output.real_values, FT_input.real_memory_allocated, 2.0f);
        const_val = make_float2(2.0f,2.0f);
        FT_complex.SetToConstant<float2>(FT_output_complex.complex_values, FT_output.real_memory_allocated, const_val);

    FT_input.FwdFFT();

    for (int i = 0; i < 5; ++i) std::cout << "FFTW fwd " << FT_input.real_values[i] << std::endl;
    std::cout << std::endl;


    bool transpose_output = false;
    bool swap_real_space_quadrants = false;
    FT.FwdFFT(swap_real_space_quadrants, transpose_output);
    FT_complex.FwdFFT(swap_real_space_quadrants, transpose_output);

    FT.CopyDeviceToHost(FT_output.real_values, false, false);
    FT_complex.CopyDeviceToHost(FT_output_complex.real_values, false, false);

    for (int i = 0; i < 10; ++i) {std::cout << "FT fwd " << FT_output.real_values[i] << std::endl;}
    for (int i = 0; i < 10; ++i) {std::cout << "FT complex fwd "<< FT_output_complex.real_values[i].x << "," << FT_output_complex.real_values[i].y << std::endl;}
  

    FT_input.InvFFT();

    for (int i = 0; i < 5; ++i) {std::cout << "FFTW inv " << FT_input.real_values[i] << std::endl;}
    std::cout << std::endl;



    FT.InvFFT(transpose_output);
    FT_complex.InvFFT(transpose_output);
    FT.CopyDeviceToHost(FT_output.real_values, true, true);
    FT_complex.CopyDeviceToHost(FT_output_complex.real_values, true, true);

    for (int i = 0; i < 10; i++) {std::cout << "Ft inv " << FT_output.real_values[i] << std::endl;}
    for (int i = 0; i < 10; i++) {std::cout << "Ft complex inv " << FT_output_complex.real_values[i].x << "," << FT_output_complex.real_values[i].y << std::endl;}


  }


}

int main(int argc, char** argv) 
{

  std::printf("Entering main in tests.cpp\n");
  std::printf("Standard is %i\n\n",__cplusplus);


  bool run_validation_tests;
  bool run_performance_tests;

  if (argc > 1)
  {
    run_validation_tests = false;
    run_performance_tests = true;
    std::cout << "Running performance tests.\n";
  }
  else
  {
    run_validation_tests = true;
    run_performance_tests = false;
  }
  // Input and output dimensions, with simple checks. I'm sure there are better checks on argv.
  short4 input_size;
  short4 output_size;

  std::vector<int> test_size = { 16, 32, 64, 128, 256, 512, 1024, 2048, 4096};
  std::vector<int> test_size_3d = { 16, 32, 64, 128, 256, 512};

  // The launch parameters fail for 4096 -> < 64 for r2c_decrease, not sure if it is the elements_per_thread or something else.
  // For now, just over-ride these small sizes
  std::vector<int> test_size_for_decrease = { 64, 128, 256, 512, 1024, 2048, 4096};



  if (run_validation_tests)  {

    // change onde these to just report the pass/fail.
    // run_oned(test_size);
    // exit(0);

    bool do_3d = true;
    const_image_test(test_size_3d, do_3d);
    exit(0);

    do_3d = false;
    const_image_test(test_size, do_3d);
    unit_impulse_test(test_size, true);
    unit_impulse_test(test_size_for_decrease, false);


  } // end of validation tests


  if (run_performance_tests) {

    #ifdef HEAVYERRORCHECKING_FFT
      std::cout << "Running performance tests with heavy error checking.\n";
      std::cout << "This doesn't make sense as the synchronizations are invalidating.\n";
      // exit(1);
    #endif

    int size_change_type = 0; // no change

    // compare_libraries(test_size, size_change_type);

    // size_change_type = 1; // increase
    // compare_libraries(test_size, size_change_type);

    size_change_type = -1; // decrease
    compare_libraries(test_size, size_change_type);


  }
  return 0;
};

