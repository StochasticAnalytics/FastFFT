#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <cmath>
#include <string>

// sudo apt-get install libfftw3-dev libfftw3-doc
#include <fftw3.h>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "FastFFT.cu"

// To print a message and some number n_to_print complex values to stdout
void print_values_complex(float* input, std::string msg, int n_to_print)
{
  for (int i = 0; i < n_to_print*2; i+=2) { std::cout << msg << i/2 << "  " << input[i] << " " << input[i+1] << std::endl ;}
}

// Return sum of real values
float ReturnSumOfReal(float* input, short4 size)
{
  double temp_sum = 0;
  long address = 0;
  int padding_jump_val = size.w*2 - size.x;
  for (int k = 0; k < size.z; k++)
  {
    for (int j = 0; j < size.y; j++)
    {
      for (int i = 0; i < size.x; i++)
      {
        temp_sum += (input[address]);
        address++;
      }
      address += padding_jump_val;
    }
  }

  return float(temp_sum);
}

// Return the sum of the complex values
float2 ReturnSumOfComplex(float2* input, int n_to_print)
{
  double sum_x = 0;
  double sum_y = 0;

  for (int i = 0; i < n_to_print; i++) 
  {
    sum_x += input[i].x;
    sum_y += input[i].y;
  }
  
  return make_float2(float(sum_x), float(sum_y));  
}

void ClipInto(const float* array_to_paste, float* array_to_paste_into, short4 size_from, short4 size_into, short4 wanted_center, float wanted_padding_value)
{


	long pixel_counter = 0;

	int kk;
	int k;
	int kk_logi;

	int jj;
	int jj_logi;
	int j;

	int ii;
	int ii_logi;
	int i;

	double junk;

  short4 center_to_paste_into = make_short4(size_into.x/2, size_into.y/2, size_into.z/2, 0);
  short4 center_to_paste = make_short4(size_from.x/2, size_from.y/2, size_from.z/2, 0);
  int padding_jump_value;

  if (size_into.x % 2 == 0) padding_jump_value = 2;
  else padding_jump_value = 1;

  for (kk = 0; kk < size_into.z; kk++)
  {
    kk_logi = kk - center_to_paste_into.z;
    k = center_to_paste.z + wanted_center.z + kk_logi;

    for (jj = 0; jj < size_into.y; jj++)
    {
      jj_logi = jj - center_to_paste_into.y;
      j = center_to_paste.y + wanted_center.y + jj_logi;

      for (ii = 0; ii < size_into.x; ii++)
      {
        ii_logi = ii - center_to_paste_into.x;
        i = center_to_paste.x + wanted_center.x + ii_logi;

        if (k < 0 || k >= size_from.z || j < 0 || j >= size_from.y || i < 0 || i >= size_from.x)
        {
          array_to_paste_into[pixel_counter] = wanted_padding_value;
        }
        else
        {
          array_to_paste_into[pixel_counter] = array_to_paste[ k*(size_from.w*2 *size_from.y) + j*(size_from.x+padding_jump_value) + i];
        }

        pixel_counter++;
      }

      pixel_counter+=padding_jump_value;
    }
  }
	


} // end of clip into

void unit_impulse_test(short4 input_size, short4 output_size)
{
  // Pointers to the arrays on the host -- maybe make this a struct of some sort? I'm sure there is a parallel in cuda, look into cuarray/texture code
  float* host_input;
  float* host_output;
  float2* host_input_complex;
  float2* host_output_complex;
  int host_input_memory_allocated;
  int host_output_memory_allocated;


  // Pointers to the arrays on the device
  float* device_input;
  float* device_output;
  float2* device_input_complex;
  float2* device_output_complex;
  int device_memory_allocated;

  float sum;
  float2 sum_complex;


  // We just make one instance of the FourierTransformer class, with calc type float.
  // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
  FastFFT::FourierTransformer FT(FastFFT::FourierTransformer::DataType::fp32);


  // Determine how much memory we need, working with FFTW/CUDA style in place transform padding.
  // Note: there is no reason we really need this, because the xforms will always be out of place. 
  //       For now, this is just in place because all memory in cisTEM is allocated accordingly.
  host_input_memory_allocated = FT.ReturnPaddedMemorySize(input_size);
  host_output_memory_allocated = FT.ReturnPaddedMemorySize(output_size);
  
  // On the device, we will always allocate enough memory for the larger of input/output including the buffer array.
  // Minmize the number of calls to malloc which are slow and can lead to fragmentation.
  device_memory_allocated = std::max(host_input_memory_allocated, host_output_memory_allocated);


  // In your own programs, you will be handling this memory allocation yourself. We'll just make something here.
  // I think fftwf_malloc may potentially create a different alignment than new/delete, but kinda doubt it. For cisTEM consistency...
	host_input = (float *) fftwf_malloc(sizeof(float) * host_input_memory_allocated);
	host_input_complex = (float2*) host_input;  // Set the complex_values to point at the newly allocated real values;

  host_output = (float *) fftwf_malloc(sizeof(float) * host_output_memory_allocated);
	host_output_complex = (float2*) host_input;  // Set the complex_values to point at the newly allocated real values;
  
  // Make FFTW plans for comparing CPU to GPU xforms.
  // This is nearly verbatim from cisTEM::Image::Allocate - I do not know if FFTW_ESTIMATE is the best option.
  // In cisTEM we almost always use MKL, so this might be worth testing. I always used exhaustive in Matlab/emClarity.
  fftwf_plan plan_fwd = NULL;
  fftwf_plan plan_bwd = NULL;
	plan_fwd = fftwf_plan_dft_r2c_3d(output_size.z, output_size.y, output_size.x, host_input, reinterpret_cast<fftwf_complex*>(host_input_complex), FFTW_ESTIMATE);
  plan_bwd = fftwf_plan_dft_c2r_3d(output_size.z, output_size.y, output_size.x, reinterpret_cast<fftwf_complex*>(host_input_complex), host_input, FFTW_ESTIMATE);
  
  // Set our input host memory to a constant. Then FFT[0] = host_input_memory_allocated
  FT.SetToConstant<float>(host_input, host_input_memory_allocated, 1.0f);
  
  // short4 wanted_center = make_short4(input_size.x/2, input_size.y/2, input_size.z/2, 0);
  short4 wanted_center = make_short4(0,0,0, 0);
  ClipInto(host_input, host_output, input_size, output_size, wanted_center, 0.0f);
  // int padding_jump_value;
  // if (output_size.x % 2 == 0) padding_jump_value = 2;
  // else padding_jump_value = 1;
  // for (int i = 0; i < output_size.x; i++) 
  // { 
  //   for (int j = 0; j < output_size.y; j++)
  //   {
  //     std::cout << host_output[i + j*(padding_jump_value+output_size.x)] << " "; 
  //   }
  //   std::cout << std::endl; 
  // }
  // exit(-1);

  
  // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
	FT.SetInputDimensionsAndType(output_size.x,output_size.y,output_size.z,true, false,FastFFT::FourierTransformer::DataType::fp32, FastFFT::FourierTransformer::OriginType::natural);
	FT.SetOutputDimensionsAndType(output_size.x,output_size.y,output_size.z,true,FastFFT::FourierTransformer::DataType::fp32, FastFFT::FourierTransformer::OriginType::natural);
  
  // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
  // ensures faster transfer. If false, it will be pinned for you.
	FT.SetInputPointer(&host_output[0], false);
  sum = ReturnSumOfReal(host_output, output_size);
  MyFFTDebugAssertTestTrue( sum == input_size.x*input_size.y*input_size.z,"Unit impulse Init ");

  // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
	FT.CopyHostToDevice();
  
  // Now let's do the forward FFT on the host and check that the result is correct.
  fftwf_execute_dft_r2c(plan_fwd, host_output, reinterpret_cast<fftwf_complex*>(host_output_complex));
  print_values_complex(host_output, "fftw ", 5);
  
  sum_complex = ReturnSumOfComplex(host_output_complex, FT.output_memory_allocated/2);
  std::cout << sum_complex.x << " " << sum_complex.y << std::endl;
  MyFFTDebugAssertTestTrue( sum_complex.x == FT.input_number_of_real_values && sum_complex.y == 0, "FFTW unit impulse forward FFT");
  FT.SetToConstant<float>(host_output, host_output_memory_allocated, 2.0f);

  // Forward FFT ;
  // FT.FFT_R2C_WithPadding_Transposed();
  // FT.FFT_C2C_WithPadding();
  FT.FwdFFT();

  // in buffer, do not deallocate, do not unpin memory
	FT.CopyDeviceToHost(false, false, false);
  sum_complex = ReturnSumOfComplex(host_output_complex, FT.output_memory_allocated/2);
  // std::cout << sum_complex.x << " " << powf(input_size.x*input_size.y*input_size.z,2) << " " << std::endl;

  // for (int i = 0; i < output_size.w*output_size.x*2; i++) { std::cout << host_input[i] << " "; }
  MyFFTDebugAssertTestTrue( sum_complex.x == FT.output_number_of_real_values && sum_complex.y == 0, "FastFFT unit impulse forward FFT");
  FT.SetToConstant<float>(host_input, host_input_memory_allocated, 2.0f);

  // FT.FFT_C2C();
  // FT.FFT_C2R_Transposed();
  FT.InvFFT();

	FT.CopyDeviceToHost(false, true, true);

  // Assuming the outputs are always even dimensions, padding_jump_val is always 2.
  sum = ReturnSumOfReal(host_input, output_size);
  // std::cout << sum << " " << powf(input_size.x*input_size.y*input_size.z,2) << " " << std::endl;
  // for (int i = 0; i < output_size.w*output_size.x*2; i++) { std::cout << host_input[i] << " "; }

  MyFFTDebugAssertTestTrue( sum == powf(input_size.x*input_size.y*input_size.z,2),"FastFFT unit impulse round trip FFT");

  fftwf_free(host_input);
  fftwf_destroy_plan(plan_fwd);
  fftwf_destroy_plan(plan_bwd);

}

int main(int argc, char** argv) {

  std::printf("Entering main in tests.cpp\n");
  std::printf("Standard is %i\n\n",__cplusplus);

  // Input and output dimensions, with simple checks. I'm sure there are better checks on argv.
  short4 input_size;
  short4 output_size;

  constexpr const int n_tests = 4;
  int test_size[n_tests] = {64, 128, 256, 512};
  // for (int iSize = 0; iSize < n_tests; iSize++) {

  //   std::cout << std::endl << "Testing " << test_size[iSize] << " x" << std::endl;
  //   input_size = make_short4(test_size[iSize],test_size[iSize],1,0);
  //   output_size = make_short4(test_size[iSize],test_size[iSize],1,0);

  //   unit_impulse_test(input_size, output_size);

  // }

  for (int iSize = 0; iSize < n_tests - 1; iSize++) {
    int oSize = iSize + 1;
    while (oSize > iSize)
    {
      std::cout << std::endl << "Testing padding from   " << test_size[iSize] << " to " << test_size[oSize] << std::endl;
      input_size = make_short4(test_size[iSize],test_size[iSize],1,0);
      output_size = make_short4(test_size[oSize],test_size[oSize],1,0);
  
      unit_impulse_test(input_size, output_size);
    }


  }
  
}
