#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <cmath>
#include <string>

// sudo apt-get install libfftw3-dev libfftw3-doc
#include <fftw3.h>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "FastFFT.cu"

void print_values(float* input, std::string msg, int n_to_print)
{
  for (int i = 0; i < n_to_print; i++) { std::cout << msg << i << "  " << input[i] << std::endl ;}
}

void print_values_complex(float* input, std::string msg, int n_to_print)
{
  for (int i = 0; i < n_to_print*2; i+=2) { std::cout << msg << i/2 << "  " << input[i] << " " << input[i+1] << std::endl ;}
}

void print_values_matrix(float* input, std::string msg, int n_row, int n_col)
{
  std::cout << msg << std::endl;
  int padding_val;
  if (n_row % 2 == 0) { padding_val =  2; }
  else { padding_val =  1; }

  int address = (n_row + padding_val) * n_col;

  // print matrix rotated 90 degrees
  for (int y = n_col-1; y >= 0; y--) 
  {
    address -= (n_row+padding_val);
    for (int x = 0; x < n_row; x++)
    {
      std::cout << input[address] << " ";
      address++;
    }
    std::cout << std::endl;
    address -= (n_row+padding_val);
  }
  
}

void ReturnSumOfComplex(float2* input, float2& sum, int n_to_print)
{
  sum.x = 0.f;
  sum.y = 0.f;
  for (int i = 0; i < n_to_print; i++) 
  {
    sum.x += input[i].x;
    sum.y += input[i].y;
  }
  
}
int main(int argc, char** argv) {

  std::printf("Entering main in tests.cpp\n");
  std::printf("Standard is %i\n\n",__cplusplus);

  // Input and output dimensions, with simple checks. I'm sure there are better checks on argv.
  short4 input_size;
  short4 output_size;

  if ( argc != 4 && argc != 7) 
  { 
    std::cout << argc << std::endl;
    std::cout << "Usage: ./tests n_x n_y n_z [optionally 3 larger or smaller sizes, otherwise input_size=output_size]" << std::endl;
    exit(1);
  }
  else
  {
    for (int i = 1; i < 4; i++) { if (atoi(argv[i]) <=0) { std::cout << "Error: " << argv[i] << " is not a positive integer" << std::endl; exit(1);}}
    input_size  = make_short4( atoi(argv[1]), atoi(argv[2]), atoi(argv[3]), 0 );

    if (argc == 7) 
    {
      for (int i = 1; i < 4; i++) { if (atoi(argv[i+3]) <=0) { std::cout << "Error: " << argv[i+3] << " is not a positive integer" << std::endl; exit(1);}}
      output_size = make_short4( atoi(argv[4]), atoi(argv[5]), atoi(argv[6]), 0 );
    }
    else
    {
      output_size = input_size;
    }
  }


  // Pointers to the arrays on the host -- maybe make this a struct of some sort? I'm sure there is a parallel in cuda, look into cuarray/texture code
  float* host_input;
  float* host_output;
  float2* host_input_complex;
  float2* host_output_complex;
  int host_input_memory_allocated;
  int host_output_memory_allocated;


  // Pointers to the arrays on the device
  float* device_input;
  float* device_output;
  float2* device_input_complex;
  float2* device_output_complex;
  int device_memory_allocated;

  float2 sum;


  // We just make one instance of the FourierTransformer class, with calc type float.
  // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
  FastFFT::FourierTransformer FT(FastFFT::FourierTransformer::DataType::fp32);

  // Determine how much memory we need, working with FFTW/CUDA style in place transform padding.
  // Note: there is no reason we really need this, because the xforms will always be out of place. 
  //       For now, this is just in place because all memory in cisTEM is allocated accordingly.
  host_input_memory_allocated = FT.ReturnPaddedMemorySize(input_size);
  host_output_memory_allocated = FT.ReturnPaddedMemorySize(output_size);
  
  // On the device, we will always allocate enough memory for the larger of input/output including the buffer array.
  // Minmize the number of calls to malloc which are slow and can lead to fragmentation.
  device_memory_allocated = std::max(host_input_memory_allocated, host_output_memory_allocated);


  // In your own programs, you will be handling this memory allocation yourself. We'll just make something here.
  // I think fftwf_malloc may potentially create a different alignment than new/delete, but kinda doubt it. For cisTEM consistency...
	host_input = (float *) fftwf_malloc(sizeof(float) * host_input_memory_allocated);
	host_input_complex = (float2*) host_input;  // Set the complex_values to point at the newly allocated real values;
  
  // Make FFTW plans for comparing CPU to GPU xforms.
  // This is nearly verbatim from cisTEM::Image::Allocate - I do not know if FFTW_ESTIMATE is the best option.
  // In cisTEM we almost always use MKL, so this might be worth testing. I always used exhaustive in Matlab/emClarity.
  fftwf_plan plan_fwd = NULL;
  fftwf_plan plan_bwd = NULL;
	plan_fwd = fftwf_plan_dft_r2c_3d(input_size.z, input_size.y, input_size.x, host_input, reinterpret_cast<fftwf_complex*>(host_input_complex), FFTW_ESTIMATE);
  plan_bwd = fftwf_plan_dft_c2r_3d(input_size.z, input_size.y, input_size.x, reinterpret_cast<fftwf_complex*>(host_input_complex), host_input, FFTW_ESTIMATE);
  
  // Set our input host memory to a constant. Then FFT[0] = host_input_memory_allocated
  FT.SetToConstant<float>(host_input, host_input_memory_allocated, 1.0f);
  

  
  // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
	FT.SetInputDimensionsAndType(input_size.x,input_size.y,input_size.z,true, false,FastFFT::FourierTransformer::DataType::fp32, FastFFT::FourierTransformer::OriginType::natural);
	FT.SetOutputDimensionsAndType(input_size.x,input_size.y,input_size.z,true,FastFFT::FourierTransformer::DataType::fp32, FastFFT::FourierTransformer::OriginType::natural);
  
  // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
  // ensures faster transfer. If false, it will be pinned for you.
	FT.SetInputPointer(&host_input[0], false);
  ReturnSumOfComplex(host_input_complex, sum, FT.input_memory_allocated/2);
  MyFFTDebugAssertTestTrue( sum.x == FT.input_memory_allocated/2 && sum.y == FT.input_memory_allocated/2, "Unit impulse init");

  // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
	FT.CopyHostToDevice();
  
  // Now let's do the forward FFT on the host and check that the result is correct.
  fftwf_execute_dft_r2c(plan_fwd, host_input, reinterpret_cast<fftwf_complex*>(host_input_complex));
  print_values_complex(host_input, "fftw ", FT.output_memory_allocated/2);
  
  ReturnSumOfComplex(host_input_complex, sum, FT.output_memory_allocated/2);
  std::cout << sum.x << " " << sum.y << std::endl;
  MyFFTDebugAssertTestTrue( sum.x == FT.output_number_of_real_values && sum.y == 0, "FFTW unit impulse forward FFT");
  FT.SetToConstant<float>(host_input, host_input_memory_allocated, 2.0f);

  // FT.SimpleFFT_NoPadding();
  FT.FFT_R2C_Transposed();
  FT.FFT_C2C_WithPadding(true);
	FT.CopyDeviceToHost(false, true, true);

  ReturnSumOfComplex(host_input_complex, sum, FT.output_memory_allocated/2);
  std::cout << sum.x << " " << sum.y << std::endl;
  MyFFTDebugAssertTestTrue( sum.x == FT.output_number_of_real_values  && sum.y == 0,"FastFFT unit impulse forward FFT");

  fftwf_free(host_input);
  fftwf_destroy_plan(plan_fwd);
  fftwf_destroy_plan(plan_bwd);
  
}
