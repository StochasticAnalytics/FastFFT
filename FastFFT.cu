#include "hip/hip_runtime.h"
// Insert some license stuff here

// #include <string>
#include <iostream>
#include <string>
#include <stdio.h>
#include <cufftdx.hpp>

#include "FastFFT.h"
#include "FastFFT.cuh"



namespace FastFFT {



  ///////////////////////////////////////////////
  ///////////////////////////////////////////////



FourierTransformer::FourierTransformer(DataType wanted_calc_data_type) 
{


  MyFFTPrint( "Initializing FourierTrasnformer...");
  calc_data_type = wanted_calc_data_type;
  // Plan to allow fp16 and bf16
  MyFFTDebugAssertTrue(calc_data_type == DataType::fp32, "Only F32 is supported at the moment");
  SetDefaults();
}

FourierTransformer::~FourierTransformer() 
{
  Deallocate();
  UnPinHostMemory();
}

void FourierTransformer::SetDefaults()
{
  DataType input_data_type = fp32;
  DataType output_data_type = fp32;

  // booleans to track state, could be bit fields but that seem opaque to me.
  is_in_memory_host_pointer = false;
  is_in_memory_device_pointer = false;

  is_fftw_padded_input = false;
  is_fftw_padded_output = false;
  is_fftw_padded_buffer = false;

  is_set_input_params = false;
  is_set_output_params = false;

  is_host_memory_pinned = false;
}

void FourierTransformer::SetInputDimensionsAndType(size_t input_logical_x_dimension, 
                                                   size_t input_logical_y_dimension, 
                                                   size_t input_logical_z_dimension, 
                                                   bool is_padded_input, 
                                                   bool is_host_memory_pinned, 
                                                   DataType input_data_type,
                                                   OriginType input_origin_type)
{

  MyFFTDebugAssertTrue(input_logical_x_dimension > 0, "Input logical x dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_y_dimension > 0, "Input logical y dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_z_dimension > 0, "Input logical z dimension must be > 0");
  MyFFTDebugAssertTrue(is_padded_input, "The input memory must be fftw padded");

  short int w;
  if (is_padded_input)
  {
    if (input_logical_x_dimension % 2 == 0) w = 2;
    else w = 1;
  }
  else w = 0;

  dims_in = make_short4(input_logical_x_dimension, input_logical_y_dimension, input_logical_z_dimension,input_logical_x_dimension + w);

  input_memory_allocated = ReturnPaddedMemorySize(dims_in);
  this->input_origin_type = input_origin_type;
  is_set_input_params = true;
}

void FourierTransformer::SetOutputDimensionsAndType(size_t output_logical_x_dimension, 
                                                    size_t output_logical_y_dimension, 
                                                    size_t output_logical_z_dimension, 
                                                    bool is_padded_output, 
                                                    DataType output_data_type,
                                                    OriginType output_origin_type)
{
  MyFFTDebugAssertTrue(output_logical_x_dimension > 0, "output logical x dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_y_dimension > 0, "output logical y dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_z_dimension > 0, "output logical z dimension must be > 0");
  MyFFTDebugAssertTrue(is_padded_output, "The output memory must be fftw padded");

  short int w;
  if (is_padded_output)
  {
    if (output_logical_x_dimension % 2 == 0) w = 2;
    else w = 1;
  }
  else w = 0;

  dims_out = make_short4(output_logical_x_dimension, output_logical_y_dimension, output_logical_z_dimension,output_logical_x_dimension + w);

  output_memory_allocated = ReturnPaddedMemorySize(dims_out);

  this->output_origin_type = output_origin_type;
  is_set_output_params = true;
}




void FourierTransformer::SetInputPointer(float* input_pointer, bool is_input_on_device) 
{ 
  MyFFTDebugAssertTrue(calc_data_type == DataType::fp32, "Only F32 is supported at the moment");
  MyFFTDebugAssertTrue(is_set_input_params, "Input parameters not set");

  if ( is_input_on_device) 
  {
    // We'll need a check on compute type, and a conversion if needed prior to this.
    device_pointer_fp32 = input_pointer;
  }
  else
  {
    host_pointer = input_pointer;
  }

  // Check to see if the host memory is pinned.
  if ( ! is_host_memory_pinned)
  {
    precheck
    cudaErr(hipHostRegister(host_pointer, sizeof(float)*input_memory_allocated, hipHostRegisterDefault));
    postcheck

    precheck
    cudaErr(hipHostGetDevicePointer( &pinnedPtr, host_pointer, 0));
    postcheck

    is_host_memory_pinned = true;
  }
  is_in_memory_host_pointer = true;
  
}

void FourierTransformer::CopyHostToDevice()
{
 
	MyFFTDebugAssertTrue(is_in_memory_host_pointer, "Host memory not allocated");
  MyFFTDebugAssertTrue(is_set_output_params, "Output parameters need to be set");
  MyFFTDebugAssertTrue(is_set_input_params, "Input parameters need to be set");
  MyFFTPrintWithDetails("Copying host to device");
  MyFFTPrint(std::to_string(output_memory_allocated) + " bytes of host memory to device");
	if ( ! is_in_memory_device_pointer )
	{
    // Allocate enough for the out of place buffer as well.
    MyFFTPrintWithDetails("Allocating device memory for input pointer");
    precheck
		cudaErr(hipMalloc(&device_pointer_fp32, 2*output_memory_allocated*sizeof(float)));
    postcheck

		device_pointer_fp32_complex = (float2 *)device_pointer_fp32;

    buffer_fp32 = &device_pointer_fp32[output_memory_allocated];
    buffer_fp32_complex = (float2 *)buffer_fp32;
 
		is_in_memory_device_pointer = true;
	}


  precheck
  // This will be too big on the output memory if padded
  cudaErr(hipMemcpyAsync(device_pointer_fp32, pinnedPtr, output_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
  bool should_block_until_complete = true;
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));
  postcheck


}

void FourierTransformer::CopyDeviceToHost(bool is_in_buffer, bool free_gpu_memory, bool unpin_host_memory)
{
 
	MyFFTDebugAssertTrue(is_in_memory_device_pointer, "GPU memory not allocated");

  float* copy_pointer;
  if (is_in_buffer) copy_pointer = buffer_fp32;
  else copy_pointer = device_pointer_fp32;

  precheck
	cudaErr(hipMemcpyAsync(pinnedPtr, copy_pointer, output_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
  postcheck

  // Just set true her for now
  bool should_block_until_complete = true;
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));
  	// TODO add asserts etc.
	if (free_gpu_memory) { Deallocate();}
  if (unpin_host_memory) { UnPinHostMemory();}


}


void FourierTransformer::Deallocate()
{

	if (is_in_memory_device_pointer) 
	{
    precheck
		cudaErr(hipFree(device_pointer_fp32));
    postcheck
		is_in_memory_device_pointer = false;
	}	
}

void FourierTransformer::UnPinHostMemory()
{
  if (is_host_memory_pinned)
	{
    precheck
		cudaErr(hipHostUnregister(host_pointer));
    postcheck
		is_host_memory_pinned = false;
	} 
}


void FourierTransformer::SimpleFFT_NoPadding()
{

  using namespace cufftdx;
	int threadsPerBlock = dims_in.x; // FIXME make sure its a multiple of 32
	int gridDims = 1;

	using FFT = decltype( FFT_64_fp32() + Type<fft_type::r2c>() + Direction<fft_direction::forward>() );
  using complex_type = typename FFT::value_type;
  using scalar_type    = typename complex_type::value_type;

  precheck
	SimpleFFT_NoPaddingKernel<FFT, complex_type, scalar_type>
  << <gridDims,  FFT::block_dim, FFT::shared_memory_size, hipStreamPerThread>> > ( (scalar_type*)device_pointer_fp32, (complex_type*)buffer_fp32_complex, dims_in, dims_out);
  postcheck


}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void SimpleFFT_NoPaddingKernel(ScalarType* real_input, ComplexType* complex_output, short4 dims_in, short4 dims_out)
{

	// Initialize the shared memory, assuming everying matches the input data X size in
	// Check that setting hipFuncSetSharedMemConfig  to 8byte makes any diff for complex reads
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  extern __shared__  complex_type shared_mem[];
  complex_type thread_data[FFT::storage_size];

  io<FFT>::load_r2c(real_input, thread_data, 0);
  FFT().execute(thread_data, shared_mem);
  io<FFT>::store_r2c(thread_data, complex_output,  0);


}

void FourierTransformer::FFT_R2C_Transposed()
{

  // TODO add asserts

  // TODO padding or maybe that is a separate funcitno.
	// For the twiddle factors ahead of the P size ffts
	float twiddle_in = -2*PIf/dims_out.x;
	int   Q = dims_out.x / dims_in.x; // FIXME assuming for now this is already divisible


	dim3 threadsPerBlock = dim3(dims_in.x/elements_per_thread_real, 1, 1); // FIXME make sure its a multiple of 32
	dim3 gridDims = dim3(1,dims_in.y, 1); // TODO allow 3d and also confirm this isn't used in any artifacts leftover 

  using FFT = decltype( FFT_64_fp32() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() );
	using complex_type = typename FFT::value_type;
	using scalar_type    = typename complex_type::value_type;

  int shared_mem = dims_in.x*sizeof(scalar_type) + FFT::shared_memory_size;
  precheck
  block_fft_kernel_R2C_Transposed<FFT,complex_type,scalar_type><< <gridDims,  threadsPerBlock, shared_mem, hipStreamPerThread>> >
  ( (scalar_type *) device_pointer_fp32,  (complex_type*) buffer_fp32_complex, dims_in, dims_out,twiddle_in,Q);
  postcheck

}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C_Transposed(ScalarType* input_values, ComplexType* output_values, short4 dims_in, short4 dims_out, float twiddle_in, int Q)
{

  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_input[];
  complex_type* shared_mem = (complex_type*)&shared_input[dims_in.x];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data ... these really could be short ints, but I don't know how that will perform. TODO benchmark
  // It is also questionable whether storing these vs, recalculating makes more sense.
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  // multiply Q*dims_out.w because x maps to y in the output transposed FFT
  io<FFT>::load_r2c_shared(&input_values[blockIdx.y*dims_in.w*2], shared_input, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);

	// In the first FFT the modifying twiddle factor is 1 so the data are real
	FFT().execute(thread_data, shared_mem);

  io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, dims_out.y);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	    io<FFT>::copy_from_shared(shared_input, thread_data, input_MAP);

    printf("I SHOULD NOT BE HERA\n");
		// cufftDX expects packed real data for a real xform, but we modify with a complex twiddle factor.
		// to get around this, split the complex fft into the sum of the real and imaginary parts
		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		  // increment the output map. 
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem);

    io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, dims_out.y);

	}


} // end of block_fft_kernel_R2C_Transposed

void FourierTransformer::FFT_C2C_WithPadding(bool forward_transform)
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform


	float twiddle_in;
	int Q;
	int shared_mem;
	dim3 threadsPerBlock;
	dim3 gridDims;

  using FFT = decltype( FFT_64_fp32() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() );
	using complex_type = typename FFT::value_type;
	using scalar_type    = typename complex_type::value_type;


  // For the twiddle factors ahead of the P size ffts
  twiddle_in = -2*PIf/dims_out.y;
  Q = dims_out.y / dims_in.y; // FIXME assuming for now this is already divisible
    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.

  threadsPerBlock = dim3(dims_in.y/elements_per_thread_complex, 1, 1); // FIXME make sure its a multiple of 32
  gridDims = dim3(1,dims_out.w,1);

  // Aggregate the transformed frequency data in shared memory so that we can write to global coalesced.
  shared_mem = dims_out.y*sizeof(complex_type) + dims_in.y*sizeof(complex_type) + FFT::shared_memory_size;

  // When it is the output dims being smaller, may need a logical or different method
  precheck
  block_fft_kernel_C2C_WithPadding<FFT,complex_type><< <gridDims,  threadsPerBlock, shared_mem, hipStreamPerThread>> >
  ( (complex_type*)buffer_fp32_complex,  (complex_type*)device_pointer_fp32_complex, dims_in, dims_out,twiddle_in,Q);
  postcheck

}

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding(ComplexType* input_values, ComplexType* output_values, short4 dims_in, short4 dims_out, float twiddle_in, int Q)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
	complex_type* shared_output = (complex_type*)&shared_input_complex[dims_in.y]; // storage for the coalesced output data. This may grow too large, 
	complex_type* shared_mem = (complex_type*)&shared_output[dims_out.y];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load_shared(&input_values[blockIdx.y*dims_out.y], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q, 1);


	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem);

	io<FFT>::store(thread_data,shared_output,output_MAP,1);


    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);

		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		    // increment the output map. Note this only works for the leading non-zero case
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem);

		io<FFT>::store(thread_data,shared_output,output_MAP,1);

	}

  // TODO confirm this is needed
	__syncthreads();

	// Now that the memory output can be coalesced send to global
  // FIXME is this actually coalced?
	// int this_idx;
	for (int sub_fft = 0; sub_fft < Q; sub_fft++)
	{
    io<FFT>::store_coalesced(shared_output, &output_values[blockIdx.y * dims_out.y], sub_fft, dims_in.y);
		// for (int i = 0; i < FFT::elements_per_thread; i++)
		// {
		// 	this_idx = input_MAP[i] + dims_in.x*sub_fft;
		// 	if (this_idx < dims_out.w)
		// 	{
		// 		output_values[blockIdx.y * dims_out.w + this_idx] = shared_output[this_idx];
		// 	}
		// }
	}


} // end of block_fft_kernel_C2C_WithPadding

} // namespace fast_FFT



