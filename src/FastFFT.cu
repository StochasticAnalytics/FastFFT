#include "hip/hip_runtime.h"
// Insert some license stuff here

// #include <string>
#include <iostream>
#include <string>
#include <stdio.h>
#include <cufftdx.hpp>


#include "FastFFT.cuh"



namespace FastFFT {




FourierTransformer::FourierTransformer(DataType wanted_calc_data_type) 
{

  calc_data_type = wanted_calc_data_type;
  // Plan to allow fp16 and bf16
  MyFFTDebugAssertTrue(calc_data_type == DataType::fp32, "Only F32 is supported at the moment");
  SetDefaults();
}

FourierTransformer::~FourierTransformer() 
{
  Deallocate();
  UnPinHostMemory();
}


void FourierTransformer::SetDefaults()
{
  DataType input_data_type = fp32;
  DataType output_data_type = fp32;

  // booleans to track state, could be bit fields but that seem opaque to me.
  is_in_memory_host_pointer = false;
  is_in_memory_device_pointer = false;

  is_fftw_padded_input = false;
  is_fftw_padded_output = false;
  is_fftw_padded_buffer = false;

  is_set_input_params = false;
  is_set_output_params = false;

  is_host_memory_pinned = false;

  is_size_validated = false;

}

void FourierTransformer::SetInputDimensionsAndType(size_t input_logical_x_dimension, 
                                                   size_t input_logical_y_dimension, 
                                                   size_t input_logical_z_dimension, 
                                                   bool is_padded_input, 
                                                   bool is_host_memory_pinned, 
                                                   DataType input_data_type,
                                                   OriginType input_origin_type)
{

  MyFFTDebugAssertTrue(input_logical_x_dimension > 0, "Input logical x dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_y_dimension > 0, "Input logical y dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_z_dimension > 0, "Input logical z dimension must be > 0");
  MyFFTDebugAssertTrue(is_padded_input, "The input memory must be fftw padded");

 
  dims_in = make_short4(input_logical_x_dimension, input_logical_y_dimension, input_logical_z_dimension,0);

  input_memory_allocated = ReturnPaddedMemorySize(dims_in);
  input_number_of_real_values = dims_in.x*dims_in.y*dims_in.z;

  this->input_origin_type = input_origin_type;
  is_set_input_params = true;
}

void FourierTransformer::SetOutputDimensionsAndType(size_t output_logical_x_dimension, 
                                                    size_t output_logical_y_dimension, 
                                                    size_t output_logical_z_dimension, 
                                                    bool is_padded_output, 
                                                    DataType output_data_type,
                                                    OriginType output_origin_type)
{
  MyFFTDebugAssertTrue(output_logical_x_dimension > 0, "output logical x dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_y_dimension > 0, "output logical y dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_z_dimension > 0, "output logical z dimension must be > 0");
  MyFFTDebugAssertTrue(is_padded_output, "The output memory must be fftw padded");


  dims_out = make_short4(output_logical_x_dimension, output_logical_y_dimension, output_logical_z_dimension,0);

  output_memory_allocated = ReturnPaddedMemorySize(dims_out);
  output_number_of_real_values = dims_out.x*dims_out.y*dims_out.z;

  this->output_origin_type = output_origin_type;
  is_set_output_params = true;
}

void FourierTransformer::CheckDimensions()
{
  // This should be run inside any public method call to ensure things ar properly setup.
  if ( ! is_size_validated )
  {
    MyFFTDebugAssertTrue(is_set_input_params, "Input parameters not set");
    MyFFTDebugAssertTrue(is_set_output_params, "Output parameters not set");
  
    if (dims_out.x > dims_in.x || dims_out.y > dims_in.y || dims_out.z > dims_in.z)
    {
      // For now we must pad in all dimensions, this is not needed and should be lifted. FIXME
      MyFFTRunTimeAssertTrue(dims_out.x >= dims_in.x, "If padding, all dimensions must be >=, x out < x in");
      MyFFTRunTimeAssertTrue(dims_out.y >= dims_in.y, "If padding, all dimensions must be >=, y out < y in");
      MyFFTRunTimeAssertTrue(dims_out.z >= dims_in.z, "If padding, all dimensions must be >=, z out < z in");
  
      size_change_type = increase;
    }
  
    MyFFTRunTimeAssertFalse(dims_out.x < dims_in.x || dims_out.y < dims_in.y || dims_out.z < dims_in.z, "Trimming (subset of output points) is yet to be implemented.");
  
    if (dims_out.x == dims_in.x && dims_out.y == dims_in.y && dims_out.z == dims_in.z)
    {
      size_change_type = none;
    }

    // check for dimensionality
    if (dims_in.z == 1 && dims_out.z == 1)
    {
      if (dims_in.y == 1 && dims_out.y == 1) 
      {
        transform_dimension = 1;
      }
      else 
      {
        transform_dimension = 2;
      }
    }
    else 
    {
      transform_dimension = 3;
    }
  
    is_size_validated = true;
  }

}


void FourierTransformer::SetInputPointer(float* input_pointer, bool is_input_on_device) 
{ 
  MyFFTDebugAssertTrue(calc_data_type == DataType::fp32, "Only F32 is supported at the moment");
  MyFFTDebugAssertTrue(is_set_input_params, "Input parameters not set");

  if ( is_input_on_device) 
  {
    // We'll need a check on compute type, and a conversion if needed prior to this.
    device_pointer_fp32 = input_pointer;
  }
  else
  {
    host_pointer = input_pointer;
  }

  // Check to see if the host memory is pinned.
  if ( ! is_host_memory_pinned)
  {
    precheck
    cudaErr(hipHostRegister(host_pointer, sizeof(float)*input_memory_allocated, hipHostRegisterDefault));
    postcheck

    precheck
    cudaErr(hipHostGetDevicePointer( &pinnedPtr, host_pointer, 0));
    postcheck

    is_host_memory_pinned = true;
  }
  is_in_memory_host_pointer = true;
  
}




void FourierTransformer::CopyHostToDevice()
{
 
	MyFFTDebugAssertTrue(is_in_memory_host_pointer, "Host memory not allocated");
  MyFFTDebugAssertTrue(is_set_output_params, "Output parameters need to be set");
  MyFFTDebugAssertTrue(is_set_input_params, "Input parameters need to be set");
  // MyFFTPrintWithDetails("Copying host to device");
  // MyFFTPrint(std::to_string(output_memory_allocated) + " bytes of host memory to device");
	if ( ! is_in_memory_device_pointer )
	{
    // Allocate enough for the out of place buffer as well.
    // MyFFTPrintWithDetails("Allocating device memory for input pointer");
    precheck
		cudaErr(hipMalloc(&device_pointer_fp32, 2*output_memory_allocated*sizeof(float)));
    postcheck

		device_pointer_fp32_complex = (float2 *)device_pointer_fp32;

    buffer_fp32 = &device_pointer_fp32[output_memory_allocated];
    buffer_fp32_complex = (float2 *)buffer_fp32;
 
		is_in_memory_device_pointer = true;
	}


  precheck
  // This will be too big on the output memory if padded
  cudaErr(hipMemcpyAsync(device_pointer_fp32, pinnedPtr, input_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
  bool should_block_until_complete = true;
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));
  postcheck

  is_in_buffer_memory = false;


}

void FourierTransformer::CopyDeviceToHost( bool free_gpu_memory, bool unpin_host_memory)
{
 
	MyFFTDebugAssertTrue(is_in_memory_device_pointer, "GPU memory not allocated");

  float* copy_pointer;
  if (is_in_buffer_memory) copy_pointer = buffer_fp32;
  else copy_pointer = device_pointer_fp32;

  precheck
	cudaErr(hipMemcpyAsync(pinnedPtr, copy_pointer, input_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
  postcheck

  // Just set true her for now
  bool should_block_until_complete = true;
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));
  	// TODO add asserts etc.
	if (free_gpu_memory) { Deallocate();}
  if (unpin_host_memory) { UnPinHostMemory();}


}


void FourierTransformer::CopyDeviceToHost(float* output_pointer, bool free_gpu_memory, bool unpin_host_memory)
{
 
	MyFFTDebugAssertTrue(is_in_memory_device_pointer, "GPU memory not allocated");

  float* copy_pointer;
  if (is_in_buffer_memory) copy_pointer = buffer_fp32;
  else copy_pointer = device_pointer_fp32;

  // Assuming the output is not pinned, TODO change to optionally maintain as host_input as well.
  float* tmpPinnedPtr;
  precheck
  cudaErr(hipHostRegister(output_pointer, sizeof(float)*output_memory_allocated, hipHostRegisterDefault));
  postcheck
  
  precheck
  cudaErr(hipHostGetDevicePointer( &tmpPinnedPtr, output_pointer, 0));
  postcheck
  
  precheck
	cudaErr(hipMemcpyAsync(tmpPinnedPtr, copy_pointer, output_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
  postcheck

  // Just set true her for now
  bool should_block_until_complete = true;
  if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));

  precheck
  cudaErr(hipHostUnregister(tmpPinnedPtr));
  postcheck

	if (free_gpu_memory) { Deallocate();}
  if (unpin_host_memory) { UnPinHostMemory();}

}


void FourierTransformer::Deallocate()
{

	if (is_in_memory_device_pointer) 
	{
    precheck
		cudaErr(hipFree(device_pointer_fp32));
    postcheck
		is_in_memory_device_pointer = false;
	}	
}

void FourierTransformer::UnPinHostMemory()
{
  if (is_host_memory_pinned)
	{
    precheck
		cudaErr(hipHostUnregister(host_pointer));
    postcheck
		is_host_memory_pinned = false;
	} 
}


void FourierTransformer::FwdFFT(bool swap_real_space_quadrants)
{
  CheckDimensions();

  switch (transform_dimension)
  {
    case 1: {
      FFT_R2C_decomposed();
      break;
    }
    case 2: {
      switch (size_change_type)
      {
        case none: {
          FFT_R2C_Transposed();
          FFT_C2C(true);
          break;
        }
        case increase: {
          FFT_R2C_WithPadding_Transposed();
          FFT_C2C_WithPadding(swap_real_space_quadrants);
          break;
        }
        case decrease: {
          MyFFTRunTimeAssertTrue(false, "Size reduction not yet supported");
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}


void FourierTransformer::InvFFT()
{
  CheckDimensions();

  switch (transform_dimension)
  {
    case 1: {
      // In progress
      break;
    }
    case 2: {
      switch (size_change_type)
      {
        case none: {
          FFT_C2C(false);
          FFT_C2R_Transposed();
          break;
        }
        case increase: {
          FFT_C2C(false);
          FFT_C2R_Transposed();
          break;
        }
        case decrease: {
          // not defined;
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}

void FourierTransformer::CrossCorrelate(float2* image_to_search, bool swap_real_space_quadrants)
{
  CheckDimensions();
  // Checks on input ft type

  switch (transform_dimension)
  {
    case 1: {
      // In progress
      break;
    }
    case 2: {
      switch (size_change_type)
      {
        case none: {
          // not defined
        }
        case increase: {
    
          FFT_R2C_WithPadding_Transposed();
    
          FFT_C2C_WithPadding_ConjMul_C2C(image_to_search, swap_real_space_quadrants);
    
          FFT_C2R_Transposed();
          break;
        }
        case decrease: {
          // not defined;
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}
  
////////////////////////////////////////////////////
/// END PUBLIC METHODS
////////////////////////////////////////////////////


template<class FFT> void FourierTransformer::FFT_R2C_decomposed_t()
{

  // Note unlike block transforms, we get the transform size here, it must be before LaunchParams. TODO add logical checks
  GetTransformSize_thread(dims_in.x, size_of<FFT>::value);
  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_decomposed);

  using complex_type = typename FFT::value_type;
  using scalar_type = typename complex_type::value_type;


  int shared_mem = LP.mem_offsets.shared_output * sizeof(complex_type);

  // cudaErr(hipSetDevice(0));
  //  cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_R2C_Transposed<FFT,complex_type,scalar_type>,hipFuncCachePreferShared ));
  //  hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_R2C_Transposed<FFT,complex_type,scalar_type>, hipSharedMemBankSizeEightByte );

  precheck
  thread_fft_kernel_R2C_decomposed<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
  ((scalar_type*) device_pointer_fp32,  (complex_type*) buffer_fp32_complex, LP.mem_offsets, LP.twiddle_in, LP.Q);
  postcheck

  is_in_buffer_memory = true;
}

void FourierTransformer::FFT_R2C_decomposed()
{

  int device, arch;
  GetCudaDeviceArch( device, arch );
  // Get transform size is called in FFT_R2C_decomposed_t() for thread based xform.

  switch (arch)
  {
    case 700: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<700>());  FFT_R2C_decomposed_t<FFT>(); break;}
    case 750: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<750>());  FFT_R2C_decomposed_t<FFT>(); break;}
    case 800: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<800>());  FFT_R2C_decomposed_t<FFT>(); break;}
  }
  
}

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_R2C_decomposed(const ScalarType*  __restrict__ input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{

  using complex_type = ComplexType;
  using scalar_type  = ScalarType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
  complex_type twiddle;
 
  // Each thread reads in the input data at stride = Q
  unsigned int index  = threadIdx.x + blockIdx.y*mem_offsets.pixel_pitch_input;
  for (unsigned int i = 0; i < size_of<FFT>::value; i++) 
  {
    thread_data[i].x = input_values[index];
    thread_data[i].y = scalar_type(0);
    index += Q;
  }
  
  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
	FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  
  // Unroll the first loop and initialize the shared mem. 
  index = threadIdx.x * size_of<FFT>::value;
  twiddle_in *= threadIdx.x; // twiddle factor arg now just needs to multiplied by K = (index + i)
  for (unsigned int i = 0; i < size_of<FFT>::value; i++)
  {
    __sincosf( twiddle_in * (index + i) ,&twiddle.y,&twiddle.x);
    twiddle *= thread_data[i];
    if (index + i < mem_offsets.pixel_pitch_output) shared_mem[index +  i] = twiddle;
  }
  
  for (unsigned int sub_fft = 1; sub_fft < Q; sub_fft++)
  {
    // wrap around, 0 --> 1, Q-1 --> 0 etc.
    index = ((threadIdx.x + sub_fft) % Q) * size_of<FFT>::value;
    for (unsigned int i = 0; i < FFT::elements_per_thread; i++)
    {
      __sincosf( twiddle_in * (index + i) ,&twiddle.y,&twiddle.x);
      twiddle *= thread_data[i];
      if (index + i < mem_offsets.pixel_pitch_output) shared_mem[index +  i] += twiddle;
    }
  }
  

  // Each thread reads in the input data at stride = mem_offsets.Q
  index  = threadIdx.x;
  for (unsigned int i = 0; i < size_of<FFT>::value/2; i++) 
  {
    output_values[index + blockIdx.y*mem_offsets.pixel_pitch_input] = shared_mem[index];
    index += Q;
  }
  if (index < mem_offsets.pixel_pitch_output)
  {
    output_values[index + blockIdx.y*mem_offsets.pixel_pitch_input] = shared_mem[index];
  }

 
} // end of block_fft_kernel_R2C

template<class FFT>
void FourierTransformer::FFT_R2C_Transposed_t()
{

  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_transposed);

  using complex_type = typename FFT::value_type;
  using scalar_type = typename complex_type::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);

  int shared_mem = FFT::shared_memory_size;

  // cudaErr(hipSetDevice(0));
   cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_R2C_Transposed<FFT,complex_type,scalar_type>,hipFuncCachePreferShared ));
   hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_R2C_Transposed<FFT,complex_type,scalar_type>, hipSharedMemBankSizeEightByte );

  precheck
  block_fft_kernel_R2C_Transposed<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
  ((scalar_type*) device_pointer_fp32,  (complex_type*) buffer_fp32_complex, LP.mem_offsets, workspace);
  postcheck

  is_in_buffer_memory = true;
}

void FourierTransformer::FFT_R2C_Transposed()
{

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_in.x);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
      break; }

    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
    //   }
    // break; }    

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
      break; } 


    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_Transposed_t<FFT>(); break;}
      }
      break; } 
  }
}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C_Transposed(const ScalarType* __restrict__ input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];


	// Memory used by FFT
  complex_type thread_data[FFT::storage_size];


  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  // multiply Q*dims_out.w because x maps to y in the output transposed FFT
  io<FFT>::load_r2c(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data);

	// In the first FFT the modifying twiddle factor is 1 so the data are real
	FFT().execute(thread_data, shared_mem, workspace);
  
  io<FFT>::store_r2c_transposed(thread_data, output_values, mem_offsets.pixel_pitch_output);

 
} // end of block_fft_kernel_R2C_Transposed

template<class FFT>
void FourierTransformer::FFT_R2C_WithPadding_Transposed_t()
{

  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_transposed);

  using complex_type = typename FFT::value_type;
  using scalar_type = typename complex_type::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);
  cudaErr(error_code);
  int shared_mem = LP.mem_offsets.shared_input*sizeof(scalar_type) + FFT::shared_memory_size;

  precheck
  block_fft_kernel_R2C_WithPadding_Transposed<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
  ( (scalar_type*) device_pointer_fp32,  (complex_type*) buffer_fp32_complex, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
  postcheck

  is_in_buffer_memory = true;
}

void FourierTransformer::FFT_R2C_WithPadding_Transposed()
{

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_in.x);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
      break; }

    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
    //   }
    // break; }   

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
      break; } 

    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_Transposed_t<FFT>(); break;}
      }
      break; } 
  }
}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C_WithPadding_Transposed(const ScalarType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_input[];
  complex_type* shared_mem = (complex_type*)&shared_input[mem_offsets.shared_input];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data ... these really could be short ints, but I don't know how that will perform. TODO benchmark
  // It is also questionable whether storing these vs, recalculating makes more sense.
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  // multiply Q*dims_out.w because x maps to y in the output transposed FFT
  io<FFT>::load_r2c_shared(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], shared_input, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);

	// We unroll the first and last loops.
  // In the first FFT the modifying twiddle factor is 1 so the data are real
	FFT().execute(thread_data, shared_mem, workspace);  
  io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, mem_offsets.pixel_pitch_output);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q-1; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input, thread_data, input_MAP);
		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		  // increment the output mapping. 
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);
    io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, mem_offsets.pixel_pitch_output);
	}

  // For the last fragment we need to also do a bounds check.
  io<FFT>::copy_from_shared(shared_input, thread_data, input_MAP);
  for (int i = 0; i < FFT::elements_per_thread; i++)
  {
    // Pre shift with twiddle
    __sincosf(twiddle_factor_args[i]*(Q-1),&twiddle.y,&twiddle.x);
    thread_data[i] *= twiddle;
    // increment the output mapping. 
    output_MAP[i]++;
  }

  FFT().execute(thread_data, shared_mem, workspace);
  io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, mem_offsets.pixel_pitch_output, mem_offsets.shared_output);
	


} // end of block_fft_kernel_R2C_WithPadding_Transposed

template<class FFT, class invFFT> 
void FourierTransformer::FFT_C2C_WithPadding_ConjMul_C2C_t(float2* image_to_search, bool swap_real_space_quadrants)
{
  
  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, xcorr_transposed);

  // Assuming invFFT is >= in size to FFT and both are C2C
	using complex_type = typename FFT::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace_fwd = make_workspace<FFT>(error_code); // presumably larger of the two
  cudaErr(error_code);
  error_code = hipSuccess;
  auto workspace_inv = make_workspace<invFFT>(error_code); // presumably larger of the two
  cudaErr(error_code);
  int shared_mem = invFFT::shared_memory_size;

  // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>,hipFuncCachePreferShared ));
  // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>, hipSharedMemBankSizeEightByte );


  if (swap_real_space_quadrants)
  {
    precheck
    block_fft_kernel_C2C_WithPadding_ConjMul_C2C_SwapRealSpaceQuadrants<FFT,invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*) image_to_search, (complex_type*)  buffer_fp32_complex,  (complex_type*) device_pointer_fp32_complex, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace_fwd, workspace_inv);
    postcheck
  }
  else
  {
    precheck
    block_fft_kernel_C2C_WithPadding_ConjMul_C2C<FFT, invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    (  (complex_type*) image_to_search, (complex_type*)  buffer_fp32_complex,  (complex_type*) device_pointer_fp32_complex, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace_fwd, workspace_inv);
    postcheck
  }

  is_in_buffer_memory = false;

}

void FourierTransformer::FFT_C2C_WithPadding_ConjMul_C2C(float2* image_to_search, bool swap_real_space_quadrants)
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_out.y);

  switch (transform_size)
  {
    case 64: {
      using FFT_noarch    = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());       
        switch (arch)
        {
          case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
          case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
          case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        }
        break;
      }

    case 128: {
      using FFT_noarch    = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }
 
    case 256: {
      using FFT_noarch    = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }

    case 512: {
      using FFT_noarch    = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }

    // case 768: {
    //   using FFT_noarch    = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
    //   using invFFT_noarch = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());      
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}        
    //   }
    // break; } 

    case 1024: {
      using FFT_noarch    = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}        
      }
    break; } 

    // case 1536: {
    //   using FFT_noarch    = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
    //   using invFFT_noarch = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     // case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}        
    //   }
    // break; }    

    case 2048: {
      using FFT_noarch    = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }

    case 4096: {
      using FFT_noarch    = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }
     
  } // end of switch on dims_in.y

  // Relies on the debug assert above

}

template<class FFT, class invFFT, class ComplexType>
__launch_bounds__(invFFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding_ConjMul_C2C(const ComplexType* __restrict__ image_to_search, const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace_fwd, typename invFFT::workspace_type workspace_inv)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	__shared__ complex_type shared_mem[invFFT::shared_memory_size/sizeof(complex_type)]; // Storage for the input data that is re-used each blcok

  complex_type thread_data[FFT::storage_size];


  io<FFT>::load(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, mem_offsets.shared_input);

	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace_fwd);


  io<invFFT>::load_shared_and_conj_multiply(&image_to_search[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data);

  invFFT().execute(thread_data, shared_mem, workspace_inv);

  io<invFFT>::store(thread_data, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output]);



} // end of block_fft_kernel_C2C_WithPadding_ConjMul_C2C

template<class FFT, class invFFT, class ComplexType>
__launch_bounds__(invFFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding_ConjMul_C2C_SwapRealSpaceQuadrants(const ComplexType* __restrict__ image_to_search, const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace_fwd, typename invFFT::workspace_type workspace_inv)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	__shared__ complex_type shared_mem[invFFT::shared_memory_size/sizeof(complex_type)]; // Storage for the input data that is re-used each blcok

  complex_type thread_data[FFT::storage_size];


  io<FFT>::load(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, mem_offsets.shared_input);

	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace_fwd);

  // Swap real space quadrants using a phase shift by N/2 pixels 
  const unsigned int  stride = io<invFFT>::stride_size();
  int logical_y;
  for (unsigned int i = 0; i < FFT::elements_per_thread; i++) 
  {
    logical_y = threadIdx.x+ i*stride;
    if ( logical_y >= mem_offsets.pixel_pitch_output/2) logical_y -= mem_offsets.pixel_pitch_output;
    if ( (int(blockIdx.y) + logical_y) % 2 != 0) thread_data[i] *= -1.f; // FIXME TYPE
  }

  io<invFFT>::load_shared_and_conj_multiply(&image_to_search[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data);

  invFFT().execute(thread_data, shared_mem, workspace_inv);

  io<invFFT>::store(thread_data, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output]);



} // 

template <class FFT>
void FourierTransformer::FFT_C2C_WithPadding_t(bool swap_real_space_quadrants)
{

  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_padded);


	using complex_type = typename FFT::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);

  // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>,hipFuncCachePreferShared ));
  // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>, hipSharedMemBankSizeEightByte );

  int shared_mem;
  // Aggregate the transformed frequency data in shared memory so that we can write to global coalesced.
  shared_mem = LP.mem_offsets.shared_output*sizeof(complex_type) + LP.mem_offsets.shared_input*sizeof(complex_type) + FFT::shared_memory_size;
  // When it is the output dims being smaller, may need a logical or different method
  if (swap_real_space_quadrants)
  {
    precheck
    block_fft_kernel_C2C_WithPadding_SwapRealSpaceQuadrants<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)  buffer_fp32_complex,  (complex_type*) device_pointer_fp32_complex, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
    postcheck
  }
  else
  {
    precheck
    block_fft_kernel_C2C_WithPadding<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)  buffer_fp32_complex,  (complex_type*) device_pointer_fp32_complex, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
    postcheck
  }


  is_in_buffer_memory = false;


}
void FourierTransformer::FFT_C2C_WithPadding(bool swap_real_space_quadrants)
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_in.y);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }
 
    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //   }
    // break; }   

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; } 

    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }    
  }


}

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding(const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
	complex_type* shared_output = (complex_type*)&shared_input_complex[mem_offsets.shared_input]; // storage for the coalesced output data. This may grow too large, 
	complex_type* shared_mem = (complex_type*)&shared_output[mem_offsets.shared_output];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load_shared(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);


	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	// 
  io<FFT>::store(thread_data,shared_output,output_MAP);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);

		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		    // increment the output map. Note this only works for the leading non-zero case
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);

    io<FFT>::store(thread_data,shared_output,output_MAP);


	}

  // TODO confirm this is needed
	__syncthreads();

	// Now that the memory output can be coalesced send to global
  // FIXME is this actually coalced?
	for (int sub_fft = 0; sub_fft < Q; sub_fft++)
	{
    io<FFT>::store_coalesced(shared_output, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output], sub_fft*mem_offsets.shared_input);
	}


} // end of block_fft_kernel_C2C_WithPadding

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding_SwapRealSpaceQuadrants(const ComplexType*  __restrict__  input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
	complex_type* shared_output = (complex_type*)&shared_input_complex[mem_offsets.shared_input]; // storage for the coalesced output data. This may grow too large, 
	complex_type* shared_mem = (complex_type*)&shared_output[mem_offsets.shared_output];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];


  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load_shared(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);


	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	// 
  io<FFT>::store_and_swap_quadrants(thread_data,shared_output,output_MAP,mem_offsets.pixel_pitch_input/2);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);

		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		    // increment the output map. Note this only works for the leading non-zero case
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);
    io<FFT>::store_and_swap_quadrants(thread_data,shared_output,output_MAP,mem_offsets.pixel_pitch_input/2);


	}

  // TODO confirm this is needed
	__syncthreads();

	// Now that the memory output can be coalesced send to global
  // FIXME is this actually coalced?
	for (int sub_fft = 0; sub_fft < Q; sub_fft++)
	{
    io<FFT>::store_coalesced(shared_output, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output], sub_fft*mem_offsets.shared_input);
	}


} // end of block_fft_kernel_C2C_WithPadding_SwapRealSpaceQuadrants

template<class FFT_nodir>
void FourierTransformer::FFT_C2C_t( bool do_forward_transform )
{
  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c);

  
  if (do_forward_transform)
  {   
    using FFT = decltype( FFT_nodir() + Direction<fft_direction::forward>() );
    using complex_type = typename FFT::value_type;
    using scalar_type    = typename complex_type::value_type;
    hipError_t error_code = hipSuccess;
    auto workspace = make_workspace<FFT>(error_code);
    int shared_mem = FFT::shared_memory_size;
    precheck
    block_fft_kernel_C2C<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)buffer_fp32_complex,  (complex_type*)device_pointer_fp32_complex, LP.mem_offsets, workspace);
    postcheck

    is_in_buffer_memory = false;
  }
  else
  {
    using FFT = decltype( FFT_nodir() + Direction<fft_direction::inverse>() );
    using complex_type = typename FFT::value_type;
    using scalar_type    = typename complex_type::value_type;
    hipError_t error_code = hipSuccess;
    auto workspace = make_workspace<FFT>(error_code);
    int shared_mem = FFT::shared_memory_size;
    precheck
    block_fft_kernel_C2C<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)device_pointer_fp32_complex,  (complex_type*)buffer_fp32_complex, LP.mem_offsets, workspace);
    postcheck

    is_in_buffer_memory = true;
  }
  

}

void FourierTransformer::FFT_C2C( bool do_forward_transform )
{

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_out.y);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }
     
    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }   
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //   }
    // break; }  


    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }       

    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }      
  }


}

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C(const ComplexType*  __restrict__  input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_mem[]; // Storage for the input data that is re-used each blcok


	// Memory used by FFT
  complex_type thread_data[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input],  thread_data);


	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	io<FFT>::store(thread_data ,&output_values[blockIdx.y*mem_offsets.pixel_pitch_output]);


} // end of block_fft_kernel_C2C

template <class FFT>
void FourierTransformer::FFT_C2R_Transposed_t()
{
  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_transposed);

	using complex_type = typename FFT::value_type;
	using scalar_type    = typename complex_type::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);
  cudaErr(error_code);

  if (is_in_buffer_memory)
  {
    precheck
    block_fft_kernel_C2R_Transformed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, FFT::shared_memory_size, hipStreamPerThread>> >
    ( (complex_type*)buffer_fp32_complex, (scalar_type*)device_pointer_fp32, LP.mem_offsets, workspace);
    postcheck
    is_in_buffer_memory = false;
  }
  else
  {
    precheck
    block_fft_kernel_C2R_Transformed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, FFT::shared_memory_size, hipStreamPerThread>> >
    ( (complex_type*)device_pointer_fp32, (scalar_type*)buffer_fp32_complex, LP.mem_offsets, workspace);
    postcheck
    is_in_buffer_memory = true;
  }


}

void FourierTransformer::FFT_C2R_Transposed()
{


  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_out.x);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }
       
    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //   }
    // break; }  

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; } 
      
    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }      
     
  }


}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2R_Transformed(const ComplexType* __restrict__  input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

	using complex_type = ComplexType;
	using scalar_type  = ScalarType;

	extern __shared__  complex_type shared_mem[];


  complex_type thread_data[FFT::storage_size];

  io<FFT>::load_c2r_transposed(&input_values[blockIdx.y], thread_data, mem_offsets.pixel_pitch_input);

  // For loop zero the twiddles don't need to be computed
  FFT().execute(thread_data, shared_mem, workspace);

  io<FFT>::store_c2r(thread_data, &output_values[blockIdx.y*mem_offsets.pixel_pitch_output]);

} // end of block_fft_kernel_C2R_Transposed


void FourierTransformer::ClipIntoTopLeft()
{
  // TODO add some checks and logic.

  // Assuming we are calling this from R2C_Transposed and that the launch bounds are not set.
  dim3 threadsPerBlock;
  dim3 gridDims;

  threadsPerBlock = dim3(512,1,1);
  gridDims = dim3( (dims_out.x + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

  const short4 area_to_clip_from = make_short4(dims_in.x, dims_in.y, dims_in.w*2, dims_out.w*2);

  precheck
  clip_into_top_left_kernel<float, float><< < gridDims, threadsPerBlock, 0, hipStreamPerThread >> >
  (device_pointer_fp32, device_pointer_fp32, area_to_clip_from);
  postcheck
}
 
template<typename InputType, typename OutputType>
__global__ void clip_into_top_left_kernel(InputType*  input_values, OutputType* output_values, short4 dims )
{

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x > dims.w) return; // Out of bounds. 

  // dims.w is the pitch of the output array
  if (blockIdx.y > dims.y) { output_values[blockIdx.y * dims.w + x] = OutputType(0); return; }

  if (threadIdx.x > dims.x) { output_values[blockIdx.y * dims.w + x] = OutputType(0); return; }
  else 
  {
    // dims.z is the pitch of the output array
    output_values[blockIdx.y * dims.w + x] = input_values[blockIdx.y * dims.z + x];
    return;
  }
} // end of clip_into_top_left_kernel


void FourierTransformer::ClipIntoReal(int wanted_coordinate_of_box_center_x, int wanted_coordinate_of_box_center_y, int wanted_coordinate_of_box_center_z)
{
  // TODO add some checks and logic.

  // Assuming we are calling this from R2C_Transposed and that the launch bounds are not set.
  dim3 threadsPerBlock;
  dim3 gridDims;
  int3 wanted_center = make_int3(wanted_coordinate_of_box_center_x, wanted_coordinate_of_box_center_y, wanted_coordinate_of_box_center_z);
  threadsPerBlock = dim3(32,32,1);
  gridDims = dim3( (dims_out.x + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (dims_out.y + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                   1);

  const short4 area_to_clip_from = make_short4(dims_in.x, dims_in.y, dims_in.w*2, dims_out.w*2);
  float wanted_padding_value = 0.f;
  
  precheck
  clip_into_real_kernel<float, float><< < gridDims, threadsPerBlock, 0, hipStreamPerThread >> >
  (device_pointer_fp32, device_pointer_fp32, dims_in, dims_out,wanted_center, wanted_padding_value);
  postcheck

}
// Modified from GpuImage::ClipIntoRealKernel
template<typename InputType, typename OutputType>
__global__ void clip_into_real_kernel(InputType* real_values_gpu,
                                      OutputType* other_image_real_values_gpu,
                                      short4 dims, 
                                      short4 other_dims,
                                      int3 wanted_coordinate_of_box_center, 
                                      OutputType wanted_padding_value)
{
  int3 other_coord = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                               blockIdx.y*blockDim.y + threadIdx.y,
                               blockIdx.z);

  int3 coord = make_int3(0, 0, 0); 

  if (other_coord.x < other_dims.x &&
      other_coord.y < other_dims.y &&
      other_coord.z < other_dims.z)
  {

    coord.z = dims.z/2 + wanted_coordinate_of_box_center.z + 
    other_coord.z - other_dims.z/2;

    coord.y = dims.y/2 + wanted_coordinate_of_box_center.y + 
    other_coord.y - other_dims.y/2;

    coord.x = dims.x + wanted_coordinate_of_box_center.x + 
    other_coord.x - other_dims.x;

    if (coord.z < 0 || coord.z >= dims.z || 
        coord.y < 0 || coord.y >= dims.y ||
        coord.x < 0 || coord.x >= dims.x)
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = wanted_padding_value;
    }
    else
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = 
      real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(coord, dims) ];
    }

  } // end of bounds check

} // end of ClipIntoRealKernel

} // namespace fast_FFT



