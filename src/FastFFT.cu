#include "hip/hip_runtime.h"
// Insert some license stuff here

// #include <string>
#include <iostream>
#include <string>
#include <stdio.h>
#include <cufftdx.hpp>


#include "FastFFT.cuh"



namespace FastFFT {


template <class ComputeType, class InputType, class OutputType>
FourierTransformer<ComputeType, InputType, OutputType>::FourierTransformer() 
{
  SetDefaults();

  // This assumption precludes the use of a packed _half2 that is really RRII layout for two arrays of __half.
  // TODO could is_real_valued_input be constexpr?
  if constexpr(std::is_same< InputType, __half2>::value || std::is_same< InputType,float2>::value)
  {
    is_real_valued_input = false;
  }
  else
  {
    is_real_valued_input = true;
  }
  
}

template <class ComputeType, class InputType, class OutputType>
FourierTransformer<ComputeType, InputType, OutputType>::~FourierTransformer() 
{
  Deallocate();
  UnPinHostMemory();
  SetDefaults();
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetDefaults()
{

  // booleans to track state, could be bit fields but that seem opaque to me.
  is_in_memory_host_pointer = false;
  is_in_memory_device_pointer = false;

  is_fftw_padded_input = false;
  is_fftw_padded_output = false;

  is_set_input_params = false;
  is_set_output_params = false;
  is_size_validated = false;
  is_set_input_pointer = false;

  is_host_memory_pinned = false;


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::Deallocate()
{
	if (is_in_memory_device_pointer) 
	{
    precheck
		cudaErr(hipFree(d_ptr.position_space));
    postcheck
		is_in_memory_device_pointer = false;
	}	
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::UnPinHostMemory()
{
  if (is_host_memory_pinned)
	{
    precheck
		cudaErr(hipHostUnregister(host_pointer));
    postcheck
		is_host_memory_pinned = false;
	} 
}


template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetInputDimensionsAndType(size_t input_logical_x_dimension, 
                                                   size_t input_logical_y_dimension, 
                                                   size_t input_logical_z_dimension, 
                                                   bool is_padded_input, 
                                                   bool is_host_memory_pinned, 
                                                   OriginType input_origin_type)
{

  MyFFTDebugAssertTrue(input_logical_x_dimension > 0, "Input logical x dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_y_dimension > 0, "Input logical y dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_z_dimension > 0, "Input logical z dimension must be > 0");

  dims_in = make_short4(input_logical_x_dimension, input_logical_y_dimension, input_logical_z_dimension,0);
  is_fftw_padded_input = is_padded_input; // Note: Must be set before ReturnPaddedMemorySize
  MyFFTRunTimeAssertTrue(is_fftw_padded_input, "Support for input arrays that are not FFTW padded needs to be implemented."); // FIXME

  input_memory_allocated = ReturnPaddedMemorySize(dims_in);
  input_number_non_padding_values = dims_in.x*dims_in.y*dims_in.z;

  this->input_origin_type = input_origin_type;
  is_set_input_params = true;
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetOutputDimensionsAndType(size_t output_logical_x_dimension, 
                                                    size_t output_logical_y_dimension, 
                                                    size_t output_logical_z_dimension, 
                                                    bool is_padded_output, 
                                                    OriginType output_origin_type)
{
  MyFFTDebugAssertTrue(is_set_input_params, "Please set the input paramters first.")
  MyFFTDebugAssertTrue(output_logical_x_dimension > 0, "output logical x dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_y_dimension > 0, "output logical y dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_z_dimension > 0, "output logical z dimension must be > 0");
  MyFFTDebugAssertTrue(is_fftw_padded_input == is_padded_output, "If the input data are FFTW padded, so must the output.");

  dims_out = make_short4(output_logical_x_dimension, output_logical_y_dimension, output_logical_z_dimension,0);

  output_memory_allocated = ReturnPaddedMemorySize(dims_out);
  output_number_non_padding_values = dims_out.x*dims_out.y*dims_out.z;

  this->output_origin_type = output_origin_type;
  is_set_output_params = true;
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CheckDimensions()
{
  // This should be run inside any public method call to ensure things ar properly setup.

  // TODO - runtime asserts would be better as these are breaking errors that are under user control.
  // check to see if there is any measurable penalty for this.
  if ( ! is_size_validated )
  {
    MyFFTDebugAssertTrue(is_set_input_params, "Input parameters not set");
    MyFFTDebugAssertTrue(is_set_output_params, "Output parameters not set");
    MyFFTDebugAssertTrue(is_set_input_pointer, "The input data pointer is not set");

  
    if (dims_out.x > dims_in.x || dims_out.y > dims_in.y || dims_out.z > dims_in.z)
    {
      // For now we must pad in all dimensions, this is not needed and should be lifted. FIXME
      MyFFTDebugAssertTrue(dims_out.x >= dims_in.x, "If padding, all dimensions must be >=, x out < x in");
      MyFFTDebugAssertTrue(dims_out.y >= dims_in.y, "If padding, all dimensions must be >=, y out < y in");
      MyFFTDebugAssertTrue(dims_out.z >= dims_in.z, "If padding, all dimensions must be >=, z out < z in");
  
      size_change_type = increase;
    }
    else if (dims_out.x < dims_in.x || dims_out.y < dims_in.y || dims_out.z < dims_in.z)
    {
      MyFFTRunTimeAssertTrue( false, "Trimming (subset of output points) is yet to be implemented.");
      size_change_type = decrease;
    }
    else if (dims_out.x == dims_in.x && dims_out.y == dims_in.y && dims_out.z == dims_in.z)
    {
      size_change_type = none;
    }
    else
    {
      // TODO: if this is relaxed, the dimensionality check below will be invalid.
      MyFFTRunTimeAssertTrue( false, "Currently all dimensions must either increase, decrease or stay the same.");
    }

    // check for dimensionality
    // Note: this is predicated on the else clause ensuring all dimensions behave the same way w.r.t. size change.
    if (dims_in.z == 1 && dims_out.z == 1)
    {
      if (dims_in.y == 1 && dims_out.y == 1) 
      {
        transform_dimension = 1;
      }
      else 
      {
        transform_dimension = 2;
      }
    }
    else 
    {
      transform_dimension = 3;
    }
  
    is_size_validated = true;
  }

}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetInputPointer(InputType* input_pointer, bool is_input_on_device) 
{ 
  MyFFTDebugAssertTrue(is_set_input_params, "Input parameters not set");

  if ( is_input_on_device) 
  {
    // We'll need a check on compute type, and a conversion if needed prior to this.
    d_ptr.position_space = input_pointer;
  }
  else
  {
    host_pointer = input_pointer;
  }

  // Check to see if the host memory is pinned.
  if ( ! is_host_memory_pinned)
  {
    precheck
    cudaErr(hipHostRegister((void *)host_pointer, sizeof(InputType)*input_memory_allocated, hipHostRegisterDefault));
    postcheck

    precheck
    cudaErr(hipHostGetDevicePointer( &pinnedPtr, host_pointer, 0));
    postcheck

    is_host_memory_pinned = true;
  }
  is_in_memory_host_pointer = true;
  
  is_set_input_pointer = true;
}



template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CopyHostToDevice()
{
 
  CheckDimensions();
	MyFFTDebugAssertTrue(is_in_memory_host_pointer, "Host memory not allocated");

  // MyFFTPrintWithDetails("Copying host to device");
  // MyFFTPrint(std::to_string(output_memory_allocated) + " bytes of host memory to device");
	if ( ! is_in_memory_device_pointer )
	{

    // Allocate enough for the out of place buffer as well.
    // MyFFTPrintWithDetails("Allocating device memory for input pointer");
    std::cout << "Compute mem allocated , size of " << compute_memory_allocated << " " << sizeof(ComputeType) << std::endl;
    precheck
		cudaErr(hipMalloc(&d_ptr.position_space, compute_memory_allocated * sizeof(ComputeType)));
    postcheck

    size_t buffer_address;
    if (is_real_valued_input) buffer_address = compute_memory_allocated/2 ;
    else buffer_address = compute_memory_allocated/4; 

    if constexpr(std::is_same< decltype(d_ptr.momentum_space), __half2>::value )
    {
      d_ptr.momentum_space = (__half2 *)d_ptr.position_space;
      d_ptr.position_space_buffer = &d_ptr.position_space[buffer_address];
      d_ptr.momentum_space_buffer = (__half2 *)d_ptr.position_space_buffer;
    }
    else
    {
      d_ptr.momentum_space = (float2 *)d_ptr.position_space;
      d_ptr.position_space_buffer = &d_ptr.position_space[buffer_address]; // compute 
      d_ptr.momentum_space_buffer = (float2 *)d_ptr.position_space_buffer;
    }


 
		is_in_memory_device_pointer = true;
	}


  precheck
  cudaErr(hipMemcpyAsync(d_ptr.position_space, pinnedPtr, input_memory_allocated * sizeof(InputType),hipMemcpyDeviceToHost,hipStreamPerThread));
  postcheck
  is_in_buffer_memory = false;

  // TODO r/n assuming InputType is _half, _half2, float, or _float2 (real, complex, real, complex) need to handle other types and convert
  bool should_block_until_complete = true;
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));

}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CopyDeviceToHost( bool free_gpu_memory, bool unpin_host_memory)
{
 
  CheckDimensions();
	MyFFTDebugAssertTrue(is_in_memory_device_pointer, "GPU memory not allocated");

  ComputeType* copy_pointer;
  if (is_in_buffer_memory) copy_pointer = d_ptr.position_space_buffer;
  else copy_pointer = d_ptr.position_space;

  // FIXME this is assuming the input type matches the compute type.
  precheck
	cudaErr(hipMemcpyAsync(pinnedPtr, copy_pointer, input_memory_allocated*sizeof(InputType),hipMemcpyDeviceToHost,hipStreamPerThread));
  postcheck

  // Just set true her for now
  bool should_block_until_complete = true;
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));
  	// TODO add asserts etc.
	if (free_gpu_memory) { Deallocate();}
  if (unpin_host_memory) { UnPinHostMemory();}

}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CopyDeviceToHost(OutputType* output_pointer, bool free_gpu_memory, bool unpin_host_memory)
{
 
  CheckDimensions();
	MyFFTDebugAssertTrue(is_in_memory_device_pointer, "GPU memory not allocated");
  // Assuming the output is not pinned, TODO change to optionally maintain as host_input as well.
  OutputType* tmpPinnedPtr;
  precheck
  // FIXME this is assuming output type is the same as compute type.
  cudaErr(hipHostRegister(output_pointer, sizeof(OutputType)*output_memory_allocated, hipHostRegisterDefault));
  postcheck
  
  precheck
  cudaErr(hipHostGetDevicePointer( &tmpPinnedPtr, output_pointer, 0));
  postcheck
  if (is_in_buffer_memory)
  {
    precheck
    cudaErr(hipMemcpyAsync(tmpPinnedPtr, d_ptr.position_space_buffer, output_memory_allocated*sizeof(OutputType),hipMemcpyDeviceToHost,hipStreamPerThread));
    postcheck
  }
  else
  {
    precheck
    cudaErr(hipMemcpyAsync(tmpPinnedPtr, d_ptr.position_space, output_memory_allocated*sizeof(OutputType),hipMemcpyDeviceToHost,hipStreamPerThread));
    postcheck
  }


  // Just set true her for now
  bool should_block_until_complete = true;
  if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));

  precheck
  cudaErr(hipHostUnregister(tmpPinnedPtr));
  postcheck

	if (free_gpu_memory) { Deallocate();}
  if (unpin_host_memory) { UnPinHostMemory();}

}



template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FwdFFT(bool swap_real_space_quadrants, bool transpose_output)
{
  CheckDimensions();

  switch (transform_dimension)
  {
    case 1: {
      if (is_real_valued_input) FFT_R2C_decomposed(transpose_output);
      else FFT_C2C_decomposed(true);
      
      break;
    }
    case 2: {
      switch (size_change_type)
      {
        case none: {
          // FFT_R2C(transpose_output);
          // FFT_C2C(true);
          FFT_R2C_decomposed(true);
          FFT_C2C_decomposed(true);
          break;
        }
        case increase: {
          FFT_R2C_WithPadding(transpose_output);
          FFT_C2C_WithPadding(swap_real_space_quadrants);
          break;
        }
        case decrease: {
          MyFFTRunTimeAssertTrue(false, "Size reduction not yet supported");
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::InvFFT(bool transpose_output)
{
  CheckDimensions();

  switch (transform_dimension)
  {
    case 1: {
      if (is_real_valued_input) FFT_C2R_decomposed(transpose_output);
      else FFT_C2C_decomposed(false);
      break;
    }
    case 2: {
      switch (size_change_type)
      {
        case none: {
          // FFT_C2C(false);
          // FFT_C2R_Transposed();
          FFT_C2C_decomposed(false);
          FFT_C2R_decomposed(true);
          break;
        }
        case increase: {
          FFT_C2C(false);
          FFT_C2R_Transposed();
          break;
        }
        case decrease: {
          // not defined;
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CrossCorrelate(float2* image_to_search, bool swap_real_space_quadrants)
{
  CheckDimensions();
  // Checks on input ft type

  switch (transform_dimension)
  {
    case 1: {
      // In progress
      break;
    }
    case 2: {
      switch (size_change_type)
      {
        case none: {
          // not defined
        }
        case increase: {
    
          FFT_R2C_WithPadding();
    
          FFT_C2C_WithPadding_ConjMul_C2C(image_to_search, swap_real_space_quadrants);
    
          FFT_C2R_Transposed();
          break;
        }
        case decrease: {
          // not defined;
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CrossCorrelate(__half2* image_to_search, bool swap_real_space_quadrants)
{
  CheckDimensions();
  // Checks on input ft type

  switch (transform_dimension)
  {
    case 1: {
      // In progress
      break;
    }
    case 2: {
      switch (size_change_type)
      {
        case none: {
          // not defined
        }
        case increase: {
    
          FFT_R2C_WithPadding();
    
          FFT_C2C_WithPadding_ConjMul_C2C(image_to_search, swap_real_space_quadrants);
    
          FFT_C2R_Transposed();
          break;
        }
        case decrease: {
          // not defined;
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}
////////////////////////////////////////////////////
/// END PUBLIC METHODS
////////////////////////////////////////////////////
template <class ComputeType, class InputType, class OutputType>
template<class FFT> 
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_R2C_decomposed_t(bool transpose_output)
{

  // Note unlike block transforms, we get the transform size here, it must be before LaunchParams. TODO add logical checks
  GetTransformSize_thread(dims_in.x, size_of<FFT>::value);

  LaunchParams LP;
  if (transpose_output)
  {
    LP = SetLaunchParameters(elements_per_thread_complex, r2c_decomposed_transposed);
  }
  else
  {
    LP = SetLaunchParameters(elements_per_thread_complex, r2c_decomposed);
  }



  using complex_type = typename FFT::value_type;
  using scalar_type = typename complex_type::value_type;


  int shared_mem = LP.mem_offsets.shared_output * sizeof(complex_type);

  if (transpose_output)
  {
    precheck
    thread_fft_kernel_R2C_decomposed_transposed<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ((scalar_type*) d_ptr.position_space,  (complex_type*) d_ptr.momentum_space_buffer, LP.mem_offsets, LP.twiddle_in, LP.Q);
    postcheck
  }
  else
  {
    precheck
    thread_fft_kernel_R2C_decomposed<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ((scalar_type*) d_ptr.position_space,  (complex_type*) d_ptr.momentum_space_buffer, LP.mem_offsets, LP.twiddle_in, LP.Q);
    postcheck
  }

  is_in_buffer_memory = true;
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_R2C_decomposed(bool transpose_output)
{

  int device, arch;
  GetCudaDeviceArch( device, arch );
  // Get transform size is called in FFT_R2C_decomposed_t() for thread based xform.
  switch (arch)
  {
    case 700: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<700>());  FFT_R2C_decomposed_t<FFT>(transpose_output); break;}
    case 750: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<750>());  FFT_R2C_decomposed_t<FFT>(transpose_output); break;}
    case 800: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<800>());  FFT_R2C_decomposed_t<FFT>(transpose_output); break;}
  }
  
}

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_R2C_decomposed(const ScalarType*  __restrict__ input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{

  using complex_type = ComplexType;
  using scalar_type  = ScalarType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 
  io_thread<FFT>::load_r2c(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, Q);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
	FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  
  io_thread<FFT>::remap_decomposed_segments(thread_data, shared_mem, twiddle_in, Q, mem_offsets.pixel_pitch_output);


  io_thread<FFT>::store_r2c(shared_mem, &output_values[blockIdx.y*mem_offsets.pixel_pitch_output], Q, mem_offsets.shared_output);

 
} // end of block_fft_kernel_R2C

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_R2C_decomposed_transposed(const ScalarType*  __restrict__ input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{

  using complex_type = ComplexType;
  using scalar_type  = ScalarType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 
  io_thread<FFT>::load_r2c(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, Q);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
	FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments(thread_data, shared_mem, twiddle_in, Q, mem_offsets.shared_output);


  io_thread<FFT>::store_r2c_transposed(shared_mem, &output_values[blockIdx.y], Q, mem_offsets.pixel_pitch_output, mem_offsets.shared_output);

 
} // end of block_fft_kernel_R2C_transposed

template <class ComputeType, class InputType, class OutputType>
template<class FFT>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_R2C_t(bool transpose_output)
{

  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_transposed);

  using complex_type = typename FFT::value_type;
  using scalar_type = typename complex_type::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);

  int shared_mem = FFT::shared_memory_size;

  // cudaErr(hipSetDevice(0));
  //  cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_R2C<FFT,complex_type,scalar_type>,hipFuncCachePreferShared ));
  //  hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_R2C<FFT,complex_type,scalar_type>, hipSharedMemBankSizeEightByte );

  precheck
  block_fft_kernel_R2C<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
  ((scalar_type*) d_ptr.position_space,  (complex_type*) d_ptr.momentum_space_buffer, LP.mem_offsets, workspace);
  postcheck

  is_in_buffer_memory = true;
}
template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_R2C(bool transpose_output)
{

  MyFFTRunTimeAssertTrue(transpose_output, "In FFT_R2C, non-transposed output is not yet supported.");

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_in.x);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
      break; }

    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
    //   }
    // break; }    

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
      break; } 


    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_t<FFT>(transpose_output); break;}
      }
      break; } 
  }
}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C(const ScalarType* __restrict__ input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];


	// Memory used by FFT
  complex_type thread_data[FFT::storage_size];


  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  // multiply Q*dims_out.w because x maps to y in the output transposed FFT
  io<FFT>::load_r2c(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data);

	// In the first FFT the modifying twiddle factor is 1 so the data are real
	FFT().execute(thread_data, shared_mem, workspace);
  
  io<FFT>::store_r2c_transposed(thread_data, output_values, mem_offsets.pixel_pitch_output);

 
} // end of block_fft_kernel_R2C

template <class ComputeType, class InputType, class OutputType>
template<class FFT>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_R2C_WithPadding_t(bool transpose_output)
{

  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_transposed);

  using complex_type = typename FFT::value_type;
  using scalar_type = typename complex_type::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);
  cudaErr(error_code);
  int shared_mem = LP.mem_offsets.shared_input*sizeof(scalar_type) + FFT::shared_memory_size;

  precheck
  block_fft_kernel_R2C_WithPadding<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
  ( (scalar_type*) d_ptr.position_space,  (complex_type*) d_ptr.momentum_space_buffer, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
  postcheck

  is_in_buffer_memory = true;
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_R2C_WithPadding(bool transpose_output)
{

  MyFFTRunTimeAssertTrue(transpose_output, "FFT_R2C_WithPadding: transpose_output must be true");

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_in.x);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
      break; }

    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
    //   }
    // break; }   

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
      break; } 

    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_R2C_WithPadding_t<FFT>(transpose_output); break;}
      }
      break; } 
  }
}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C_WithPadding(const ScalarType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_input[];
  complex_type* shared_mem = (complex_type*)&shared_input[mem_offsets.shared_input];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data ... these really could be short ints, but I don't know how that will perform. TODO benchmark
  // It is also questionable whether storing these vs, recalculating makes more sense.
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  // multiply Q*dims_out.w because x maps to y in the output transposed FFT
  io<FFT>::load_r2c_shared(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], shared_input, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);

	// We unroll the first and last loops.
  // In the first FFT the modifying twiddle factor is 1 so the data are real
	FFT().execute(thread_data, shared_mem, workspace);  
  io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, mem_offsets.pixel_pitch_output);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q-1; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input, thread_data, input_MAP);
		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		  // increment the output mapping. 
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);
    io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, mem_offsets.pixel_pitch_output);
	}

  // For the last fragment we need to also do a bounds check.
  io<FFT>::copy_from_shared(shared_input, thread_data, input_MAP);
  for (int i = 0; i < FFT::elements_per_thread; i++)
  {
    // Pre shift with twiddle
    __sincosf(twiddle_factor_args[i]*(Q-1),&twiddle.y,&twiddle.x);
    thread_data[i] *= twiddle;
    // increment the output mapping. 
    output_MAP[i]++;
  }

  FFT().execute(thread_data, shared_mem, workspace);
  io<FFT>::store_r2c_transposed(thread_data, output_values, output_MAP, mem_offsets.pixel_pitch_output, mem_offsets.shared_output);
	


} // end of block_fft_kernel_R2C_WithPadding

template <class ComputeType, class InputType, class OutputType>
template<class FFT, class invFFT> 
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C_WithPadding_ConjMul_C2C_t(float2* image_to_search, bool swap_real_space_quadrants)
{
  
  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, xcorr_transposed);

  // Assuming invFFT is >= in size to FFT and both are C2C
	using complex_type = typename FFT::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace_fwd = make_workspace<FFT>(error_code); // presumably larger of the two
  cudaErr(error_code);
  error_code = hipSuccess;
  auto workspace_inv = make_workspace<invFFT>(error_code); // presumably larger of the two
  cudaErr(error_code);
  int shared_mem = invFFT::shared_memory_size;

  // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>,hipFuncCachePreferShared ));
  // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>, hipSharedMemBankSizeEightByte );


  if (swap_real_space_quadrants)
  {
    precheck
    block_fft_kernel_C2C_WithPadding_ConjMul_C2C_SwapRealSpaceQuadrants<FFT,invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*) image_to_search, (complex_type*)  d_ptr.momentum_space_buffer,  (complex_type*) d_ptr.momentum_space, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace_fwd, workspace_inv);
    postcheck
  }
  else
  {
    precheck
    block_fft_kernel_C2C_WithPadding_ConjMul_C2C<FFT, invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    (  (complex_type*) image_to_search, (complex_type*)  d_ptr.momentum_space_buffer,  (complex_type*) d_ptr.momentum_space, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace_fwd, workspace_inv);
    postcheck
  }

  is_in_buffer_memory = false;

}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C_WithPadding_ConjMul_C2C(float2* image_to_search, bool swap_real_space_quadrants)
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_out.y);

  switch (transform_size)
  {
    case 64: {
      using FFT_noarch    = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());       
        switch (arch)
        {
          case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
          case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
          case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        }
        break;
      }

    case 128: {
      using FFT_noarch    = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }
 
    case 256: {
      using FFT_noarch    = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }

    case 512: {
      using FFT_noarch    = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }

    // case 768: {
    //   using FFT_noarch    = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
    //   using invFFT_noarch = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());      
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}        
    //   }
    // break; } 

    case 1024: {
      using FFT_noarch    = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}        
      }
    break; } 

    // case 1536: {
    //   using FFT_noarch    = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
    //   using invFFT_noarch = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     // case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}        
    //   }
    // break; }    

    case 2048: {
      using FFT_noarch    = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }

    case 4096: {
      using FFT_noarch    = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>());
      using invFFT_noarch = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>());
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_noarch() + SM<700>()); using invFFT = decltype(invFFT_noarch() + SM<700>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_noarch() + SM<750>()); using invFFT = decltype(invFFT_noarch() + SM<750>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_noarch() + SM<800>()); using invFFT = decltype(invFFT_noarch() + SM<800>()); FFT_C2C_WithPadding_ConjMul_C2C_t<FFT, invFFT>(image_to_search,swap_real_space_quadrants); break;}
      }
      break;
    }
     
  } // end of switch on dims_in.y

  // Relies on the debug assert above

}

template<class FFT, class invFFT, class ComplexType>
__launch_bounds__(invFFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding_ConjMul_C2C(const ComplexType* __restrict__ image_to_search, const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace_fwd, typename invFFT::workspace_type workspace_inv)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	__shared__ complex_type shared_mem[invFFT::shared_memory_size/sizeof(complex_type)]; // Storage for the input data that is re-used each blcok

  complex_type thread_data[FFT::storage_size];


  io<FFT>::load(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, mem_offsets.shared_input);

	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace_fwd);


  io<invFFT>::load_shared_and_conj_multiply(&image_to_search[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data);

  invFFT().execute(thread_data, shared_mem, workspace_inv);

  io<invFFT>::store(thread_data, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output]);



} // end of block_fft_kernel_C2C_WithPadding_ConjMul_C2C

template<class FFT, class invFFT, class ComplexType>
__launch_bounds__(invFFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding_ConjMul_C2C_SwapRealSpaceQuadrants(const ComplexType* __restrict__ image_to_search, const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace_fwd, typename invFFT::workspace_type workspace_inv)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	__shared__ complex_type shared_mem[invFFT::shared_memory_size/sizeof(complex_type)]; // Storage for the input data that is re-used each blcok

  complex_type thread_data[FFT::storage_size];


  io<FFT>::load(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, mem_offsets.shared_input);

	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace_fwd);

  // Swap real space quadrants using a phase shift by N/2 pixels 
  const unsigned int  stride = io<invFFT>::stride_size();
  int logical_y;
  for (unsigned int i = 0; i < FFT::elements_per_thread; i++) 
  {
    logical_y = threadIdx.x+ i*stride;
    if ( logical_y >= mem_offsets.pixel_pitch_output/2) logical_y -= mem_offsets.pixel_pitch_output;
    if ( (int(blockIdx.y) + logical_y) % 2 != 0) thread_data[i] *= -1.f; // FIXME TYPE
  }

  io<invFFT>::load_shared_and_conj_multiply(&image_to_search[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data);

  invFFT().execute(thread_data, shared_mem, workspace_inv);

  io<invFFT>::store(thread_data, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output]);



} // 

template <class ComputeType, class InputType, class OutputType>
template <class FFT>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C_WithPadding_t(bool swap_real_space_quadrants)
{

  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_padded);


	using complex_type = typename FFT::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);

  // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>,hipFuncCachePreferShared ));
  // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_WithPadding<FFT,complex_type>, hipSharedMemBankSizeEightByte );

  int shared_mem;
  // Aggregate the transformed frequency data in shared memory so that we can write to global coalesced.
  shared_mem = LP.mem_offsets.shared_output*sizeof(complex_type) + LP.mem_offsets.shared_input*sizeof(complex_type) + FFT::shared_memory_size;
  // When it is the output dims being smaller, may need a logical or different method
  if (swap_real_space_quadrants)
  {
    precheck
    block_fft_kernel_C2C_WithPadding_SwapRealSpaceQuadrants<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)  d_ptr.momentum_space_buffer,  (complex_type*) d_ptr.momentum_space, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
    postcheck
  }
  else
  {
    precheck
    block_fft_kernel_C2C_WithPadding<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)  d_ptr.momentum_space_buffer,  (complex_type*) d_ptr.momentum_space, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
    postcheck
  }


  is_in_buffer_memory = false;


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C_WithPadding(bool swap_real_space_quadrants)
{

	// This is the first set of 1d ffts when the input data are real valued, accessing the strided dimension. Since we need the full length, it will actually run a C2C xform

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_in.y);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }
 
    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
    //   }
    // break; }   

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; } 

    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2C_WithPadding_t<FFT>(swap_real_space_quadrants); break;}
      }
      break; }    
  }


}

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding(const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
	complex_type* shared_output = (complex_type*)&shared_input_complex[mem_offsets.shared_input]; // storage for the coalesced output data. This may grow too large, 
	complex_type* shared_mem = (complex_type*)&shared_output[mem_offsets.shared_output];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load_shared(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);


	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	// 
  io<FFT>::store(thread_data,shared_output,output_MAP);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);

		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		    // increment the output map. Note this only works for the leading non-zero case
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);

    io<FFT>::store(thread_data,shared_output,output_MAP);


	}

  // TODO confirm this is needed
	__syncthreads();

	// Now that the memory output can be coalesced send to global
  // FIXME is this actually coalced?
	for (int sub_fft = 0; sub_fft < Q; sub_fft++)
	{
    io<FFT>::store_coalesced(shared_output, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output], sub_fft*mem_offsets.shared_input);
	}


} // end of block_fft_kernel_C2C_WithPadding

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_WithPadding_SwapRealSpaceQuadrants(const ComplexType*  __restrict__  input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
	complex_type* shared_output = (complex_type*)&shared_input_complex[mem_offsets.shared_input]; // storage for the coalesced output data. This may grow too large, 
	complex_type* shared_mem = (complex_type*)&shared_output[mem_offsets.shared_output];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];


  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load_shared(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);


	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	// 
  io<FFT>::store_and_swap_quadrants(thread_data,shared_output,output_MAP,mem_offsets.pixel_pitch_input/2);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);

		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			__sincosf(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		    // increment the output map. Note this only works for the leading non-zero case
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);
    io<FFT>::store_and_swap_quadrants(thread_data,shared_output,output_MAP,mem_offsets.pixel_pitch_input/2);


	}

  // TODO confirm this is needed
	__syncthreads();

	// Now that the memory output can be coalesced send to global
  // FIXME is this actually coalced?
	for (int sub_fft = 0; sub_fft < Q; sub_fft++)
	{
    io<FFT>::store_coalesced(shared_output, &output_values[blockIdx.y * mem_offsets.pixel_pitch_output], sub_fft*mem_offsets.shared_input);
	}


} // end of block_fft_kernel_C2C_WithPadding_SwapRealSpaceQuadrants

template <class ComputeType, class InputType, class OutputType>
template<class FFT_nodir>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C_t( bool do_forward_transform )
{
  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c, do_forward_transform);

  
  if (do_forward_transform)
  {   
    using FFT = decltype( FFT_nodir() + Direction<fft_direction::forward>() );
    using complex_type = typename FFT::value_type;
    using scalar_type    = typename complex_type::value_type;
    hipError_t error_code = hipSuccess;
    auto workspace = make_workspace<FFT>(error_code);
    int shared_mem = FFT::shared_memory_size;
    precheck
    block_fft_kernel_C2C<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)d_ptr.momentum_space_buffer,  (complex_type*)d_ptr.momentum_space, LP.mem_offsets, workspace);
    postcheck

    is_in_buffer_memory = false;
  }
  else
  {
    using FFT = decltype( FFT_nodir() + Direction<fft_direction::inverse>() );
    using complex_type = typename FFT::value_type;
    using scalar_type    = typename complex_type::value_type;
    hipError_t error_code = hipSuccess;
    auto workspace = make_workspace<FFT>(error_code);
    int shared_mem = FFT::shared_memory_size;
    precheck
    block_fft_kernel_C2C<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    ( (complex_type*)d_ptr.momentum_space,  (complex_type*)d_ptr.momentum_space_buffer, LP.mem_offsets, workspace);
    postcheck

    is_in_buffer_memory = true;
  }
  

}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C( bool do_forward_transform )
{

  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_out.y);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }
     
    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }   
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
    //   }
    // break; }  


    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }       

    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_t<FFT>(do_forward_transform); break;}
      }
      break; }      
  }


}

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C(const ComplexType*  __restrict__  input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

//	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_mem[]; // Storage for the input data that is re-used each blcok


	// Memory used by FFT
  complex_type thread_data[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input],  thread_data);


	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	io<FFT>::store(thread_data ,&output_values[blockIdx.y*mem_offsets.pixel_pitch_output]);


} // end of block_fft_kernel_C2C

template <class ComputeType, class InputType, class OutputType>
template<class FFT_nodir>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C_decomposed_t( bool do_forward_transform )
{

  // Note unlike block transforms, we get the transform size here, it must be before LaunchParams. TODO add logical checks
  // Temporary fix to check for 1d, this is not to be sustained. FIXME
  if (dims_in.y == 1) GetTransformSize_thread(dims_in.x, size_of<FFT_nodir>::value);
  else GetTransformSize_thread(dims_in.y, size_of<FFT_nodir>::value); // does dims_in make sense?
  

  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_decomposed, do_forward_transform);

  using complex_type = typename FFT_nodir::value_type;
  using scalar_type  = typename complex_type::value_type;

  complex_type* input_pointer;
  complex_type* output_pointer;
  if (is_in_buffer_memory)
  {
    input_pointer  = (complex_type*)d_ptr.momentum_space_buffer;
    output_pointer = (complex_type*)d_ptr.momentum_space;
    is_in_buffer_memory = false;
  }
  else
  {
    input_pointer  = (complex_type*)d_ptr.momentum_space;
    output_pointer = (complex_type*)d_ptr.momentum_space_buffer;
    is_in_buffer_memory = true;
  }
  if (do_forward_transform)
  {
    using FFT = decltype( FFT_nodir() + Direction<fft_direction::forward>() );
    int shared_mem = LP.mem_offsets.shared_output * sizeof(complex_type);

    precheck
    thread_fft_kernel_C2C_decomposed<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    (input_pointer, output_pointer, LP.mem_offsets, LP.twiddle_in, LP.Q);
    postcheck
  }
  else
  {

    using FFT = decltype( FFT_nodir() + Direction<fft_direction::inverse>() );
    int shared_mem = LP.mem_offsets.shared_output * sizeof(complex_type);

    precheck
    thread_fft_kernel_C2C_decomposed<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
    (input_pointer, output_pointer, LP.mem_offsets, LP.twiddle_in, LP.Q);
    postcheck
  }



}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2C_decomposed( bool do_forward_transform )
{

  int device, arch;
  GetCudaDeviceArch( device, arch );

    switch (arch)
    {
      case 700: { using FFT = decltype(FFT_thread_base() + Type<fft_type::c2c>() + SM<700>());  FFT_C2C_decomposed_t<FFT>(do_forward_transform); break;}
      case 750: { using FFT = decltype(FFT_thread_base() + Type<fft_type::c2c>() + SM<750>());  FFT_C2C_decomposed_t<FFT>(do_forward_transform); break;}
      case 800: { using FFT = decltype(FFT_thread_base() + Type<fft_type::c2c>() + SM<800>());  FFT_C2C_decomposed_t<FFT>(do_forward_transform); break;}
    }

}

template<class FFT, class ComplexType>
__global__
void thread_fft_kernel_C2C_decomposed(const ComplexType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{


  using complex_type = ComplexType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 
  io_thread<FFT>::load_c2c(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, Q);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
	FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments(thread_data, shared_mem, twiddle_in, Q, mem_offsets.pixel_pitch_output);


  io_thread<FFT>::store_c2c(shared_mem, &output_values[blockIdx.y*mem_offsets.pixel_pitch_output], Q);

}

template <class ComputeType, class InputType, class OutputType>
template <class FFT>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2R_Transposed_t()
{
  LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_transposed);

	using complex_type = typename FFT::value_type;
	using scalar_type    = typename complex_type::value_type;
  hipError_t error_code = hipSuccess;
  auto workspace = make_workspace<FFT>(error_code);
  cudaErr(error_code);

  if (is_in_buffer_memory)
  {
    precheck
    block_fft_kernel_C2R_Transposed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, FFT::shared_memory_size, hipStreamPerThread>> >
    ( (complex_type*)d_ptr.momentum_space_buffer, (scalar_type*)d_ptr.position_space, LP.mem_offsets, workspace);
    postcheck
    is_in_buffer_memory = false;
  }
  else
  {
    precheck
    block_fft_kernel_C2R_Transposed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, FFT::shared_memory_size, hipStreamPerThread>> >
    ( (complex_type*)d_ptr.position_space, (scalar_type*)d_ptr.momentum_space_buffer, LP.mem_offsets, workspace);
    postcheck
    is_in_buffer_memory = true;
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2R_Transposed()
{


  int device, arch;
  GetCudaDeviceArch( device, arch );
  GetTransformSize(dims_out.x);

  switch (transform_size)
  {
    case 64: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<64>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }

    case 128: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<128>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }
       
    case 256: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<256>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; } 

    case 512: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<512>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; } 

    // case 768: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     case 750: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<768>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //   }
    // break; } 

    case 1024: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<1024>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
    break; } 

    // case 1536: {
    //   switch (arch)
    //   {
    //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
    //   }
    // break; }  

    case 2048: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 750: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<2048>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; } 
      
    case 4096: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        // case 750: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<750>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<4096>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }  

    case 8192: {
      switch (arch)
      {
        case 700: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<700>());  FFT_C2R_Transposed_t<FFT>(); break;}
        case 800: { using FFT = decltype(FFT_base()  + Size<8192>() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() + SM<800>());  FFT_C2R_Transposed_t<FFT>(); break;}
      }
      break; }      
     
  }


}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2R_Transposed(const ComplexType* __restrict__  input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

	using complex_type = ComplexType;
	using scalar_type  = ScalarType;

	extern __shared__  complex_type shared_mem[];


  complex_type thread_data[FFT::storage_size];

  io<FFT>::load_c2r_transposed(&input_values[blockIdx.y], thread_data, mem_offsets.pixel_pitch_input);

  // For loop zero the twiddles don't need to be computed
  FFT().execute(thread_data, shared_mem, workspace);

  io<FFT>::store_c2r(thread_data, &output_values[blockIdx.y*mem_offsets.pixel_pitch_output]);

} // end of block_fft_kernel_C2R_Transposed

template <class ComputeType, class InputType, class OutputType>
template <class FFT>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2R_decomposed_t(bool transpose_output)
{

  GetTransformSize_thread(dims_out.x, size_of<FFT>::value);

	using complex_type = typename FFT::value_type;
	using scalar_type    = typename complex_type::value_type;

  complex_type* tmp_input_ptr;
  scalar_type* tmp_output_ptr;

  // Note TODO this in the C2R_Transposed
  if (is_in_buffer_memory) 
  {
    tmp_input_ptr = (complex_type*)d_ptr.momentum_space_buffer;
    tmp_output_ptr = (scalar_type*)d_ptr.position_space;
    is_in_buffer_memory = false;
  }
  else
  {
    tmp_input_ptr = (complex_type*)d_ptr.momentum_space;
    tmp_output_ptr = (scalar_type*)d_ptr.position_space_buffer;
    is_in_buffer_memory = true;
  }

  if (transpose_output)
  {
    std::cout << " CONFIRM TRANSOPOSE OUTPUT C2R " << std::endl;
    LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_decomposed_transposed);
    int shared_memory = LP.mem_offsets.shared_output * sizeof(scalar_type);

    precheck
    thread_fft_kernel_C2R_decomposed_transposed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
    ( tmp_input_ptr, tmp_output_ptr, LP.mem_offsets, LP.twiddle_in, LP.Q);
    postcheck
  }
  else
  {
    LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_decomposed);
    int shared_memory = LP.mem_offsets.shared_output * sizeof(scalar_type);
    precheck
    thread_fft_kernel_C2R_decomposed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
    ( tmp_input_ptr, tmp_output_ptr, LP.mem_offsets, LP.twiddle_in, LP.Q);
    postcheck
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FFT_C2R_decomposed(bool transpose_output)
{
  int device, arch;
  GetCudaDeviceArch( device, arch );

  // Since we decompose, we need to use a c2c type.
  switch (arch)
  {
    case 700: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>() + SM<700>());  FFT_C2R_decomposed_t<FFT>(transpose_output); break;}
    case 750: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>() + SM<750>());  FFT_C2R_decomposed_t<FFT>(transpose_output); break;}
    case 800: { using FFT = decltype(FFT_thread_base() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>() + SM<800>());  FFT_C2R_decomposed_t<FFT>(transpose_output); break;}
  }

}

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_C2R_decomposed(const ComplexType*  __restrict__ input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q)
{
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_mem_C2R_decomposed[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 

  io_thread<FFT>::load_c2r(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], thread_data, Q, mem_offsets.pixel_pitch_input);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
	FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments_c2r(thread_data, shared_mem_C2R_decomposed, twiddle_in, Q);

  io_thread<FFT>::store_c2r(shared_mem_C2R_decomposed, &output_values[blockIdx.y*mem_offsets.pixel_pitch_output],Q);
}

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_C2R_decomposed_transposed(const ComplexType*  __restrict__ input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q)
{

  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_mem_transposed[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 

  io_thread<FFT>::load_c2r_transposed(&input_values[blockIdx.y], thread_data, Q, mem_offsets.pixel_pitch_input, mem_offsets.pixel_pitch_output/2);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
	// FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments_c2r(thread_data, shared_mem_transposed, twiddle_in, Q);

  io_thread<FFT>::store_c2r(shared_mem_transposed, &output_values[blockIdx.y*mem_offsets.pixel_pitch_output],Q);

}


template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::ClipIntoTopLeft()
{
  // TODO add some checks and logic.

  // Assuming we are calling this from R2C_Transposed and that the launch bounds are not set.
  dim3 threadsPerBlock;
  dim3 gridDims;

  threadsPerBlock = dim3(512,1,1);
  gridDims = dim3( (dims_out.x + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

  const short4 area_to_clip_from = make_short4(dims_in.x, dims_in.y, dims_in.w*2, dims_out.w*2);

  precheck
  clip_into_top_left_kernel<float, float><< < gridDims, threadsPerBlock, 0, hipStreamPerThread >> >
  (d_ptr.position_space, d_ptr.position_space, area_to_clip_from);
  postcheck
}
 
template<typename InputType, typename OutputType>
__global__ void clip_into_top_left_kernel(InputType*  input_values, OutputType* output_values, short4 dims )
{

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x > dims.w) return; // Out of bounds. 

  // dims.w is the pitch of the output array
  if (blockIdx.y > dims.y) { output_values[blockIdx.y * dims.w + x] = OutputType(0); return; }

  if (threadIdx.x > dims.x) { output_values[blockIdx.y * dims.w + x] = OutputType(0); return; }
  else 
  {
    // dims.z is the pitch of the output array
    output_values[blockIdx.y * dims.w + x] = input_values[blockIdx.y * dims.z + x];
    return;
  }
} // end of clip_into_top_left_kernel

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::ClipIntoReal(int wanted_coordinate_of_box_center_x, int wanted_coordinate_of_box_center_y, int wanted_coordinate_of_box_center_z)
{
  // TODO add some checks and logic.

  // Assuming we are calling this from R2C_Transposed and that the launch bounds are not set.
  dim3 threadsPerBlock;
  dim3 gridDims;
  int3 wanted_center = make_int3(wanted_coordinate_of_box_center_x, wanted_coordinate_of_box_center_y, wanted_coordinate_of_box_center_z);
  threadsPerBlock = dim3(32,32,1);
  gridDims = dim3( (dims_out.x + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (dims_out.y + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                   1);

  const short4 area_to_clip_from = make_short4(dims_in.x, dims_in.y, dims_in.w*2, dims_out.w*2);
  float wanted_padding_value = 0.f;
  
  precheck
  clip_into_real_kernel<float, float><< < gridDims, threadsPerBlock, 0, hipStreamPerThread >> >
  (d_ptr.position_space, d_ptr.position_space, dims_in, dims_out,wanted_center, wanted_padding_value);
  postcheck

}
// Modified from GpuImage::ClipIntoRealKernel
template<typename InputType, typename OutputType>
__global__ void clip_into_real_kernel(InputType* real_values_gpu,
                                      OutputType* other_image_real_values_gpu,
                                      short4 dims, 
                                      short4 other_dims,
                                      int3 wanted_coordinate_of_box_center, 
                                      OutputType wanted_padding_value)
{
  int3 other_coord = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                               blockIdx.y*blockDim.y + threadIdx.y,
                               blockIdx.z);

  int3 coord = make_int3(0, 0, 0); 

  if (other_coord.x < other_dims.x &&
      other_coord.y < other_dims.y &&
      other_coord.z < other_dims.z)
  {

    coord.z = dims.z/2 + wanted_coordinate_of_box_center.z + 
    other_coord.z - other_dims.z/2;

    coord.y = dims.y/2 + wanted_coordinate_of_box_center.y + 
    other_coord.y - other_dims.y/2;

    coord.x = dims.x + wanted_coordinate_of_box_center.x + 
    other_coord.x - other_dims.x;

    if (coord.z < 0 || coord.z >= dims.z || 
        coord.y < 0 || coord.y >= dims.y ||
        coord.x < 0 || coord.x >= dims.x)
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = wanted_padding_value;
    }
    else
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = 
      real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(coord, dims) ];
    }

  } // end of bounds check

} // end of ClipIntoRealKernel

} // namespace fast_FFT



