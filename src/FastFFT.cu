#include "hip/hip_runtime.h"
// Insert some license stuff here

// #include <string>
#include <iostream>
#include <string>
#include <stdio.h>
#include <cufftdx.hpp>


#include "FastFFT.cuh"



namespace FastFFT {


template <class ComputeType, class InputType, class OutputType>
FourierTransformer<ComputeType, InputType, OutputType>::FourierTransformer() 
{
  SetDefaults();
  GetCudaDeviceProps( device_properties );
  // exit(0);
  // This assumption precludes the use of a packed _half2 that is really RRII layout for two arrays of __half.
  // TODO could is_real_valued_input be constexpr?
  if constexpr(std::is_same< InputType, __half2>::value || std::is_same< InputType,float2>::value)
  {
    is_real_valued_input = false;
  }
  else
  {
    is_real_valued_input = true;
  }
  
}

template <class ComputeType, class InputType, class OutputType>
FourierTransformer<ComputeType, InputType, OutputType>::~FourierTransformer() 
{
  Deallocate();
  UnPinHostMemory();
  SetDefaults();
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetDefaults()
{


  // booleans to track state, could be bit fields but that seem opaque to me.
  is_in_memory_host_pointer = false; // To track allocation of host side memory
  is_in_memory_device_pointer = false; // To track allocation of device side memory.
  is_in_buffer_memory = false; // To track whether the current result is in dev_ptr.position_space or dev_ptr.position_space_buffer (momemtum space/ momentum space buffer respectively.)
  transform_stage_completed = none;
  
  is_host_memory_pinned = false; // Specified in the constructor. Assuming host memory won't be pinned for many applications.
  
  is_fftw_padded_input = false; // Padding for in place r2c transforms
  is_fftw_padded_output = false; // Currently the output state will match the input state, otherwise it is an error.
  
  is_real_valued_input = true; // This is determined by the input type. If it is a float2 or __half2, then it is assumed to be a complex valued input function.
  
  is_set_input_params = false; // Yes, yes, "are" set.
  is_set_output_params = false;
  is_size_validated = false; // Defaults to false, set after both input/output dimensions are set and checked.
  is_set_input_pointer = false; // May be on the host of the device.


  compute_memory_allocated = 0;


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::Deallocate()
{
	if (is_in_memory_device_pointer) 
	{
    precheck
		cudaErr(hipFree(d_ptr.position_space));
    postcheck
		is_in_memory_device_pointer = false;
	}	
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::UnPinHostMemory()
{
  if (is_host_memory_pinned)
	{
    precheck
		cudaErr(hipHostUnregister(host_pointer));
    postcheck
		is_host_memory_pinned = false;
	} 
}


template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetForwardFFTPlan(size_t input_logical_x_dimension,  size_t input_logical_y_dimension,  size_t input_logical_z_dimension, 
                                                                               size_t output_logical_x_dimension, size_t output_logical_y_dimension, size_t output_logical_z_dimension,
                                                                               bool is_padded_input, 
                                                                               bool is_host_memory_pinned, 
                                                                               OriginType input_origin_type)
{

  MyFFTDebugAssertTrue(input_logical_x_dimension > 0, "Input logical x dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_y_dimension > 0, "Input logical y dimension must be > 0");
  MyFFTDebugAssertTrue(input_logical_z_dimension > 0, "Input logical z dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_x_dimension > 0, "output logical x dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_y_dimension > 0, "output logical y dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_z_dimension > 0, "output logical z dimension must be > 0");

  fwd_dims_in  = make_short4(input_logical_x_dimension, input_logical_y_dimension, input_logical_z_dimension,0);
  fwd_dims_out = make_short4(output_logical_x_dimension, output_logical_y_dimension, output_logical_z_dimension,0);

  is_fftw_padded_input = is_padded_input; // Note: Must be set before ReturnPaddedMemorySize
  MyFFTRunTimeAssertTrue(is_fftw_padded_input, "Support for input arrays that are not FFTW padded needs to be implemented."); // FIXME

  // ReturnPaddedMemorySize also sets FFTW padding etc.
  input_memory_allocated = ReturnPaddedMemorySize(fwd_dims_in);
  fwd_output_memory_allocated = ReturnPaddedMemorySize(fwd_dims_out); // sets .w and also increases compute_memory_allocated if needed. 

  // The compute memory allocated is the max of all possible sizes.

  this->input_origin_type = input_origin_type;
  is_set_input_params = true;
}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetInverseFFTPlan(size_t input_logical_x_dimension,  size_t input_logical_y_dimension,  size_t input_logical_z_dimension, 
                                                                               size_t output_logical_x_dimension, size_t output_logical_y_dimension, size_t output_logical_z_dimension,
                                                                               bool is_padded_output, 
                                                                               OriginType output_origin_type)
{
  MyFFTDebugAssertTrue(is_set_input_params, "Please set the input paramters first.")
  MyFFTDebugAssertTrue(output_logical_x_dimension > 0, "output logical x dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_y_dimension > 0, "output logical y dimension must be > 0");
  MyFFTDebugAssertTrue(output_logical_z_dimension > 0, "output logical z dimension must be > 0");
  MyFFTDebugAssertTrue(is_fftw_padded_input == is_padded_output, "If the input data are FFTW padded, so must the output.");

  inv_dims_in  = make_short4(input_logical_x_dimension, input_logical_y_dimension, input_logical_z_dimension,0);
  inv_dims_out = make_short4(output_logical_x_dimension, output_logical_y_dimension, output_logical_z_dimension,0);

  ReturnPaddedMemorySize(inv_dims_in); // sets .w and also increases compute_memory_allocated if needed. 
  inv_output_memory_allocated = ReturnPaddedMemorySize(inv_dims_out);
  // The compute memory allocated is the max of all possible sizes.

  this->output_origin_type = output_origin_type;
  is_set_output_params = true;
}


template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetInputPointer(InputType* input_pointer, bool is_input_on_device) 
{ 
  MyFFTDebugAssertTrue(is_set_input_params, "Input parameters not set");

  if ( is_input_on_device) 
  {
    // We'll need a check on compute type, and a conversion if needed prior to this.
    d_ptr.position_space = input_pointer;
  }
  else
  {
    host_pointer = input_pointer;
  }

  // Check to see if the host memory is pinned.
  if ( ! is_host_memory_pinned)
  {
    precheck
    cudaErr(hipHostRegister((void *)host_pointer, sizeof(InputType)*input_memory_allocated, hipHostRegisterDefault));
    postcheck

    precheck
    cudaErr(hipHostGetDevicePointer( &pinnedPtr, host_pointer, 0));
    postcheck

    is_host_memory_pinned = true;
  }
  is_in_memory_host_pointer = true;
  
  is_set_input_pointer = true;
}



template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CopyHostToDevice()
{
 
  SetDimensions(CopyFromHost);
	MyFFTDebugAssertTrue(is_in_memory_host_pointer, "Host memory not allocated");

  // FIXME switch to stream ordered malloc
	if ( ! is_in_memory_device_pointer )
	{
    // Allocate enough for the out of place buffer as well.
    // MyFFTDebugPrintWithDetails("Allocating device memory for input pointer");
    precheck
		cudaErr(hipMalloc(&d_ptr.position_space, compute_memory_allocated * sizeof(ComputeType)));
    postcheck

    size_t buffer_address;
    if (is_real_valued_input) buffer_address = compute_memory_allocated/2 ;
    else buffer_address = compute_memory_allocated/4; 

    if constexpr(std::is_same< decltype(d_ptr.momentum_space), __half2>::value )
    {
      d_ptr.momentum_space = (__half2 *)d_ptr.position_space;
      d_ptr.position_space_buffer = &d_ptr.position_space[buffer_address];
      d_ptr.momentum_space_buffer = (__half2 *)d_ptr.position_space_buffer;
    }
    else
    {
      d_ptr.momentum_space = (float2 *)d_ptr.position_space;
      d_ptr.position_space_buffer = &d_ptr.position_space[buffer_address]; // compute 
      d_ptr.momentum_space_buffer = (float2 *)d_ptr.position_space_buffer;
    }
 
		is_in_memory_device_pointer = true;
	}

  precheck
  cudaErr(hipMemcpyAsync(d_ptr.position_space, pinnedPtr, memory_size_to_copy * sizeof(InputType),hipMemcpyDeviceToHost,hipStreamPerThread));
  postcheck
  // TODO r/n assuming InputType is _half, _half2, float, or _float2 (real, complex, real, complex) need to handle other types and convert
  bool should_block_until_complete = true; // FIXME after switching to stream ordered malloc this will not be needed.
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));

}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CopyDeviceToHost( bool free_gpu_memory, bool unpin_host_memory, int n_elements_to_copy)
{
 
  SetDimensions(CopyToHost);  
  if (n_elements_to_copy != 0) memory_size_to_copy = n_elements_to_copy;
  
  // std::cout << "N elements " << n_elements_to_copy << " memory to copy " << memory_size_to_copy <<  std::endl;
	MyFFTDebugAssertTrue(is_in_memory_device_pointer, "GPU memory not allocated");
  ComputeType* copy_pointer;
  if (is_in_buffer_memory) copy_pointer = d_ptr.position_space_buffer;
  else copy_pointer = d_ptr.position_space;

  // FIXME this is assuming the input type matches the compute type.
  precheck
	cudaErr(hipMemcpyAsync(pinnedPtr, copy_pointer, memory_size_to_copy * sizeof(InputType),hipMemcpyDeviceToHost,hipStreamPerThread));
  postcheck

  // Just set true her for now
  bool should_block_until_complete = true;
	if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));

  	// TODO add asserts etc.
	if (free_gpu_memory) { Deallocate();}

  if (unpin_host_memory) { UnPinHostMemory();}


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CopyDeviceToHost(OutputType* output_pointer, bool free_gpu_memory, bool unpin_host_memory, int n_elements_to_copy)
{
 
  SetDimensions(CopyToHost);
  if (n_elements_to_copy != 0) memory_size_to_copy = n_elements_to_copy;
	MyFFTDebugAssertTrue(is_in_memory_device_pointer, "GPU memory not allocated");
  // Assuming the output is not pinned, TODO change to optionally maintain as host_input as well.
  OutputType* tmpPinnedPtr;
  precheck
  // FIXME this is assuming output type is the same as compute type.
  cudaErr(hipHostRegister(output_pointer, sizeof(OutputType)*memory_size_to_copy, hipHostRegisterDefault));
  postcheck
  
  precheck
  cudaErr(hipHostGetDevicePointer( &tmpPinnedPtr, output_pointer, 0));
  postcheck
  if (is_in_buffer_memory)
  {
    precheck
    cudaErr(hipMemcpyAsync(tmpPinnedPtr, d_ptr.position_space_buffer, memory_size_to_copy*sizeof(OutputType),hipMemcpyDeviceToHost,hipStreamPerThread));
    postcheck
  }
  else
  {
    precheck
    cudaErr(hipMemcpyAsync(tmpPinnedPtr, d_ptr.position_space, memory_size_to_copy*sizeof(OutputType),hipMemcpyDeviceToHost,hipStreamPerThread));
    postcheck
  }


  // Just set true her for now
  bool should_block_until_complete = true;
  if (should_block_until_complete) cudaErr(hipStreamSynchronize(hipStreamPerThread));

  precheck
  cudaErr(hipHostUnregister(output_pointer));
  postcheck

	if (free_gpu_memory) { Deallocate();}
  if (unpin_host_memory) { UnPinHostMemory();}

}



template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::FwdFFT(bool swap_real_space_quadrants, bool transpose_output)
{
  SetDimensions(FwdTransform);
  static constexpr bool use_thread_method = false;
  bool do_forward_transform = true;

  // SetPrecisionAndExectutionMethod(KernelType kernel_type, bool do_forward_transform, bool use_thread_method)
  switch (transform_dimension)
  {
    case 1: {
        // FIXME there is some redundancy in specifying _decomposed and use_thread_method
        // Note: the only time the non-transposed method should be used is for 1d data.
        if constexpr (use_thread_method)
        {
          if (is_real_valued_input) SetPrecisionAndExectutionMethod(r2c_decomposed, do_forward_transform); //FFT_R2C_decomposed(transpose_output);
          else SetPrecisionAndExectutionMethod(c2c_decomposed, do_forward_transform);
          transform_stage_completed = TransformStageCompleted::fwd;

        }
        else
        {
          if (is_real_valued_input) 
          {
            switch (fwd_size_change_type)
            {
              case SizeChangeType::no_change:{ SetPrecisionAndExectutionMethod<false>(r2c_none_XY); break; }
              case SizeChangeType::decrease: { SetPrecisionAndExectutionMethod<false>(r2c_decrease); break; }
              case SizeChangeType::increase: { SetPrecisionAndExectutionMethod<false>(r2c_increase); break; }
              default: { MyFFTDebugAssertTrue(false, "Invalid size change type"); }
            }
          }
          else
          {
            switch (fwd_size_change_type)
            {
              case SizeChangeType::no_change:{ SetPrecisionAndExectutionMethod<false>(c2c_fwd_none); break; }
              case SizeChangeType::decrease: { SetPrecisionAndExectutionMethod<false>(c2c_fwd_decrease); break; }
              case SizeChangeType::increase: { SetPrecisionAndExectutionMethod<false>(c2c_fwd_increase); break; }
              default: { MyFFTDebugAssertTrue(false, "Invalid size change type"); }
            }
          }
          transform_stage_completed = TransformStageCompleted::fwd;
        }

        break;
    }
    case 2: {
      switch (fwd_size_change_type)
      {
        case no_change: {
          // FIXME there is some redundancy in specifying _decomposed and use_thread_method
          // Note: the only time the non-transposed method should be used is for 1d data.
          if (use_thread_method)
          {
            SetPrecisionAndExectutionMethod(r2c_decomposed_transposed, do_forward_transform);
            transform_stage_completed = TransformStageCompleted::fwd; // technically not complete, needed for copy on validation of partial fft.
            SetPrecisionAndExectutionMethod(c2c_decomposed, do_forward_transform);
          }
          else
          {
            SetPrecisionAndExectutionMethod(r2c_none_XY);
            transform_stage_completed = TransformStageCompleted::fwd; // technically not complete, needed for copy on validation of partial fft.
            SetPrecisionAndExectutionMethod(c2c_fwd_none);
          }
          break;
        }
        case increase: {
          SetPrecisionAndExectutionMethod(r2c_increase);
          transform_stage_completed = TransformStageCompleted::fwd; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2c_fwd_increase);   

          break;
        }
        case decrease: {

          SetPrecisionAndExectutionMethod(r2c_decrease);

          transform_stage_completed = TransformStageCompleted::fwd; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2c_fwd_decrease); 
 
          break;
        }
      }
      break; // case 2
    }
    case 3: {
      switch (fwd_size_change_type)
      {
        case no_change: {
          SetPrecisionAndExectutionMethod(r2c_none_XZ);
          transform_stage_completed = TransformStageCompleted::fwd; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2c_fwd_none_Z);
          SetPrecisionAndExectutionMethod(c2c_fwd_none);
          break;
        }
        case increase: {
          SetPrecisionAndExectutionMethod(r2c_increase);
          transform_stage_completed = TransformStageCompleted::fwd; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2c_fwd_increase_Z);   
          break;
        }
        case decrease: {
          // Not yet supported
          MyFFTRunTimeAssertTrue(false, "3D FFT fwd no change not yet supported");
          break;
        }
      } 
    }
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::InvFFT(bool transpose_output)
{
  SetDimensions(InvTransform);
  constexpr const bool use_thread_method = false;
  bool do_forward_transform = false;

  switch (transform_dimension)
  {
    case 1: {

              // FIXME there is some redundancy in specifying _decomposed and use_thread_method
        // Note: the only time the non-transposed method should be used is for 1d data.
        if constexpr (use_thread_method)
        {
          if (is_real_valued_input) SetPrecisionAndExectutionMethod(c2r_decomposed, do_forward_transform); //FFT_R2C_decomposed(transpose_output);
          else SetPrecisionAndExectutionMethod(c2c_decomposed, do_forward_transform);
          transform_stage_completed = TransformStageCompleted::inv;

        }
        else
        {
          if (is_real_valued_input) 
          {
            switch (inv_size_change_type)
            {
              case SizeChangeType::no_change:{ SetPrecisionAndExectutionMethod<false>(c2r_none_XY); break; }
              case SizeChangeType::decrease: { SetPrecisionAndExectutionMethod<false>(c2r_decrease); break; }
              case SizeChangeType::increase: { SetPrecisionAndExectutionMethod<false>(c2r_increase); break; }
              default: { MyFFTDebugAssertTrue(false, "Invalid size change type"); }
            }
          }
          else
          {
            switch (inv_size_change_type)
            {
              case SizeChangeType::no_change:{ SetPrecisionAndExectutionMethod<false>(c2c_inv_none); break; }
              case SizeChangeType::decrease: { SetPrecisionAndExectutionMethod<false>(c2c_inv_decrease); break; }
              case SizeChangeType::increase: { SetPrecisionAndExectutionMethod<false>(c2c_inv_increase); break; }
              default: { MyFFTDebugAssertTrue(false, "Invalid size change type"); }
            }
          }
          transform_stage_completed = TransformStageCompleted::inv;
        }

        break;
    }
    case 2: {
      switch (inv_size_change_type)
      {
        case no_change: {
          // FIXME there is some redundancy in specifying _decomposed and use_thread_method
          // Note: the only time the non-transposed method should be used is for 1d data.
          if (use_thread_method)
          {
            SetPrecisionAndExectutionMethod(c2c_decomposed,            do_forward_transform);
            transform_stage_completed = TransformStageCompleted::inv; // technically not complete, needed for copy on validation of partial fft.
            SetPrecisionAndExectutionMethod(c2r_decomposed_transposed, do_forward_transform);

          }
          else
          {
            SetPrecisionAndExectutionMethod(c2c_inv_none);
            transform_stage_completed = TransformStageCompleted::inv; // technically not complete, needed for copy on validation of partial fft.
            SetPrecisionAndExectutionMethod(c2r_none_XY);

          }          
          break;
        }
        case increase: {
          SetPrecisionAndExectutionMethod(c2c_inv_increase);
          transform_stage_completed = TransformStageCompleted::inv; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2r_increase); 
          
          // FFT_C2C(false);
          // FFT_C2R_Transposed();
          break;
        }
        case decrease: {
          SetPrecisionAndExectutionMethod(c2c_inv_decrease);
          transform_stage_completed = TransformStageCompleted::inv; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2r_decrease); 
          break;
        }
        default: {
          MyFFTDebugAssertTrue(false, "Invalid size change type");
        }
      } // switch on inv size change type
      break; // case 2
    }
    case 3: {
      switch (inv_size_change_type) 
      {
        case no_change: {
          SetPrecisionAndExectutionMethod(c2c_inv_none);
          transform_stage_completed = TransformStageCompleted::inv; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2c_inv_none_Z);
          SetPrecisionAndExectutionMethod(c2r_none);
          break;
        }
        case increase: {
          SetPrecisionAndExectutionMethod(r2c_increase);
          transform_stage_completed = TransformStageCompleted::fwd; // technically not complete, needed for copy on validation of partial fft.
          SetPrecisionAndExectutionMethod(c2c_fwd_increase_Z);   
          break;
        }
        case decrease: {
          // Not yet supported
          MyFFTRunTimeAssertTrue(false, "3D FFT inv no decrease not yet supported");
          break;
        }
        default: {
          MyFFTDebugAssertTrue(false, "Invalid dimension");
          break;
        }
      }
    }
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CrossCorrelate(float2* image_to_search, bool swap_real_space_quadrants)
{

  // Set the member pointer to the passed pointer
  d_ptr.image_to_search = image_to_search;
 

  switch (transform_dimension)
  {
   

    case 1: {
     

      MyFFTRunTimeAssertTrue(false, "1D FFT Cross correlation not yet supported");
      break;
    }
    case 2: {
     
      switch (fwd_size_change_type)
      {
          case no_change: {
           

            SetDimensions(FwdTransform);
            SetPrecisionAndExectutionMethod(r2c_none_XY,   true);
            switch (inv_size_change_type)
            {
              case no_change: {
                MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation no change/ nochange not yet supported");
                break;
              }
              case increase: {
                MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation no change/increase not yet supported");
                break;
              }
              case decrease: {
                           
                SetPrecisionAndExectutionMethod(xcorr_fwd_none_inv_decrease, true);
                SetPrecisionAndExectutionMethod(c2r_decrease,   false);
                break;
              }
              default: {
                MyFFTDebugAssertTrue(false, "Invalid size change type");
                break;
              }
            } // switch on inv size change type
            break;
          } // case fwd no change
          case increase: {
           
            SetDimensions(FwdTransform);
            SetPrecisionAndExectutionMethod(r2c_increase,   true);
            switch (inv_size_change_type)
            {
              case no_change: {
               

                SetPrecisionAndExectutionMethod(xcorr_fwd_increase_inv_none, true);
                SetPrecisionAndExectutionMethod(c2r_none_XY,   false);
                transform_stage_completed = TransformStageCompleted::inv;
                break;
              }
              case increase: {
                // I don't see where increase increase makes any sense
                // FIXME add a check on this in the validation step.
                MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation with fwd and inv size increase is not supported");
                break;
              }
              case decrease: {
                // with FwdTransform set, call c2c
                // Set InvTransform
                // Call new kernel that handles the conj mul inv c2c trimmed, and inv c2r in one go.
                MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation with fwd increase and inv size decrease is a work in progress");

                break;
              }
              default: {
                MyFFTRunTimeAssertTrue(false, "Invalid size change type");
              }
            } // switch on inv size change type
      
            // FFT_R2C_WithPadding();   
            // FFT_C2C_INCREASE_ConjMul_C2C(image_to_search, swap_real_space_quadrants);  
            // FFT_C2R_Transposed();
            break;
          }
          case decrease: {

            SetDimensions(FwdTransform);
            SetPrecisionAndExectutionMethod(r2c_decrease,   true);
            switch (inv_size_change_type)
            {
              case no_change: {
                SetPrecisionAndExectutionMethod(xcorr_fwd_increase_inv_none, true);
                SetPrecisionAndExectutionMethod(c2r_none_XY,   false); // TODO the output could be smaller
                transform_stage_completed = TransformStageCompleted::inv;
                break;
              }
              case increase: {

                MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation with fwd and inv size increase is not supported");
                break;
              }
              case decrease: {

                MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation with fwd decrease and inv size decrease is a work in progress");
                break;
              }
              default: {
                MyFFTRunTimeAssertTrue(false, "Invalid inv size change type");
              }
            break;
          }
          break;
        } // case decrease
        default: {
          MyFFTRunTimeAssertTrue(false, "Invalid fwd size change type");
        }
      } // switch on fwd size change type

      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::CrossCorrelate(__half2* image_to_search, bool swap_real_space_quadrants)
{

  // Set the member pointer to the passed pointer
  d_ptr.image_to_search = image_to_search;
  switch (transform_dimension)
  {
    case 1: {
      MyFFTRunTimeAssertTrue(false, "1D FFT Cross correlation not yet supported");
      break;
    }
    case 2: {
      switch (fwd_size_change_type)
      {
        case no_change: {
          MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation without size change not yet supported");
          break;
        }
        case increase: {
          SetDimensions(FwdTransform);
          SetPrecisionAndExectutionMethod(r2c_increase,   true);

          switch (inv_size_change_type)
          {
            case no_change: {
              SetPrecisionAndExectutionMethod(xcorr_fwd_increase_inv_none, true);
              SetPrecisionAndExectutionMethod(c2r_none_XY,   false); // TODO the output could be smaller
              transform_stage_completed = TransformStageCompleted::inv;

              break;
            }
            case increase: {
              // I don't see where increase increase makes any sense
              // FIXME add a check on this in the validation step.
              MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation with fwd and inv size increase is not supported");
              break;
            }
            case decrease: {
              // with FwdTransform set, call c2c
              // Set InvTransform
              // Call new kernel that handles the conj mul inv c2c trimmed, and inv c2r in one go.
              MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation with fwd increase and inv size decrease is a work in progress");

              break;
            }
          } // inv size change type
        } // case fwd_size_change = increase
        case decrease: {
          MyFFTRunTimeAssertTrue(false, "2D FFT Cross correlation without size decrease not yet supported");
          break;
        }
      } // fwd size change type
      break; // case 2
    }
    case 3: {
      // Not yet supported
      MyFFTRunTimeAssertTrue(false, "3D FFT not yet supported");
      break;
    }
  }


}
////////////////////////////////////////////////////
/// END PUBLIC METHODS
////////////////////////////////////////////////////
template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::ValidateDimensions()
{
  // TODO - runtime asserts would be better as these are breaking errors that are under user control.
  // check to see if there is any measurable penalty for this.

  MyFFTDebugAssertTrue(is_set_input_params, "Input parameters not set");
  MyFFTDebugAssertTrue(is_set_output_params, "Output parameters not set");
  MyFFTDebugAssertTrue(is_set_input_pointer, "The input data pointer is not set");

  MyFFTRunTimeAssertTrue( fwd_dims_out.x == inv_dims_in.x &&
                          fwd_dims_out.y == inv_dims_in.y &&
                          fwd_dims_out.z == inv_dims_in.z, "Error in validating the dimension: Currently all fwd out should match inv in.");

  // Validate the forward transform
  if (fwd_dims_out.x > fwd_dims_in.x || fwd_dims_out.y > fwd_dims_in.y || fwd_dims_out.z > fwd_dims_in.z)
  {
    // For now we must pad in all dimensions, this is not needed and should be lifted. FIXME
    MyFFTDebugAssertTrue(fwd_dims_out.x >= fwd_dims_in.x, "If padding, all dimensions must be >=, x out < x in");
    MyFFTDebugAssertTrue(fwd_dims_out.y >= fwd_dims_in.y, "If padding, all dimensions must be >=, y out < y in");
    MyFFTDebugAssertTrue(fwd_dims_out.z >= fwd_dims_in.z, "If padding, all dimensions must be >=, z out < z in");

    fwd_size_change_type = increase;
  }
  else if (fwd_dims_out.x < fwd_dims_in.x || fwd_dims_out.y < fwd_dims_in.y || fwd_dims_out.z < fwd_dims_in.z)
  {
    // For now we must pad in all dimensions, this is not needed and should be lifted. FIXME
    MyFFTDebugAssertTrue(fwd_dims_out.x <= fwd_dims_in.x, "If padding, all dimensions must be <=, x out > x in");
    MyFFTDebugAssertTrue(fwd_dims_out.y <= fwd_dims_in.y, "If padding, all dimensions must be <=, y out > y in");
    MyFFTDebugAssertTrue(fwd_dims_out.z <= fwd_dims_in.z, "If padding, all dimensions must be <=, z out > z in");

    fwd_size_change_type = decrease;
  }
  else if (fwd_dims_out.x == fwd_dims_in.x && fwd_dims_out.y == fwd_dims_in.y && fwd_dims_out.z == fwd_dims_in.z)
  {
    fwd_size_change_type = no_change;
  }
  else
  {
    // TODO: if this is relaxed, the dimensionality check below will be invalid.
    MyFFTRunTimeAssertTrue( false, "Error in validating fwd plan: Currently all dimensions must either increase, decrease or stay the same.");
  }

  // Validate the inverse transform
  if (inv_dims_out.x > inv_dims_in.x || inv_dims_out.y > inv_dims_in.y || inv_dims_out.z > inv_dims_in.z)
  {
    // For now we must pad in all dimensions, this is not needed and should be lifted. FIXME
    MyFFTDebugAssertTrue(inv_dims_out.x >= inv_dims_in.x, "If padding, all dimensions must be >=, x out < x in");
    MyFFTDebugAssertTrue(inv_dims_out.y >= inv_dims_in.y, "If padding, all dimensions must be >=, y out < y in");
    MyFFTDebugAssertTrue(inv_dims_out.z >= inv_dims_in.z, "If padding, all dimensions must be >=, z out < z in");

    inv_size_change_type = increase;
  }
  else if (inv_dims_out.x < inv_dims_in.x || inv_dims_out.y < inv_dims_in.y || inv_dims_out.z < inv_dims_in.z)
  {
    inv_size_change_type = decrease;
  }
  else if (inv_dims_out.x == inv_dims_in.x && inv_dims_out.y == inv_dims_in.y && inv_dims_out.z == inv_dims_in.z)
  {
    inv_size_change_type = no_change;
  }
  else
  {
    // TODO: if this is relaxed, the dimensionality check below will be invalid.
    MyFFTRunTimeAssertTrue( false, "Error in validating inv plan: Currently all dimensions must either increase, decrease or stay the same.");
  }

  // check for dimensionality
  // Note: this is predicated on the else clause ensuring all dimensions behave the same way w.r.t. size change.
  if (fwd_dims_in.z == 1 && fwd_dims_out.z == 1)
  {
    MyFFTRunTimeAssertTrue(inv_dims_in.z == 1 && inv_dims_out.z == 1, "Fwd/Inv dimensionality may not change from 1d,2d,3d (z dimension)");
    if (fwd_dims_in.y == 1 && fwd_dims_out.y == 1) 
    {
      MyFFTRunTimeAssertTrue(inv_dims_in.y == 1 && inv_dims_out.y == 1, "Fwd/Inv dimensionality may not change from 1d,2d,3d (y dimension)");
      transform_dimension = 1;
    }
    else 
    {
      transform_dimension = 2;
    }
  }
  else 
  {
    transform_dimension = 3;
  }

  is_size_validated = true;

}
template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::SetDimensions(DimensionCheckType check_op_type)
{
  // This should be run inside any public method call to ensure things ar properly setup.
  if ( ! is_size_validated ) ValidateDimensions();

  switch (check_op_type)
  {
    case CopyFromHost: {
      MyFFTDebugAssertTrue(transform_stage_completed == none, "When copying from host, the transform stage should be none, something has gone wrong.");
      memory_size_to_copy = input_memory_allocated;
      break;
    }

    case CopyToHost: {
      // FIXME currently there is no check that the right amount of memory is allocated on the host side array.
      switch (transform_stage_completed)
      {
        case no_change: {
          memory_size_to_copy = input_memory_allocated;
          break;
        }
        case fwd: {
          memory_size_to_copy = fwd_output_memory_allocated; 
          break;
        }
        case inv: {
          memory_size_to_copy = inv_output_memory_allocated;
          break;
        }
      } // switch transform_stage_completed
      break;
    } // case CopToHose

    case FwdTransform: {
      MyFFTDebugAssertTrue(transform_stage_completed == none || transform_stage_completed == inv, "When doing a forward transform, the transform stage completed should be none, something has gone wrong.");
      break;
    }

    case InvTransform: {
      MyFFTDebugAssertTrue(transform_stage_completed == fwd, "When doing an inverse transform, the transform stage completed should be fwd, something has gone wrong.");
      break;
    }
  } // end switch on operation type  

}

////////////////////////////////////////////////////
/// Transform kernels
////////////////////////////////////////////////////

// R2C_decomposed

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_R2C_decomposed(const ScalarType*  __restrict__ input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{

  using complex_type = ComplexType;
  using scalar_type  = ScalarType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 
  io_thread<FFT>::load_r2c(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], thread_data, Q);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
  FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P  
  io_thread<FFT>::remap_decomposed_segments(thread_data, shared_mem, twiddle_in, Q, mem_offsets.physical_x_output);


  io_thread<FFT>::store_r2c(shared_mem, &output_values[Return1DFFTAddress(mem_offsets.physical_x_output)], Q, mem_offsets.physical_x_output);

 
} // end of thread_fft_kernel_R2C

// R2C_decomposed_transposed

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_R2C_decomposed_transposed(const ScalarType*  __restrict__ input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{

  using complex_type = ComplexType;
  using scalar_type  = ScalarType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 
  io_thread<FFT>::load_r2c(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], thread_data, Q);

    // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
    FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments(thread_data, shared_mem, twiddle_in, Q, mem_offsets.physical_x_output);


  io_thread<FFT>::store_r2c_transposed_xy(shared_mem, &output_values[ ReturnZplane(blockDim.y, mem_offsets.physical_x_output)], Q, gridDim.y, mem_offsets.physical_x_output);

 
} // end of thread_fft_kernel_R2C_transposed

// R2C

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C_NONE_XY(const ScalarType* __restrict__ input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];


	// Memory used by FFT
  complex_type thread_data[FFT::storage_size];


  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  // multiply Q*fwd_dims_out.w because x maps to y in the output transposed FFT
  io<FFT>::load_r2c(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], thread_data);
  // io<FFT>::load_r2c(&input_values[blockIdx.y*mem_offsets.physical_x_input], thread_data);

  // In the first FFT the modifying twiddle factor is 1 so the data are real
  FFT().execute(thread_data, shared_mem, workspace);

  io<FFT>::store_r2c_transposed_xy(thread_data, &output_values[ ReturnZplane(gridDim.y, mem_offsets.physical_x_output)], gridDim.y);

 
} // end of block_fft_kernel_R2C_NONE_XY

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C_NONE_XZ(const ScalarType*  __restrict__ input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT
  complex_type thread_data[FFT::storage_size];

  io<FFT>::load_r2c(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], thread_data);

  FFT().execute(thread_data, shared_mem, workspace);

  // Transpose XZ, so the proper Z dimension now comes from X
  io<FFT>::store_r2c_transposed_xz(thread_data, output_values);

 
} // end of block_fft_kernel_R2C_NONE_XY

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_R2C_INCREASE_XY(const ScalarType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_input[];
  complex_type* shared_mem = (complex_type*)&shared_input[mem_offsets.shared_input];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data ... these really could be short ints, but I don't know how that will perform. TODO benchmark
  // It is also questionable whether storing these vs, recalculating makes more sense.
  int input_MAP[FFT::storage_size];
  int output_MAP[FFT::storage_size];
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  // multiply Q*fwd_dims_out.w because x maps to y in the output transposed FFT
  io<FFT>::load_r2c_shared(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], shared_input, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);

  // We unroll the first and last loops.
  // In the first FFT the modifying twiddle factor is 1 so the data are real
  FFT().execute(thread_data, shared_mem, workspace);  

  io<FFT>::store_r2c_transposed_xy(thread_data, &output_values[ ReturnZplane(blockDim.y, mem_offsets.physical_x_output)], output_MAP, gridDim.y);


  // For the other fragments we need the initial twiddle
  for (int sub_fft = 1; sub_fft < Q-1; sub_fft++)
  {

      io<FFT>::copy_from_shared(shared_input, thread_data, input_MAP);
      for (int i = 0; i < FFT::elements_per_thread; i++)
      {
        // Pre shift with twiddle
        SINCOS(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
        thread_data[i] *= twiddle;
        // increment the output mapping. 
        output_MAP[i]++;
      }
  
      FFT().execute(thread_data, shared_mem, workspace);


    io<FFT>::store_r2c_transposed_xy(thread_data, &output_values[ ReturnZplane(blockDim.y, mem_offsets.physical_x_output)], output_MAP, gridDim.y);
  }

  // For the last fragment we need to also do a bounds check.
  io<FFT>::copy_from_shared(shared_input, thread_data, input_MAP);
  for (int i = 0; i < FFT::elements_per_thread; i++)
  {
    // Pre shift with twiddle
    SINCOS(twiddle_factor_args[i]*(Q-1),&twiddle.y,&twiddle.x);
    thread_data[i] *= twiddle;
    // increment the output mapping. 
    output_MAP[i]++;
  }

  FFT().execute(thread_data, shared_mem, workspace);

  io<FFT>::store_r2c_transposed_xy(thread_data, &output_values[ ReturnZplane(blockDim.y, mem_offsets.physical_x_output)], output_MAP, gridDim.y, mem_offsets.physical_x_output);

} // end of block_fft_kernel_R2C_INCREASE_XY

// __launch_bounds__(FFT::max_threads_per_block)  we don't know this because it is threadDim.x * threadDim.z - this could be templated if it affects performance significantly
template<class FFT, class ComplexType, class ScalarType>
__global__
void block_fft_kernel_R2C_DECREASE_XY(const ScalarType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{
  // Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The shared memory is used for storage, shuffling and fft ops at different stages and includes room for bank padding.
	extern __shared__  complex_type shared_mem[];

  complex_type thread_data[FFT::storage_size];

  // Load in natural order
  io<FFT>::load_r2c_shared_and_pad(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], shared_mem);

  // DIT shuffle, bank conflict free
  io<FFT>::copy_from_shared(shared_mem, thread_data, Q);

  // The FFT operator has no idea we are using threadIdx.z to get multiple sub transforms, so we need to 
  // segment the shared memory it accesses to avoid conflicts.
  constexpr const unsigned int fft_shared_mem_num_elements = FFT::shared_memory_size / sizeof(complex_type);
  FFT().execute(thread_data, &shared_mem[fft_shared_mem_num_elements * threadIdx.z], workspace);
  __syncthreads();

  // Full twiddle multiply and store in natural order in shared memory
  io<FFT>::reduce_block_fft(thread_data, shared_mem, twiddle_in, Q);
 
  // Reduce from shared memory into registers, ending up with only P valid outputs.
  io<FFT>::store_r2c_reduced(thread_data, &output_values[ mem_offsets.physical_x_output * threadIdx.z ], gridDim.y, mem_offsets.physical_x_output);

} // end of block_fft_kernel_R2C_DECREASE_XY

// decomposed with conj multiplication

template<class FFT, class invFFT, class ComplexType>
__global__
void thread_fft_kernel_C2C_decomposed_ConjMul(const ComplexType* __restrict__ image_to_search, const ComplexType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{


  using complex_type = ComplexType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 
  io_thread<FFT>::load_c2c(&input_values[Return1DFFTAddress(size_of<FFT>::value)*Q], thread_data, Q);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
  FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments(thread_data, shared_mem, twiddle_in, Q, size_of<FFT>::value*Q);
  
  #if DEBUG_FFT_STAGE > 3
    io_thread<invFFT>::load_shared_and_conj_multiply(&image_to_search[Return1DFFTAddress(size_of<FFT>::value*Q)], shared_mem, thread_data, Q);
  #endif

  #if DEBUG_FFT_STAGE > 4
	  invFFT().execute(thread_data);
    // Now we need to aggregate each of the Q transforms into each output block of size P
    io_thread<invFFT>::remap_decomposed_segments(thread_data, shared_mem, -twiddle_in, Q,size_of<FFT>::value*Q);
  #endif

  io_thread<invFFT>::store_c2c(shared_mem, &output_values[Return1DFFTAddress(size_of<FFT>::value*Q)], Q);

}

// C2C with conj multiplication

template<class FFT, class invFFT, class ComplexType>
__launch_bounds__(invFFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul(const ComplexType* __restrict__ image_to_search, const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, 
                                                        Offsets mem_offsets, int apparent_Q, typename FFT::workspace_type workspace_fwd, typename invFFT::workspace_type workspace_inv)
{

  //	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	// __shared__ complex_type shared_mem[invFFT::shared_memory_size/sizeof(complex_type)]; // Storage for the input data that is re-used each blcok
	extern __shared__ complex_type shared_mem[]; // Storage for the input data that is re-used each blcok

  complex_type thread_data[FFT::storage_size];

  // For simplicity, we explicitly zeropad the input data to the size of the FFT.
  // It may be worth trying to use threadIdx.z as in the DECREASE methods.
  // Until then, this 
  io<FFT>::load(&input_values[Return1DFFTAddress(size_of<FFT>::value / apparent_Q)], thread_data, size_of<FFT>::value / apparent_Q);

  // In the first FFT the modifying twiddle factor is 1 so the data are reeal
  FFT().execute(thread_data, shared_mem, workspace_fwd);

  #if DEBUG_FFT_STAGE > 3
    //  * apparent_Q
    io<invFFT>::load_shared_and_conj_multiply(&image_to_search[Return1DFFTAddress(size_of<FFT>::value)], thread_data);
  #endif

  #if DEBUG_FFT_STAGE > 4
    invFFT().execute(thread_data, shared_mem, workspace_inv);
  #endif

  //  * apparent_Q
  io<invFFT>::store(thread_data, &output_values[Return1DFFTAddress(size_of<FFT>::value)]);


} // end of block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul

template<class FFT, class invFFT, class ComplexType>
__launch_bounds__(invFFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul_SwapRealSpaceQuadrants(const ComplexType* __restrict__ image_to_search, const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace_fwd, typename invFFT::workspace_type workspace_inv)
{

  //	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	// __shared__ complex_type shared_mem[invFFT::shared_memory_size/sizeof(complex_type)]; // Storage for the input data that is re-used each blcok
	extern __shared__ complex_type shared_mem[]; // Storage for the input data that is re-used each blcok

  complex_type thread_data[FFT::storage_size];


  io<FFT>::load(&input_values[Return1DFFTAddress(size_of<FFT>::value)], thread_data, size_of<FFT>::value);

  // In the first FFT the modifying twiddle factor is 1 so the data are reeal
  FFT().execute(thread_data, shared_mem, workspace_fwd);

  #if DEBUG_FFT_STAGE > 3
    // Swap real space quadrants using a phase shift by N/2 pixels 
    const unsigned int  stride = io<invFFT>::stride_size();
    int logical_y;
    for (unsigned int i = 0; i < FFT::elements_per_thread; i++) 
    {
      logical_y = threadIdx.x+ i*stride;
      // FIXME, not sure the physical_x_output is updated to replace the previous terms appropriately. This is supposed to be setting the conjugate terms.
      if ( logical_y >= mem_offsets.physical_x_output) logical_y -= mem_offsets.physical_x_output;
      if ( (int(blockIdx.y) + logical_y) % 2 != 0) thread_data[i] *= -1.f; // FIXME TYPE
    }

    io<invFFT>::load_shared_and_conj_multiply(&image_to_search[Return1DFFTAddress(size_of<FFT>::value * Q)], thread_data);
  #endif


  #if DEBUG_FFT_STAGE > 4
    invFFT().execute(thread_data, shared_mem, workspace_inv);
  #endif

  io<invFFT>::store(thread_data, &output_values[Return1DFFTAddress(size_of<FFT>::value * Q)]);



} // 

template<class FFT, class invFFT, class ComplexType>
__global__
void _INV_DECREASE_ConjMul( const ComplexType* __restrict__ image_to_search, const ComplexType* __restrict__  input_values, ComplexType* __restrict__  output_values, 
                                                Offsets mem_offsets, float twiddle_in, int apparent_Q, typename FFT::workspace_type workspace_fwd, typename invFFT::workspace_type workspace_inv)
{
  
    using complex_type = ComplexType;

    extern __shared__  complex_type shared_mem[]; 
  
    complex_type thread_data[FFT::storage_size];
  
    // Load in natural order
    io<FFT>::load(&input_values[Return1DFFTAddress(size_of<FFT>::value)], thread_data);

    // io<FFT>::load_c2c_shared_and_pad(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], shared_mem);
  
    // // DIT shuffle, bank conflict free
    // io<FFT>::copy_from_shared(shared_mem, thread_data, Q);
  
    // constexpr const unsigned int fft_shared_mem_num_elements = FFT::shared_memory_size / sizeof(complex_type);
    // FFT().execute(thread_data, &shared_mem[fft_shared_mem_num_elements * threadIdx.z], workspace_fwd);
    // __syncthreads();
    FFT().execute(thread_data, shared_mem, workspace_fwd);


    // // Full twiddle multiply and store in natural order in shared memory
    // io<FFT>::reduce_block_fft(thread_data, shared_mem, twiddle_in, Q);

    #if DEBUG_FFT_STAGE > 3
      // Load in imageFFT to search
      io<invFFT>::load_shared_and_conj_multiply(&image_to_search[Return1DFFTAddress(size_of<FFT>::value)], thread_data);
    #endif

    #if DEBUG_FFT_STAGE > 4
      // Run the inverse FFT
      // invFFT().execute(thread_data, &shared_mem[fft_shared_mem_num_elements * threadIdx.z], workspace_inv);
      invFFT().execute(thread_data, shared_mem, workspace_inv);

    #endif

    // // The reduced store considers threadIdx.z to ignore extra threads
    // io<invFFT>::store_c2c_reduced(thread_data, &output_values[blockIdx.y * gridDim.y]);
    #if DEBUG_FFT_STAGE < 5
      // There is no size reduction for this debug stage, so we need to use the pixel_pitch of the input array.
      io<invFFT>::store(thread_data, &output_values[Return1DFFTAddress(size_of<FFT>::value)]);
    #else
      // In the current simplified version of the kernel, I am not using any transform decomposition (this is because of the difficulties with resrved threadIdx.x/y in the cufftdx lib)
      // So the full thing is calculated and only truncated on output.
      io<invFFT>::store(thread_data, &output_values[Return1DFFTAddress(size_of<FFT>::value / apparent_Q)], size_of<FFT>::value / apparent_Q);
    #endif


} // end of _INV_DECREASE_ConjMul

// C2C

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_NONE(const ComplexType*  __restrict__  input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

 //	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_mem[]; // Storage for the input data that is re-used each blcok


	// Memory used by FFT
  complex_type thread_data[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load(&input_values[Return1DFFTAddress(size_of<FFT>::value)],  thread_data);

  // Since the memory ops are super straightforward this is an okay compromise.
	FFT().execute(thread_data, shared_mem, workspace);

	io<FFT>::store(thread_data ,&output_values[Return1DFFTAddress(size_of<FFT>::value)]);


} // end of block_fft_kernel_C2C_NONE

// C2C decomposed

// __launch_bounds__(FFT::max_threads_per_block)  we don't know this because it is threadDim.x * threadDim.z - this could be templated if it affects performance significantly
template<class FFT, class ComplexType>
__global__
void block_fft_kernel_C2C_DECREASE(const ComplexType* __restrict__  input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{
  //	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

  extern __shared__  complex_type shared_mem[]; 

  complex_type thread_data[FFT::storage_size];

  // Load in natural order
  io<FFT>::load_c2c_shared_and_pad(&input_values[Return1DFFTAddress(size_of<FFT>::value * Q)], shared_mem);

  // DIT shuffle, bank conflict free
  io<FFT>::copy_from_shared(shared_mem, thread_data, Q);


  constexpr const unsigned int fft_shared_mem_num_elements = FFT::shared_memory_size / sizeof(complex_type);
  FFT().execute(thread_data, &shared_mem[fft_shared_mem_num_elements * threadIdx.z], workspace);
  __syncthreads();

  // Full twiddle multiply and store in natural order in shared memory
  io<FFT>::reduce_block_fft(thread_data, shared_mem, twiddle_in, Q);

  // Reduce from shared memory into registers, ending up with only P valid outputs.
  io<FFT>::store_c2c_reduced(thread_data, &output_values[Return1DFFTAddress(size_of<FFT>::value)]);


}

// __launch_bounds__(FFT::max_threads_per_block)  we don't know this because it is threadDim.x * threadDim.z - this could be templated if it affects performance significantly
template<class FFT, class ComplexType>
__global__
void block_fft_kernel_C2C_INCREASE(const ComplexType*  __restrict__ input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{
 //	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
	complex_type* shared_output = (complex_type*)&shared_input_complex[mem_offsets.shared_input]; // storage for the coalesced output data. This may grow too large, 
	complex_type* shared_mem = (complex_type*)&shared_output[mem_offsets.shared_output];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];

  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load_shared(&input_values[Return1DFFTAddress(size_of<FFT>::value)], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);

	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	// 
  io<FFT>::store(thread_data,shared_output,output_MAP);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);

		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			SINCOS(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		    // increment the output map. Note this only works for the leading non-zero case
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);

    io<FFT>::store(thread_data,shared_output,output_MAP);


	}

  // TODO confirm this is needed
	__syncthreads();

	// Now that the memory output can be coalesced send to global
  // FIXME is this actually coalced?
	for (int sub_fft = 0; sub_fft < Q; sub_fft++)
	{
    io<FFT>::store_coalesced(shared_output, &output_values[Return1DFFTAddress(size_of<FFT>::value * Q)], sub_fft*mem_offsets.shared_input);
	}


} // end of block_fft_kernel_C2C_INCREASE

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_INCREASE_SwapRealSpaceQuadrants(const ComplexType*  __restrict__  input_values, ComplexType*  __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{

 //	// Initialize the shared memory, assuming everyting matches the input data X size in
  using complex_type = ComplexType;

	extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
	complex_type* shared_output = (complex_type*)&shared_input_complex[mem_offsets.shared_input]; // storage for the coalesced output data. This may grow too large, 
	complex_type* shared_mem = (complex_type*)&shared_output[mem_offsets.shared_output];


	// Memory used by FFT
	complex_type twiddle;
  complex_type thread_data[FFT::storage_size];

  // To re-map the thread index to the data
  int input_MAP[FFT::storage_size];
  // To re-map the decomposed frequency to the full output frequency
  int output_MAP[FFT::storage_size];
  // For a given decomposed fragment
  float twiddle_factor_args[FFT::storage_size];


  // No need to __syncthreads as each thread only accesses its own shared mem anyway
  io<FFT>::load_shared(&input_values[Return1DFFTAddress(size_of<FFT>::value)], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q);

	// In the first FFT the modifying twiddle factor is 1 so the data are reeal
	FFT().execute(thread_data, shared_mem, workspace);

	// FIXME I have not confirmed on switch to physical_x_output that this represents the index of the first negative frequency in Y as it should.
  io<FFT>::store_and_swap_quadrants(thread_data,shared_output,output_MAP,size_of<FFT>::value * Q);

    // For the other fragments we need the initial twiddle
	for (int sub_fft = 1; sub_fft < Q; sub_fft++)
	{

	  io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);

		for (int i = 0; i < FFT::elements_per_thread; i++)
		{
			// Pre shift with twiddle
			SINCOS(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
			thread_data[i] *= twiddle;
		    // increment the output map. Note this only works for the leading non-zero case
			output_MAP[i]++;
		}

		FFT().execute(thread_data, shared_mem, workspace);
    io<FFT>::store_and_swap_quadrants(thread_data,shared_output,output_MAP,size_of<FFT>::value * Q);


	}

  // TODO confirm this is needed
	__syncthreads();

	// Now that the memory output can be coalesced send to global
  // FIXME is this actually coalced?
	for (int sub_fft = 0; sub_fft < Q; sub_fft++)
	{
    io<FFT>::store_coalesced(shared_output, &output_values[Return1DFFTAddress(size_of<FFT>::value * Q)], sub_fft*mem_offsets.shared_input);
	}


} // end of block_fft_kernel_C2C_INCREASE_SwapRealSpaceQuadrants

template<class FFT, class ComplexType>
__global__
void thread_fft_kernel_C2C_decomposed(const ComplexType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, float twiddle_in, int Q)
{


  using complex_type = ComplexType;
  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  complex_type shared_mem[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 
  io_thread<FFT>::load_c2c(&input_values[Return1DFFTAddress(size_of<FFT>::value)], thread_data, Q);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
  FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments(thread_data, shared_mem, twiddle_in, Q, size_of<FFT>::value * Q);

  io_thread<FFT>::store_c2c(shared_mem, &output_values[Return1DFFTAddress(size_of<FFT>::value * Q)], Q);

}


template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_NONE_XY(const ComplexType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

  //	// Initialize the shared memory, assuming everyting matches the input data X size in
   using complex_type = ComplexType;
 
   extern __shared__  complex_type shared_mem[]; // Storage for the input data that is re-used each blcok
 
   // Memory used by FFT
   complex_type thread_data[FFT::storage_size];
 
   // No need to __syncthreads as each thread only accesses its own shared mem anyway
   io<FFT>::load_Z(&input_values[ Return1DFFTAddress_YX() ], thread_data, gridDim.y * gridDim.z );
 
   // Since the memory ops are super straightforward this is an okay compromise.
   FFT().execute(thread_data, shared_mem, workspace);
   
   io<FFT>::store_Z(thread_data, &output_values[ Return1DFFTAddress_YX_to_XY() ], gridDim.y * gridDim.z);
  
}

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_NONE_XYZ(const ComplexType* __restrict__  input_values, ComplexType* __restrict__  output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

  //	// Initialize the shared memory, assuming everyting matches the input data X size in
   using complex_type = ComplexType;
 
   extern __shared__  complex_type shared_mem[]; // Storage for the input data that is re-used each blcok
 
   // Memory used by FFT
   complex_type thread_data[FFT::storage_size];
 
   // No need to __syncthreads as each thread only accesses its own shared mem anyway
   io<FFT>::load_Z(&input_values[ Return1DFFTColumn_XZ_transpose(size_of<FFT>::value) ], thread_data );
 
   // Since the memory ops are super straightforward this is an okay compromise.
   FFT().execute(thread_data, shared_mem, workspace);
 
   io<FFT>::store_Z(thread_data, &output_values[ Return1DFFTColumn_XZ_to_XY() ], gridDim.y * gridDim.z);
  
}

template<class FFT, class ComplexType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2C_INCREASE_Z(const ComplexType* __restrict__  input_values, ComplexType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q, typename FFT::workspace_type workspace)
{
  // //	// Initialize the shared memory, assuming everyting matches the input data X size in
  //  using complex_type = ComplexType;
 
  //  extern __shared__  complex_type shared_input_complex[]; // Storage for the input data that is re-used each blcok
  //  complex_type* shared_mem = (complex_type*)&shared_input_complex[size_of<FFT>::value];
 
 
  //  // Memory used by FFT
  //  complex_type twiddle;
  //  complex_type thread_data[FFT::storage_size];
 
  //  // To re-map the thread index to the data
  //  int input_MAP[FFT::storage_size];
  //  // To re-map the decomposed frequency to the full output frequency
  //  int output_MAP[FFT::storage_size];
  //  // For a given decomposed fragment
  //  float twiddle_factor_args[FFT::storage_size];
 
  //  // mem_offsets.pixel_pitch_output is the number of elements nw * ny;
  // //  io<FFT>::load_shared_Z(&input_values[blockIdx.y*mem_offsets.pixel_pitch_input], shared_input_complex, thread_data, twiddle_factor_args, twiddle_in, input_MAP, output_MAP, Q, mem_offsets.pixel_pitch_output);
 
  //  // In the first FFT the modifying twiddle factor is 1 so the data are reeal
  //  FFT().execute(thread_data, shared_mem, workspace);
 
  //  // 
  //  io<FFT>::store(thread_data, output_values, output_MAP);
 
  //    // For the other fragments we need the initial twiddle
  //  for (int sub_fft = 1; sub_fft < Q; sub_fft++)
  //  {
 
  //    io<FFT>::copy_from_shared(shared_input_complex, thread_data, input_MAP);
 
  //    for (int i = 0; i < FFT::elements_per_thread; i++)
  //    {
  //      // Pre shift with twiddle
  //      SINCOS(twiddle_factor_args[i]*sub_fft,&twiddle.y,&twiddle.x);
  //      thread_data[i] *= twiddle;
  //        // increment the output map. Note this only works for the leading non-zero case
  //      output_MAP[i]++;
  //    }
 
  //    FFT().execute(thread_data, shared_mem, workspace);
 
  //    io<FFT>::store(thread_data, output_values, output_MAP);
 
  //  }
 

  
} // end of block_fft_kernel_C2C_INCREASE_Z 


template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2R_NONE(const ComplexType*  __restrict__ input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

	using complex_type = ComplexType;
	using scalar_type  = ScalarType;

	extern __shared__  complex_type shared_mem[];


  complex_type thread_data[FFT::storage_size];

  io<FFT>::load_c2r(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], thread_data);

  // For loop zero the twiddles don't need to be computed
  FFT().execute(thread_data, shared_mem, workspace);
  
  io<FFT>::store_c2r(thread_data, &output_values[Return1DFFTAddress(mem_offsets.physical_x_output)], size_of<FFT>::value);

}

template<class FFT, class ComplexType, class ScalarType>
__launch_bounds__(FFT::max_threads_per_block) __global__
void block_fft_kernel_C2R_NONE_XY(const ComplexType* __restrict__  input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, typename FFT::workspace_type workspace)
{

	using complex_type = ComplexType;
	using scalar_type  = ScalarType;

	extern __shared__  complex_type shared_mem[];


  complex_type thread_data[FFT::storage_size];

  io<FFT>::load_c2r_transposed(&input_values[ReturnZplane(gridDim.y, mem_offsets.physical_x_input)], thread_data, gridDim.y);

  // For loop zero the twiddles don't need to be computed
  FFT().execute(thread_data, shared_mem, workspace);
  
  io<FFT>::store_c2r(thread_data, &output_values[Return1DFFTAddress(mem_offsets.physical_x_output)], size_of<FFT>::value);

} // end of block_fft_kernel_C2R_NONE_XY

template<class FFT, class ComplexType, class ScalarType>
__global__
void block_fft_kernel_C2R_DECREASE_XY(const ComplexType*  __restrict__ input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, const float twiddle_in, const unsigned int Q, typename FFT::workspace_type workspace)
{

	using complex_type = ComplexType;
	using scalar_type  = ScalarType;

	extern __shared__  complex_type shared_mem[];

  complex_type thread_data[FFT::storage_size];


  io<FFT>::load_c2r_transposed(&input_values[ReturnZplane(gridDim.y, mem_offsets.physical_x_input)], thread_data, gridDim.y);
  
  // For loop zero the twiddles don't need to be computed
  FFT().execute(thread_data, shared_mem, workspace);

  io<FFT>::store_c2r(thread_data, &output_values[Return1DFFTAddress(mem_offsets.physical_x_output)], size_of<FFT>::value);

  // // Load transposed data into shared memory in natural order.
  // io<FFT>::load_c2r_shared_and_pad(&input_values[blockIdx.y], shared_mem, mem_offsets.physical_x_input);

  // // DIT shuffle, bank conflict free
  // io<FFT>::copy_from_shared(shared_mem, thread_data, Q);

  // constexpr const unsigned int fft_shared_mem_num_elements = FFT::shared_memory_size / sizeof(complex_type);
  // FFT().execute(thread_data, &shared_mem[fft_shared_mem_num_elements * threadIdx.z], workspace);
  // __syncthreads();

  // // Full twiddle multiply and store in natural order in shared memory
  // io<FFT>::reduce_block_fft(thread_data, shared_mem, twiddle_in, Q);

  // // Reduce from shared memory into registers, ending up with only P valid outputs.
  // io<FFT>::store_c2r_reduced(thread_data, &output_values[Return1DFFTAddress(mem_offsets.physical_x_output)]);


} // end of block_fft_kernel_C2R_DECREASE_XY

// C2R decomposed

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_C2R_decomposed(const ComplexType*  __restrict__ input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q)
{
  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_mem_C2R_decomposed[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 

  io_thread<FFT>::load_c2r(&input_values[Return1DFFTAddress(mem_offsets.physical_x_input)], thread_data, Q, mem_offsets.physical_x_input);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
  FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments_c2r(thread_data, shared_mem_C2R_decomposed, twiddle_in, Q);

  io_thread<FFT>::store_c2r(shared_mem_C2R_decomposed, &output_values[Return1DFFTAddress(mem_offsets.physical_x_output)],Q);
}

template<class FFT, class ComplexType, class ScalarType>
__global__
void thread_fft_kernel_C2R_decomposed_transposed(const ComplexType*  __restrict__ input_values, ScalarType*  __restrict__ output_values, Offsets mem_offsets, float twiddle_in, int Q)
{

  using complex_type = ComplexType;
  using scalar_type  = ScalarType;

  // The data store is non-coalesced, so don't aggregate the data in shared mem.
	extern __shared__  scalar_type shared_mem_transposed[];

	// Memory used by FFT - for Thread() type, FFT::storage_size == FFT::elements_per_thread == size_of<FFT>::value
  complex_type thread_data[FFT::storage_size];
 

  io_thread<FFT>::load_c2r_transposed(&input_values[ReturnZplane(blockDim.y, mem_offsets.physical_x_input)], thread_data, Q, gridDim.y, mem_offsets.physical_x_input);

  // We then have Q FFTs of size size_of<FFT>::value (P in the paper)
  FFT().execute(thread_data);

  // Now we need to aggregate each of the Q transforms into each output block of size P
  io_thread<FFT>::remap_decomposed_segments_c2r(thread_data, shared_mem_transposed, twiddle_in, Q);

  io_thread<FFT>::store_c2r(shared_mem_transposed, &output_values[Return1DFFTAddress(mem_offsets.physical_x_output)],Q);

}

// FIXME assumed FWD 
template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::ClipIntoTopLeft()
{
  // TODO add some checks and logic.

  // Assuming we are calling this from R2C_Transposed and that the launch bounds are not set.
  dim3 threadsPerBlock;
  dim3 gridDims;

  threadsPerBlock = dim3(512,1,1);
  gridDims = dim3( (fwd_dims_out.x + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

  const short4 area_to_clip_from = make_short4(fwd_dims_in.x, fwd_dims_in.y, fwd_dims_in.w*2, fwd_dims_out.w*2);

  precheck
  clip_into_top_left_kernel<float, float><< < gridDims, threadsPerBlock, 0, hipStreamPerThread >> >
  (d_ptr.position_space, d_ptr.position_space, area_to_clip_from);
  postcheck
}

// FIXME assumed FWD 
template<typename InputType, typename OutputType>
__global__ void clip_into_top_left_kernel(InputType*  input_values, OutputType* output_values, short4 dims )
{

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x > dims.w) return; // Out of bounds. 

  // dims.w is the pitch of the output array
  if (blockIdx.y > dims.y) { output_values[blockIdx.y * dims.w + x] = OutputType(0); return; }

  if (threadIdx.x > dims.x) { output_values[blockIdx.y * dims.w + x] = OutputType(0); return; }
  else 
  {
    // dims.z is the pitch of the output array
    output_values[blockIdx.y * dims.w + x] = input_values[blockIdx.y * dims.z + x];
    return;
  }
} // end of clip_into_top_left_kernel

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::ClipIntoReal(int wanted_coordinate_of_box_center_x, int wanted_coordinate_of_box_center_y, int wanted_coordinate_of_box_center_z)
{
  // TODO add some checks and logic.

  // Assuming we are calling this from R2C_Transposed and that the launch bounds are not set.
  dim3 threadsPerBlock;
  dim3 gridDims;
  int3 wanted_center = make_int3(wanted_coordinate_of_box_center_x, wanted_coordinate_of_box_center_y, wanted_coordinate_of_box_center_z);
  threadsPerBlock = dim3(32,32,1);
  gridDims = dim3( (fwd_dims_out.x + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (fwd_dims_out.y + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                   1);

  const short4 area_to_clip_from = make_short4(fwd_dims_in.x, fwd_dims_in.y, fwd_dims_in.w*2, fwd_dims_out.w*2);
  float wanted_padding_value = 0.f;
  
  precheck
  clip_into_real_kernel<float, float><< < gridDims, threadsPerBlock, 0, hipStreamPerThread >> >
  (d_ptr.position_space, d_ptr.position_space, fwd_dims_in, fwd_dims_out,wanted_center, wanted_padding_value);
  postcheck

}
// Modified from GpuImage::ClipIntoRealKernel
template<typename InputType, typename OutputType>
__global__ void clip_into_real_kernel(InputType* real_values_gpu,
                                      OutputType* other_image_real_values_gpu,
                                      short4 dims, 
                                      short4 other_dims,
                                      int3 wanted_coordinate_of_box_center, 
                                      OutputType wanted_padding_value)
{
  int3 other_coord = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                               blockIdx.y*gridDim.y + threadIdx.y,
                               blockIdx.z);

  int3 coord = make_int3(0, 0, 0); 

  if (other_coord.x < other_dims.x &&
      other_coord.y < other_dims.y &&
      other_coord.z < other_dims.z)
  {

    coord.z = dims.z/2 + wanted_coordinate_of_box_center.z + 
    other_coord.z - other_dims.z/2;

    coord.y = dims.y/2 + wanted_coordinate_of_box_center.y + 
    other_coord.y - other_dims.y/2;

    coord.x = dims.x + wanted_coordinate_of_box_center.x + 
    other_coord.x - other_dims.x;

    if (coord.z < 0 || coord.z >= dims.z || 
        coord.y < 0 || coord.y >= dims.y ||
        coord.x < 0 || coord.x >= dims.x)
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = wanted_padding_value;
    }
    else
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = 
      real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(coord, dims) ];
    }

  } // end of bounds check

} // end of ClipIntoRealKernel

template <class ComputeType, class InputType, class OutputType>
template <bool use_thread_method>
void FourierTransformer<ComputeType, InputType, OutputType>::SetPrecisionAndExectutionMethod(KernelType kernel_type, bool do_forward_transform)
{
  // For kernels with fwd and inv transforms, we want to not set the direction yet.
  

  static const bool is_half = std::is_same_v<ComputeType, __half>;
  static const bool is_float = std::is_same_v<ComputeType, float>;
  static_assert( is_half || is_float , "FourierTransformer::SetPrecisionAndExectutionMethod: Unsupported ComputeType");


  if constexpr (use_thread_method)
  {
    using FFT = decltype(Thread() + Size<32>() + Precision<ComputeType>());
    SelectSizeAndType<FFT>(kernel_type, do_forward_transform);

  }
  else
  {
    using FFT = decltype( Block() + Precision<ComputeType>()  + FFTsPerBlock<1>() );
    SelectSizeAndType<FFT>(kernel_type, do_forward_transform);
  }
  

}

template <class ComputeType, class InputType, class OutputType>
template <class FFT_base>
void FourierTransformer<ComputeType, InputType, OutputType>::SelectSizeAndType(KernelType kernel_type, bool do_forward_transform)
{


  if constexpr (detail::is_operator<fft_operator::thread, FFT_base>::value)
  {
    GetTransformSize_thread(kernel_type, size_of<FFT_base>::value);
    switch (device_properties.device_arch)
    {
      case 700: { using FFT = decltype(FFT_base() + SM<700>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      case 750: { using FFT = decltype(FFT_base() + SM<750>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      case 800: { using FFT = decltype(FFT_base() + SM<800>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
    }
  }
  else
  {
    GetTransformSize(kernel_type);

    switch (transform_size.P)
    {
      case 16: {
        elements_per_thread_complex = elements_per_thread_16;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<16>()  + SM<700>() + ElementsPerThread<elements_per_thread_16>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<16>()  + SM<750>() + ElementsPerThread<elements_per_thread_16>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<16>()  + SM<800>() + ElementsPerThread<elements_per_thread_16>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; }

      case 32: {
        elements_per_thread_complex = elements_per_thread_32;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<32>()  + SM<700>() + ElementsPerThread<elements_per_thread_32>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<32>()  + SM<750>() + ElementsPerThread<elements_per_thread_32>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<32>()  + SM<800>() + ElementsPerThread<elements_per_thread_32>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; }

      case 64: {
        elements_per_thread_complex = elements_per_thread_64;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<64>()  + SM<700>() + ElementsPerThread<elements_per_thread_64>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<64>()  + SM<750>() + ElementsPerThread<elements_per_thread_64>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<64>()  + SM<800>() + ElementsPerThread<elements_per_thread_64>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; }
  
      case 128: {
        elements_per_thread_complex = elements_per_thread_128;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<128>()  + SM<700>() + ElementsPerThread<elements_per_thread_128>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<128>()  + SM<750>() + ElementsPerThread<elements_per_thread_128>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<128>()  + SM<800>() + ElementsPerThread<elements_per_thread_128>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; }
  
      case 256: {
        elements_per_thread_complex = elements_per_thread_256;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<256>()  + SM<700>() + ElementsPerThread<elements_per_thread_256>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<256>()  + SM<750>() + ElementsPerThread<elements_per_thread_256>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<256>()  + SM<800>() + ElementsPerThread<elements_per_thread_256>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; } 
  
      case 512: {
        elements_per_thread_complex = elements_per_thread_512;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<512>()  + SM<700>() + ElementsPerThread<elements_per_thread_512>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<512>()  + SM<750>() + ElementsPerThread<elements_per_thread_512>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<512>()  + SM<800>() + ElementsPerThread<elements_per_thread_512>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; } 
  
      // case 768: {
      //   switch (device_properties.device_arch)
      //   {
      //     case 700: { using FFT = decltype(FFT_base()  + Size<768>()  + SM<700>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      //     case 750: { using FFT = decltype(FFT_base()  + Size<768>()  + SM<750>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      //     case 800: { using FFT = decltype(FFT_base()  + Size<768>()  + SM<800>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      //   }
      // break; } 
  
      case 1024: {
        elements_per_thread_complex = elements_per_thread_1024;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<1024>()  + SM<700>() + ElementsPerThread<elements_per_thread_1024>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<1024>()  + SM<750>() + ElementsPerThread<elements_per_thread_1024>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<1024>()  + SM<800>() + ElementsPerThread<elements_per_thread_1024>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
      break; } 
  
      // case 1536: {
      //   switch (device_properties.device_arch)
      //   {
      //     case 700: { using FFT = decltype(FFT_base()  + Size<1536>()  + SM<700>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      //     // case 750: { using FFT = decltype(FFT_base()  + Size<1536>()  + SM<750>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      //     case 800: { using FFT = decltype(FFT_base()  + Size<1536>()  + SM<800>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
      //   }
      // break; }    
  
      case 2048: {
        elements_per_thread_complex = elements_per_thread_2048;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<2048>()  + SM<700>() + ElementsPerThread<elements_per_thread_2048>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 750: { using FFT = decltype(FFT_base()  + Size<2048>()  + SM<750>() + ElementsPerThread<elements_per_thread_2048>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<2048>()  + SM<800>() + ElementsPerThread<elements_per_thread_2048>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; } 
  
  
      case 4096: {
        elements_per_thread_complex = elements_per_thread_4096;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<4096>()  + SM<700>() + ElementsPerThread<elements_per_thread_4096>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          // case 750: { using FFT = decltype(FFT_base()  + Size<4096>()  + SM<750>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<4096>()  + SM<800>() + ElementsPerThread<elements_per_thread_4096>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; }  
  
      case 8192: {
        elements_per_thread_complex = elements_per_thread_8192;
        switch (device_properties.device_arch)
        {
          case 700: { using FFT = decltype(FFT_base()  + Size<8192>()  + SM<700>() + ElementsPerThread<elements_per_thread_8192>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
          case 800: { using FFT = decltype(FFT_base()  + Size<8192>()  + SM<800>() + ElementsPerThread<elements_per_thread_8192>());  SetAndLaunchKernel<FFT>(kernel_type, do_forward_transform); break;}
        }
        break; } 

      default: {
        MyFFTRunTimeAssertTrue(false, "FFT size not supported");
      }
    }

  }

}

template <class ComputeType, class InputType, class OutputType>
template <class FFT_base_arch, bool use_thread_method>
void FourierTransformer<ComputeType, InputType, OutputType>::SetAndLaunchKernel(KernelType kernel_type, bool do_forward_transform)
{

  using complex_type = typename FFT_base_arch::value_type;
	using scalar_type    = typename complex_type::value_type;

  complex_type* complex_input;
  complex_type* complex_output;
  scalar_type*  scalar_input;
  scalar_type*  scalar_output;

  // Make sure we are in the right chunk of the memory pool.
  if (is_in_buffer_memory) 
  {
    complex_input  = (complex_type*)d_ptr.momentum_space_buffer;
    complex_output = (complex_type*)d_ptr.momentum_space;

    scalar_input   = (scalar_type*)d_ptr.position_space_buffer;
    scalar_output  = (scalar_type*)d_ptr.position_space;

    is_in_buffer_memory = false;
  }
  else
  {
    complex_input  = (complex_type*)d_ptr.momentum_space;
    complex_output = (complex_type*)d_ptr.momentum_space_buffer;

    scalar_input   = (scalar_type*)d_ptr.position_space;
    scalar_output  = (scalar_type*)d_ptr.position_space_buffer;

    is_in_buffer_memory = true;
  }

  
  if constexpr (detail::is_operator<fft_operator::thread, FFT_base_arch>::value)
  {
    switch (kernel_type)
    {
      case r2c_decomposed: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() ); 

        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_decomposed);  
      
        int shared_mem = LP.mem_offsets.shared_output * sizeof(complex_type);
        CheckSharedMemory(shared_mem, device_properties);
        cudaErr(hipFuncSetAttribute((void*)thread_fft_kernel_R2C_decomposed<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem));        
        #if DEBUG_FFT_STAGE > 0
          precheck
          thread_fft_kernel_R2C_decomposed<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
          (scalar_input, complex_output, LP.mem_offsets, LP.twiddle_in, LP.Q);
          postcheck
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif    
        break; 
      }

      case r2c_decomposed_transposed: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() ); 

        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_decomposed_transposed);
      
        int shared_mem = LP.mem_offsets.shared_output * sizeof(complex_type);
        CheckSharedMemory(shared_mem, device_properties);
        cudaErr(hipFuncSetAttribute((void*)thread_fft_kernel_R2C_decomposed_transposed<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem));        
        #if DEBUG_FFT_STAGE > 0
          precheck
          thread_fft_kernel_R2C_decomposed_transposed<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_mem, hipStreamPerThread>> >
          (scalar_input, complex_output, LP.mem_offsets, LP.twiddle_in, LP.Q);
          postcheck
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break; 
      }
    case c2r_decomposed: {

      // Note that unlike the block C2R we require a C2C sub xform.
      using FFT = decltype(FFT_base_arch() + Direction<fft_direction::inverse>() + Type<fft_type::c2c>());
      // TODO add completeness check.

      LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_decomposed);
      int shared_memory = LP.mem_offsets.shared_output * sizeof(scalar_type);
      CheckSharedMemory(shared_memory, device_properties);
      cudaErr(hipFuncSetAttribute((void*)thread_fft_kernel_C2R_decomposed<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
      #if DEBUG_FFT_STAGE > 6
        precheck
        thread_fft_kernel_C2R_decomposed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
        (complex_input, complex_output, LP.mem_offsets, LP.twiddle_in, LP.Q);
        postcheck
      #else
        is_in_buffer_memory = ! is_in_buffer_memory;
      #endif

      break; 
    }
    case c2r_decomposed_transposed: {  
      // Note that unlike the block C2R we require a C2C sub xform.
      using FFT = decltype(FFT_base_arch() + Direction<fft_direction::inverse>() + Type<fft_type::c2c>());

      LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_decomposed_transposed);
      int shared_memory = LP.mem_offsets.shared_output * sizeof(scalar_type);
      CheckSharedMemory(shared_memory, device_properties);
      cudaErr(hipFuncSetAttribute((void*)thread_fft_kernel_C2R_decomposed_transposed<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
      #if DEBUG_FFT_STAGE > 6
        precheck
        thread_fft_kernel_C2R_decomposed_transposed<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
        (complex_input, complex_output, LP.mem_offsets, LP.twiddle_in, LP.Q);
        postcheck
      #else
        is_in_buffer_memory = ! is_in_buffer_memory;
      #endif
    
      break; 
    } 
    case xcorr_decomposed: {

      using    FFT = decltype( FFT_base_arch() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() );  
      using invFFT = decltype( FFT_base_arch() + Type<fft_type::c2c>() + Direction<fft_direction::inverse>() ); 

      LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, xcorr_decomposed);

      int shared_memory = LP.mem_offsets.shared_output * sizeof(complex_type);
      CheckSharedMemory(shared_memory, device_properties);

      // FIXME
      bool swap_real_space_quadrants = false;

      if (swap_real_space_quadrants)
      {
        MyFFTRunTimeAssertTrue(false, "decomposed xcorr with swap real space quadrants is not implemented.");
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul_SwapRealSpaceQuadrants<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        

        // precheck
        // block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul_SwapRealSpaceQuadrants<FFT,invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
        // ( (complex_type*) image_to_search, (complex_type*)  d_ptr.momentum_space_buffer,  (complex_type*) d_ptr.momentum_space, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace_fwd, workspace_inv);
        // postcheck
      }
      else
      {
        cudaErr(hipFuncSetAttribute((void*)thread_fft_kernel_C2C_decomposed_ConjMul<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        

        #if DEBUG_FFT_STAGE > 2
          // the image_to_search pointer is set during call to CrossCorrelate,
          precheck
          thread_fft_kernel_C2C_decomposed_ConjMul<FFT, invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( (complex_type *)d_ptr.image_to_search, complex_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q);
          postcheck
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
      }
      
      break; 
    }
    case c2c_decomposed: {
      using FFT_nodir = decltype(FFT_base_arch() + Type<fft_type::c2c>() );

      LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_decomposed, do_forward_transform);

      if (do_forward_transform)
      {
        using FFT = decltype( FFT_nodir() + Direction<fft_direction::forward>() );
        int shared_memory = LP.mem_offsets.shared_output * sizeof(complex_type);
        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)thread_fft_kernel_C2C_decomposed<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
        #if DEBUG_FFT_STAGE > 2
          precheck
          thread_fft_kernel_C2C_decomposed<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, complex_output, LP.mem_offsets, LP.twiddle_in, LP.Q);
          postcheck
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
      }
      else
      {
    
        using FFT = decltype( FFT_nodir() + Direction<fft_direction::inverse>() );
        int shared_memory = LP.mem_offsets.shared_output * sizeof(complex_type);
        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)thread_fft_kernel_C2C_decomposed<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
        #if DEBUG_FFT_STAGE > 4
          precheck
          thread_fft_kernel_C2C_decomposed<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, complex_output, LP.mem_offsets, LP.twiddle_in, LP.Q);
          postcheck
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;  
        #endif
      }
    }
    
    break; 
    }    
  }
  else // Block
  {
    switch (kernel_type)
    {
      case r2c_none_XY: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() );  
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_none_XY);

        int shared_memory = FFT::shared_memory_size;
        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_R2C_NONE_XY<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
   
        // cudaErr(hipSetDevice(0));
        //  cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_R2C_NONE_XY<FFT,complex_type,scalar_type>,hipFuncCachePreferShared ));
        //  hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_R2C_NONE_XY<FFT,complex_type,scalar_type>, hipSharedMemBankSizeEightByte );

        #if DEBUG_FFT_STAGE > 0
          precheck
          block_fft_kernel_R2C_NONE_XY<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          (scalar_input, complex_output, LP.mem_offsets, workspace);
          postcheck 
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif


           
        break;
      }

      case r2c_none_XZ: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() );  
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_none_XZ);

        int shared_memory = FFT::shared_memory_size;
        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_R2C_NONE_XZ<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
   
        // cudaErr(hipSetDevice(0));
        //  cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_R2C_NONE_XZ<FFT,complex_type,scalar_type>,hipFuncCachePreferShared ));
        //  hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_R2C_NONE_XZ<FFT,complex_type,scalar_type>, hipSharedMemBankSizeEightByte );


        #if DEBUG_FFT_STAGE > 0
          precheck
          block_fft_kernel_R2C_NONE_XZ<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          (scalar_input, complex_output, LP.mem_offsets, workspace);
          postcheck 
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif


           
        break;
      }      
      
      case r2c_decrease: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() );  
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_decrease);

        // the shared mem is mixed between storage, shuffling and FFT. For this kernel we need to add padding to avoid bank conlicts (N/32)
        int shared_memory = std::max( FFT::shared_memory_size * LP.threadsPerBlock.z, (LP.mem_offsets.shared_input + LP.mem_offsets.shared_input/32) * (unsigned int)sizeof(complex_type));

        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_R2C_DECREASE_XY<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));  

        #if DEBUG_FFT_STAGE > 0
          precheck
          block_fft_kernel_R2C_DECREASE_XY<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( scalar_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
          postcheck 
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
        break;
      }

      case r2c_increase: {
        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() );  
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, r2c_increase);

        int shared_memory = LP.mem_offsets.shared_input*sizeof(scalar_type) + FFT::shared_memory_size;

        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_R2C_INCREASE_XY<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
    
        #if DEBUG_FFT_STAGE > 0
          precheck
          block_fft_kernel_R2C_INCREASE_XY<FFT,complex_type,scalar_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( scalar_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
          postcheck 
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break;
      }

      case c2c_fwd_none: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() ); 
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_fwd_none);

        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        int shared_memory = FFT::shared_memory_size;

        #if DEBUG_FFT_STAGE > 2
          CheckSharedMemory(shared_memory, device_properties);
          cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_NONE<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
          precheck
          block_fft_kernel_C2C_NONE<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input,  complex_output, LP.mem_offsets, workspace);
          postcheck
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break;
      }

      case c2c_fwd_none_Z: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>() + Type<fft_type::c2c>() ); 
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_fwd_none_Z);

        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        int shared_memory = FFT::shared_memory_size;

        #if DEBUG_FFT_STAGE > 1
          CheckSharedMemory(shared_memory, device_properties);
          cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_NONE_XYZ<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory)); 
          precheck
          block_fft_kernel_C2C_NONE_XYZ<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input,  complex_output, LP.mem_offsets, workspace);
          postcheck
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break;
      }

      case c2c_fwd_decrease: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() );  
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_fwd_decrease);

        #if DEBUG_FFT_STAGE > 2
          // the shared mem is mixed between storage, shuffling and FFT. For this kernel we need to add padding to avoid bank conlicts (N/32)
          // For decrease methods, the shared_input > shared_output
          int shared_memory = std::max( FFT::shared_memory_size * LP.threadsPerBlock.z, (LP.mem_offsets.shared_input + LP.mem_offsets.shared_input/32) * (unsigned int)sizeof(complex_type));

          CheckSharedMemory(shared_memory, device_properties);
          cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_DECREASE<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
      
          precheck
          block_fft_kernel_C2C_DECREASE<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
          postcheck 
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break;
      }
      case c2c_fwd_increase: {
  
        using FFT = decltype(FFT_base_arch() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() );  
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_fwd_increase);
        
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;        
        // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>,hipFuncCachePreferShared ));
          // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>, hipSharedMemBankSizeEightByte )
        #if DEBUG_FFT_STAGE > 2
          int shared_memory;
          // Aggregate the transformed frequency data in shared memory so that we can write to global coalesced.
          shared_memory = LP.mem_offsets.shared_output*sizeof(complex_type) + LP.mem_offsets.shared_input*sizeof(complex_type) + FFT::shared_memory_size;

          CheckSharedMemory(shared_memory, device_properties);
    
          // std::cout << "shared_memory " << shared_memory << std::endl;
          // When it is the output dims being smaller, may need a logical or different method
          //FIXME
          bool swap_real_space_quadrants = false;
          if (swap_real_space_quadrants)
          {
            MyFFTRunTimeAssertTrue(false, "c2c_fwd_increase with swap_real_space_quadrants == true, is not yet implemented.");
            cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_INCREASE_SwapRealSpaceQuadrants<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));

            precheck
            block_fft_kernel_C2C_INCREASE_SwapRealSpaceQuadrants<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
            ( complex_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
            postcheck
          }
          else
          {
            cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));

            precheck
            block_fft_kernel_C2C_INCREASE<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
            ( complex_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
            postcheck
          }
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
             
        // do something
        break; 
      }

      case c2c_fwd_increase_Z: {
  
        using FFT = decltype(FFT_base_arch() + Direction<fft_direction::forward>()+ Type<fft_type::c2c>() );  
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_fwd_increase_Z);
        
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;        
        // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>,hipFuncCachePreferShared ));
          // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>, hipSharedMemBankSizeEightByte )
        #if DEBUG_FFT_STAGE > 2
          int shared_memory;
          // Aggregate the transformed frequency data in shared memory so that we can write to global coalesced.
          shared_memory = size_of<FFT>::value*sizeof(complex_type) + FFT::shared_memory_size;

          CheckSharedMemory(shared_memory, device_properties);
    
          // std::cout << "shared_memory " << shared_memory << std::endl;
          // When it is the output dims being smaller, may need a logical or different method
          //FIXME
          bool swap_real_space_quadrants = false;
          if (swap_real_space_quadrants)
          {
            MyFFTRunTimeAssertTrue(false, "c2c_fwd_increase with swap_real_space_quadrants == true, is not yet implemented.");

          }
          else
          {
            cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_INCREASE_Z<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));

            precheck
            block_fft_kernel_C2C_INCREASE_Z<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
            ( complex_input, complex_output, LP.mem_offsets, LP.twiddle_in, LP.Q, workspace);
            postcheck
          }
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
             
        // do something
        break; 
      }

      case c2c_inv_none: {

        using FFT = decltype( FFT_base_arch() + Type<fft_type::c2c>() + Direction<fft_direction::inverse>() );

  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_inv_none);

        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;        
        
          int shared_memory = FFT::shared_memory_size;

          CheckSharedMemory(shared_memory, device_properties);
          cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_NONE<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
        #if DEBUG_FFT_STAGE > 4  
          precheck
          block_fft_kernel_C2C_NONE<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, complex_output, LP.mem_offsets, workspace);
          postcheck
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;         
        #endif
        
      
        // do something
        break; 
      }
      case c2c_inv_none_Z: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::inverse>() + Type<fft_type::c2c>() ); 
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_inv_none_Z);

        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        int shared_memory = FFT::shared_memory_size;
        #if DEBUG_FFT_STAGE > 5
          CheckSharedMemory(shared_memory, device_properties);
          cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_NONE_XY<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory)); 
          precheck
          block_fft_kernel_C2C_NONE_XY<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input,  complex_output, LP.mem_offsets, workspace);
          postcheck
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break;
      }

      case c2c_inv_decrease: {

        using FFT = decltype( FFT_base_arch() + Direction<fft_direction::inverse>()+ Type<fft_type::c2c>() );  
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2c_inv_decrease);

        #if DEBUG_FFT_STAGE > 4
          // the shared mem is mixed between storage, shuffling and FFT. For this kernel we need to add padding to avoid bank conlicts (N/32)
          // For decrease methods, the shared_input > shared_output
          int shared_memory = std::max( FFT::shared_memory_size * LP.threadsPerBlock.z, (LP.mem_offsets.shared_input + LP.mem_offsets.shared_input/32) * (unsigned int)sizeof(complex_type));

          CheckSharedMemory(shared_memory, device_properties);
          cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_DECREASE<FFT,complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
      
          precheck
          block_fft_kernel_C2C_DECREASE<FFT,complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace);
          postcheck 
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break;
      }

      case c2c_inv_increase: {
        MyFFTRunTimeAssertTrue(false, "c2c_inv_increase is not yet implemented.");

        #if DEBUG_FFT_STAGE > 4
        #else
          // Since we skip the memory ops, unlike the other kernels, we need to flip the buffer pinter
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif

        break;
      }

      case c2r_none: {
  
        using FFT = decltype(FFT_base_arch() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() ); 
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_none);
      
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;        cudaErr(error_code);

        int shared_memory = FFT::shared_memory_size;

        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2R_NONE<FFT,complex_type, scalar_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));  
        #if DEBUG_FFT_STAGE > 6
          precheck
          block_fft_kernel_C2R_NONE<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, scalar_output, LP.mem_offsets, workspace);
          postcheck
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
  
        break; 
      }

      case c2r_none_XY: {
  
        using FFT = decltype(FFT_base_arch() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() ); 
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_none_XY);
      
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;        cudaErr(error_code);

        int shared_memory = FFT::shared_memory_size;

        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2R_NONE_XY<FFT,complex_type, scalar_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));  
        #if DEBUG_FFT_STAGE > 6
          precheck
          block_fft_kernel_C2R_NONE_XY<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, scalar_output, LP.mem_offsets, workspace);
          postcheck
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
  
        break; 
      }

      case c2r_decrease: {
        using FFT = decltype(FFT_base_arch() + Direction<fft_direction::inverse>()+ Type<fft_type::c2r>() ); 
  
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, c2r_decrease);
      
        hipError_t error_code = hipSuccess;
        auto workspace = make_workspace<FFT>(error_code); // std::cout << " EPT: " << FFT::elements_per_thread << "kernel " << KernelName[kernel_type] << std::endl;        cudaErr(error_code);

        int shared_memory = std::max( FFT::shared_memory_size * LP.gridDims.z , (LP.mem_offsets.shared_input + LP.mem_offsets.shared_input/32) * (unsigned int)sizeof(complex_type));

        CheckSharedMemory(shared_memory, device_properties);
        cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2R_DECREASE_XY<FFT,complex_type, scalar_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory)); 


        #if DEBUG_FFT_STAGE > 6
          precheck
          block_fft_kernel_C2R_DECREASE_XY<FFT, complex_type, scalar_type><< <LP.gridDims, LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
          ( complex_input, scalar_output, LP.mem_offsets, LP.twiddle_in, LP.Q, workspace);
          postcheck

          transform_stage_completed = TransformStageCompleted::inv;
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;  
        #endif
   
        break;
      }

      case c2r_increase: {
        MyFFTRunTimeAssertTrue(false, "c2r_increase is not yet implemented.");
        break;
      }

      case xcorr_fwd_increase_inv_none: {
  
        using FFT    = decltype( FFT_base_arch() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() ); 
        using invFFT = decltype( FFT_base_arch() + Type<fft_type::c2c>() + Direction<fft_direction::inverse>() ); 
          
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, xcorr_fwd_increase_inv_none);

        hipError_t error_code = hipSuccess;
        auto workspace_fwd = make_workspace<FFT>(error_code); // presumably larger of the two
        cudaErr(error_code);
        error_code = hipSuccess;
        auto workspace_inv = make_workspace<invFFT>(error_code); // presumably larger of the two
        cudaErr(error_code);
  
        int shared_memory = invFFT::shared_memory_size;
        CheckSharedMemory(shared_memory, device_properties);
          // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>,hipFuncCachePreferShared ));
          // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>, hipSharedMemBankSizeEightByte );  
        // FIXME
        #if DEBUG_FFT_STAGE > 2
          bool swap_real_space_quadrants = false;   
          if (swap_real_space_quadrants)
          {
            MyFFTRunTimeAssertTrue(false, "Swapping real space quadrants is not yet implemented.");
            // cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul_SwapRealSpaceQuadrants<FFT,invFFT, complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        

            // precheck
            // block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul_SwapRealSpaceQuadrants<FFT,invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
            // ( (complex_type *)d_ptr.image_to_search, complex_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace_fwd, workspace_inv);
            // postcheck
          }
          else
          {

            cudaErr(hipFuncSetAttribute((void*)block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul<FFT, invFFT, complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
            precheck

            // Right now, because of the n_threads == size_of<FFT> requirement, we are explicitly zero padding, so we need to send an "apparent Q" to know the input size.
            // Could send the actual size, but later when converting to use the transform decomp with different sized FFTs this will be a more direct conversion.
            int apperent_Q = size_of<FFT>::value / fwd_dims_in.y;
   
            block_fft_kernel_C2C_FWD_INCREASE_INV_NONE_ConjMul<FFT, invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
            ( (complex_type *)d_ptr.image_to_search, complex_input, complex_output , LP.mem_offsets,apperent_Q, workspace_fwd, workspace_inv);
            postcheck
          }
        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
            
        // do something
        break; 
      }

      case xcorr_fwd_none_inv_decrease: {
        using FFT    = decltype( FFT_base_arch() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() ); 
        using invFFT = decltype( FFT_base_arch() + Type<fft_type::c2c>() + Direction<fft_direction::inverse>() ); 
          
        LaunchParams LP = SetLaunchParameters(elements_per_thread_complex, xcorr_fwd_none_inv_decrease);
  
        hipError_t error_code = hipSuccess;
        auto workspace_fwd = make_workspace<FFT>(error_code); // presumably larger of the two
        cudaErr(error_code);
        error_code = hipSuccess;
        auto workspace_inv = make_workspace<invFFT>(error_code); // presumably larger of the two
        cudaErr(error_code);
  
        // Max shared memory needed to store the full 1d fft remaining on the forward transform
        unsigned int shared_memory = FFT::shared_memory_size + (unsigned int)sizeof(complex_type) * LP.mem_offsets.physical_x_input;
        // shared_memory = std::max( shared_memory, std::max( invFFT::shared_memory_size * LP.threadsPerBlock.z, (LP.mem_offsets.shared_input + LP.mem_offsets.shared_input/32) * (unsigned int)sizeof(complex_type)));

        CheckSharedMemory(shared_memory, device_properties);
 
        
          // cudaErr(hipFuncSetCacheConfig( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>,hipFuncCachePreferShared ));
          // hipFuncSetSharedMemConfig ( (void*)block_fft_kernel_C2C_INCREASE<FFT,complex_type>, hipSharedMemBankSizeEightByte );  
        // FIXME
        #if DEBUG_FFT_STAGE > 2


          bool swap_real_space_quadrants = false;   
          if (swap_real_space_quadrants)
          {
            // cudaErr(hipFuncSetAttribute((void*)_INV_DECREASE_ConjMul_SwapRealSpaceQuadrants<FFT,invFFT, complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
            MyFFTDebugAssertFalse(swap_real_space_quadrants, "Swap real space quadrants not yet implemented in xcorr_fwd_none_inv_decrease.");

            // precheck
            // _INV_DECREASE_ConjMul_SwapRealSpaceQuadrants<FFT,invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
            // ( (complex_type *)d_ptr.image_to_search, complex_input, complex_output, LP.mem_offsets, LP.twiddle_in,LP.Q, workspace_fwd, workspace_inv);
            // postcheck
          }
          else
          {
            cudaErr(hipFuncSetAttribute((void*)_INV_DECREASE_ConjMul<FFT, invFFT, complex_type>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));        
            // Right now, because of the n_threads == size_of<FFT> requirement, we are explicitly zero padding, so we need to send an "apparent Q" to know the input size.
            // Could send the actual size, but later when converting to use the transform decomp with different sized FFTs this will be a more direct conversion.
            int apparent_Q = size_of<FFT>::value / inv_dims_out.y;
            precheck
            _INV_DECREASE_ConjMul<FFT, invFFT, complex_type><< <LP.gridDims,  LP.threadsPerBlock, shared_memory, hipStreamPerThread>> >
            ( (complex_type *)d_ptr.image_to_search, complex_input, complex_output , LP.mem_offsets, LP.twiddle_in, apparent_Q, workspace_fwd, workspace_inv);
            postcheck
          }
          transform_stage_completed = TransformStageCompleted::fwd;

        #else
          is_in_buffer_memory = ! is_in_buffer_memory;
        #endif
            
        // do something
        break; 
      } // end case xcorr_fwd_none_inv_decrease    
      default:
        // throw something
        break;
  
    }
  }


    



  // 
} // end set and launc kernel

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Some helper functions that are annoyingly long to have in the header.
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::GetTransformSize(KernelType kernel_type)
{
  // Set member variable transform_size.N (.P .L .Q)

  if ( IsR2CType(kernel_type) )
  {
    AssertDivisibleAndFactorOf2( std::max(fwd_dims_in.x, fwd_dims_out.x), std::min(fwd_dims_in.x, fwd_dims_out.x) );
  }
  else if ( IsC2RType(kernel_type) )
  {
    // FIXME
    if (kernel_type == c2r_decrease)
    {
      AssertDivisibleAndFactorOf2( std::max(inv_dims_in.x, inv_dims_out.x), std::max(inv_dims_in.x, inv_dims_out.x) );
    }
    else
    {
      AssertDivisibleAndFactorOf2( std::max(inv_dims_in.x, inv_dims_out.x), std::min(inv_dims_in.x, inv_dims_out.x) );
    }
  }
  else
  {
    // C2C type
    if ( IsForwardType( kernel_type) )
    {
      switch (transform_dimension)
      {
        case 1: { AssertDivisibleAndFactorOf2( std::max(fwd_dims_in.x, fwd_dims_out.x),  std::min(fwd_dims_in.x, fwd_dims_out.x) ); break; }
        case 2: { 
          if (kernel_type == xcorr_fwd_increase_inv_none)
          {
            // FIXME
            AssertDivisibleAndFactorOf2( std::max(fwd_dims_in.y, fwd_dims_out.y),  std::max(fwd_dims_in.y, fwd_dims_out.y) ); 
          }
          else
          {
            AssertDivisibleAndFactorOf2( std::max(fwd_dims_in.y, fwd_dims_out.y),  std::min(fwd_dims_in.y, fwd_dims_out.y) ); 
          }
          break; 
        }
        case 3:  {
          if (IsTransormAlongZ(kernel_type)) 
          {
            AssertDivisibleAndFactorOf2( std::max(fwd_dims_in.z, fwd_dims_out.z),  std::min(fwd_dims_in.z, fwd_dims_out.z) ); 
          }
          else
          {
            AssertDivisibleAndFactorOf2( std::max(fwd_dims_in.y, fwd_dims_out.y),  std::min(fwd_dims_in.y, fwd_dims_out.y) ); 
          }
          
          break;
        }

        default: { MyFFTDebugAssertTrue(false, "ERROR: Invalid transform dimension for c2c fwd type.\n"); }
      }
    }
    else
    {
      switch (transform_dimension)
      {
        case 1: { AssertDivisibleAndFactorOf2( std::max(inv_dims_in.x, inv_dims_out.x),  std::min(inv_dims_in.x, inv_dims_out.x) ); break; }
        case 2: { 
          if (kernel_type == xcorr_fwd_none_inv_decrease)
          {
            // FIXME, for now using full transform
            AssertDivisibleAndFactorOf2( std::max(inv_dims_in.y, inv_dims_out.y),  std::max(inv_dims_in.y, inv_dims_out.y) );
          }
          else
          {
            AssertDivisibleAndFactorOf2( std::max(inv_dims_in.y, inv_dims_out.y),  std::min(inv_dims_in.y, inv_dims_out.y) ); 
          }
          break; 
        }
        case 3:  {
          if (IsTransormAlongZ(kernel_type)) 
          {
            AssertDivisibleAndFactorOf2( std::max(inv_dims_in.z, inv_dims_out.z),  std::min(inv_dims_in.z, inv_dims_out.z) ); 
          }
          else
          {
            AssertDivisibleAndFactorOf2( std::max(inv_dims_in.y, inv_dims_out.y),  std::min(inv_dims_in.y, inv_dims_out.y) ); 
          }
          
          break;
        }

        default: { MyFFTDebugAssertTrue(false, "ERROR: Invalid transform dimension for c2c inverse type.\n"); }
      }
    }


  }

} // end GetTransformSize function

template <class ComputeType, class InputType, class OutputType>
void FourierTransformer<ComputeType, InputType, OutputType>::GetTransformSize_thread(KernelType kernel_type, int thread_fft_size)
{

  transform_size.P = thread_fft_size;
  
  switch (kernel_type)
  {
    case r2c_decomposed:
      transform_size.N = fwd_dims_in.x;
      break;
    case r2c_decomposed_transposed:
      transform_size.N = fwd_dims_in.x;
      break; 
    case c2c_decomposed:
    // FIXME fwd vs inv
      if (fwd_dims_in.y == 1) transform_size.N = fwd_dims_in.x;
      else transform_size.N = fwd_dims_in.y;
      break;
    case c2r_decomposed:
      transform_size.N = inv_dims_out.x;
      break;
    case c2r_decomposed_transposed:
      transform_size.N = inv_dims_out.x;
      break;
    case xcorr_decomposed:
        // FIXME fwd vs inv
      if (fwd_dims_in.y == 1) transform_size.N = fwd_dims_out.x; // FIXME should probably throw an error for now.
      else transform_size.N = fwd_dims_out.y; // does fwd_dims_in make sense?

      break;
    default:
      std::cerr << "Function GetTransformSize_thread does not recognize the kernel type ( " << KernelName[kernel_type] << " )" << std::endl;
      exit(-1);
  }

  if (transform_size.N % transform_size.P != 0) { std::cerr << "Thread based decompositions must factor by thread_fft_size (" << thread_fft_size << ") in the current implmentations." << std::endl; exit(-1); }
  transform_size.Q = transform_size.N / transform_size.P;
} // end GetTransformSize_thread function

template <class ComputeType, class InputType, class OutputType>
LaunchParams FourierTransformer<ComputeType, InputType, OutputType>::SetLaunchParameters(const int& ept, KernelType kernel_type, bool do_forward_transform)
{
  /*
    Assuming:
    1) r2c/c2r imply forward/inverse transform. 
       c2c_padded implies forward transform.
    2) for 2d or 3d transforms the x/y dimensions are transposed in momentum space during store on the 1st set of 1ds transforms.
    3) if 1d then z = y = 1.

    threadsPerBlock = size/threads_per_fft (for thread based transforms)
                    = size of fft ( for block based transforms ) NOTE: Something in cufftdx seems to be very picky about this. Launching > threads seem to cause problems.
    gridDims = number of 1d FFTs, placed on blockDim perpendicular
    shared_input/output = number of elements reserved in dynamic shared memory. TODO add a check on minimal (48k) and whether this should be increased (depends on Arch)
    physical_x_input/output = number of elements along the fast (x) dimension, depends on fftw padding && whether the memory is currently transposed in x/y
    twiddle_in = +/- 2*PI/Largest dimension : + for the inverse transform
    Q = number of sub-transforms
  */
  LaunchParams L;

  // This is the same for all kernels as set in AssertDivisibleAndFactorOf2()
  L.Q = transform_size.Q;

  // Set the twiddle factor, only differ in sign between fwd/inv transforms.
  // For mixed kernels (eg. xcorr_* the size type is defined by where the size change happens.
  // FIXME fwd_increase (oversampling) xcorr -> inv decrease (peak search) is a likely algorithm, that will not fit with this logic.
  SizeChangeType size_change_type;
  if ( IsForwardType(kernel_type) ) 
  {
    size_change_type = fwd_size_change_type;
    L.twiddle_in = L.twiddle_in = -2*PIf/transform_size.N ;
  }
  else 
  {
    size_change_type = inv_size_change_type;
    L.twiddle_in = L.twiddle_in = 2*PIf/transform_size.N ;
  }

  // Set the thread block dimensions
  if ( IsThreadType(kernel_type) ) {
      L.threadsPerBlock = dim3(transform_size.Q, 1, 1);
  }
  else {
    if (size_change_type == decrease) {
      L.threadsPerBlock = dim3(transform_size.P/ept, 1, transform_size.Q);
    }
    else {
      // In the current xcorr methods that have INCREASE, explicit zero padding is used, so this will be overridden (overrode?) with transform_size.N
      L.threadsPerBlock = dim3(transform_size.P/ept, 1, 1);
    }
  }

  
  // Set the shared mem sizes, which depend on the size_change_type
  switch (size_change_type)
  {
    case no_change: {
      // no shared memory is needed outside that for the FFT itself.
      // For C2C kernels of size_type increase, the shared output may be reset below in order to store for coalesced global writes.
      L.mem_offsets.shared_input  = 0;
      L.mem_offsets.shared_output = 0;
      break;
    }
    case decrease: {
      // Prior to reduction, we must be able to store the full transform. An alternate algorithm with multiple reads would relieve this dependency and 
      // may be worth considering if L2 cache residence on Ampere is an effective way to reduce repeated Globabl memory access.
      // Note: that this shared memory is not static, in the sense that it is used both for temporory fast storage, as well as the calculation of the FFT. The max of those two requirments is calculated per kernel.
      L.mem_offsets.shared_input  = transform_size.N;
      if (IsR2CType(kernel_type)) { L.mem_offsets.shared_output = 0; } 
      else { L.mem_offsets.shared_output = transform_size.N; } // TODO this line is just from case increase, haven't thought about it.
      break;
    }
    case increase: {
      // We want to re-use the input memory as we loop over construction of the full FFT. This shared memory is independent of the
      // memory used for the FFT itself.
      L.mem_offsets.shared_input  = transform_size.P;
      if (IsR2CType(kernel_type)) { L.mem_offsets.shared_output = 0; } 
      else { L.mem_offsets.shared_output = transform_size.N; }
      // Note: This is overwritten in the C2C methods as it depends on 1d vs 2d and fwd vs inv.
      break;
    }
    default: {
      MyFFTDebugAssertTrue(false, "Unknown size_change_type ( " + std::to_string(size_change_type) + " )");
    }
  } // switch on size change

  
  // Set the grid dimensions and pixel pitch
  if (IsR2CType(kernel_type)) 
  {
    L.gridDims = dim3(1, fwd_dims_in.y, fwd_dims_in.z); 
    L.mem_offsets.physical_x_input = fwd_dims_in.w*2; // scalar type, natural 
    L.mem_offsets.physical_x_output = fwd_dims_out.w;

  } 
  else if (IsC2RType(kernel_type)) 
  {
    // This is always the last op, so if there is a size change, it will have happened once on C2C, reducing the number of blocks
    L.gridDims = dim3(1, inv_dims_out.y, inv_dims_out.z);
    L.mem_offsets.physical_x_input = inv_dims_in.w;
    L.mem_offsets.physical_x_output = inv_dims_out.w*2;      
  }
  else // C2C type
  {
    // All dimensions have the same physical x dims for C2C transforms
    if ( IsForwardType(kernel_type) ) 
    {
      // If 1d, this is implicitly a complex valued input, s.t. fwd_dims_in.x = fwd_dims_in.w.) But if fftw_padding is allowed false this may not be true.
      L.mem_offsets.physical_x_input =  fwd_dims_in.w; 
      L.mem_offsets.physical_x_output = fwd_dims_out.w;
    }
    else
    {
      L.mem_offsets.physical_x_input =  inv_dims_in.w;
      L.mem_offsets.physical_x_output = inv_dims_out.w;
    }

    switch (transform_dimension)
    {
      case 1: {
        L.gridDims = dim3(1, 1, 1);
        break;
      }
      case 2: {
        if ( IsForwardType(kernel_type) ) 
        {
          L.gridDims = dim3(1, fwd_dims_out.w, fwd_dims_out.z);
        }
        else
        {
          L.gridDims = dim3(1, inv_dims_out.w, inv_dims_out.z);
        }
        break;
      }
      case 3: {
        if (IsTransormAlongZ(kernel_type))
        {
          // When transforming along the Z-dimension, The Z grid dimensions for a 3d kernel are used to indicate the transposed x coordinate.
          if ( IsForwardType(kernel_type) ) 
          {
            L.gridDims = dim3(1, fwd_dims_out.y, fwd_dims_out.w);
          }
          else
          {
            L.gridDims = dim3(1, inv_dims_out.y, inv_dims_out.w);
          }
        }
        else
        {
          if ( IsForwardType(kernel_type) ) 
          {
            L.gridDims = dim3(1, fwd_dims_out.w, fwd_dims_out.z);
          }
          else
          {
            L.gridDims = dim3(1, inv_dims_out.w, inv_dims_out.z);
          }
        }          
        break;
      } // 3 dimensional case
      default: {
        MyFFTDebugAssertTrue(false, "Unknown transform_dimension ( " + std::to_string(transform_dimension) + " )");
      }
    }
  }

  // FIXME
  // Some shared memory over-rides
  if (  kernel_type == c2c_inv_decrease || kernel_type == c2c_inv_increase )
  {
    L.mem_offsets.shared_output = inv_dims_out.y;
  }

  // FIXME
  // Some xcorr overrides TODO try the DECREASE approcae
  if (kernel_type == xcorr_fwd_increase_inv_none)
  {
    // FIXME not correct for 3D
    L.threadsPerBlock = dim3(transform_size.N/ept, 1, 1);
  }

  if (kernel_type == xcorr_fwd_none_inv_decrease)
  {
    // FIXME not correct for 3D

    L.threadsPerBlock = dim3(transform_size.N/ept, 1, 1);
    // FIXME
    L.gridDims = dim3(1, fwd_dims_out.w, 1);
    L.mem_offsets.physical_x_input = inv_dims_in.y;
    L.mem_offsets.physical_x_output = inv_dims_out.y;
  }
 
  return L;
}

} // namespace fast_FFT



